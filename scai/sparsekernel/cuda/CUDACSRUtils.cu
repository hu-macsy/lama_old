#include "hip/hip_runtime.h"
/**
 * @file CUDACSRUtils.cu
 *
 * @license
 * Copyright (c) 2009-2016
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * This file is part of the SCAI framework LAMA.
 *
 * LAMA is free software: you can redistribute it and/or modify it under the
 * terms of the GNU Affero General Public License as published by the Free
 * Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * LAMA is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU Affero General Public License for
 * more details.
 *
 * You should have received a copy of the GNU Affero General Public License
 * along with LAMA. If not, see <http://www.gnu.org/licenses/>.
 *
 * Other Usage
 * Alternatively, this file may be used in accordance with the terms and
 * conditions contained in a signed written agreement between you and
 * Fraunhofer SCAI. Please contact our distributor via info[at]scapos.com.
 * @endlicense
 *
 * @brief Implementation of CSR utilities with CUDA
 * @author Bea Hornef, Thomas Brandes, Jiri Kraus
 * @date 04.07.2012
 */

// hpp
#include <scai/sparsekernel/cuda/CUDACSRUtils.hpp>

// local library
#include <scai/sparsekernel/cuda/CUDACSRUtils.hpp>
#include <scai/sparsekernel/cuda/CUDACOOUtils.hpp>
#include <scai/sparsekernel/CSRKernelTrait.hpp>

// internal scai library
#include <scai/utilskernel/cuda/CUDAUtils.hpp>

#include <scai/hmemo/Memory.hpp>
#include <scai/kregistry/KernelRegistry.hpp>

#include <scai/tasking/cuda/CUDAStreamSyncToken.hpp>

#include <scai/tracing.hpp>

#include <scai/common/cuda/CUDATexVector.hpp>
#include <scai/common/cuda/CUDASettings.hpp>
#include <scai/common/cuda/CUDAUtils.hpp>
#include <scai/common/SCAITypes.hpp>
#include <scai/common/bind.hpp>
#include <scai/common/Constants.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/launchHelper.hpp>

#include <scai/common/macros/unused.hpp>

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// thrust
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>

// Parameters for Matrix Multiplication
#define NUM_HASH_RETRIES 16
#define NUM_ELEMENTS_PER_CHUNK 512
#define NUM_ELEMENTS_IN_SHARED 512
#define NUM_BLOCKS 9216
#define NUM_THREADS 32
#define NUM_WARPS NUM_THREADS/32
#define HASH_A 684
#define HASH_B 46165
#define HASH_P 88651
#define HASH_C0 1
#define HASH_C1 1
#define NUM_CHUNKS_PER_WARP 128

using namespace scai::common;
using namespace scai::hmemo;

namespace scai
{

using utilskernel::CUDAUtils;

using tasking::SyncToken;
using tasking::CUDAStreamSyncToken;

using common::CUDASettings;


namespace sparsekernel
{

SCAI_LOG_DEF_LOGGER( CUDACSRUtils::logger, "CUDA.CSRUtils" )

// not yet: __device__ const IndexType cudaNIndex = std::numeric_limits<IndexType>::max();

#define cudaNIndex static_cast<IndexType>( -1 )

IndexType CUDACSRUtils::sizes2offsets( IndexType array[], const IndexType n )
{
    SCAI_LOG_INFO( logger, "sizes2offsets " << " #n = " << n )
    SCAI_CHECK_CUDA_ACCESS
    thrust::device_ptr<IndexType> array_ptr( array );
    thrust::exclusive_scan( array_ptr, array_ptr + n + 1, array_ptr );
    thrust::host_vector<IndexType> numValues( array_ptr + n, array_ptr + n + 1 );
    return numValues[0];
}

/* --------------------------------------------------------------------------- */
/*     CUDA Kernels                                                            */
/* --------------------------------------------------------------------------- */

__global__
static void offsets2sizes_kernel( IndexType sizes[], const IndexType offsets[], const IndexType n )
{
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < n )
    {
        sizes[i] = offsets[i + 1] - offsets[i];
    }
}

/* --------------------------------------------------------------------------- */
/*     offsets2sizes                                                           */
/* --------------------------------------------------------------------------- */

void CUDACSRUtils::offsets2sizes( IndexType sizes[], const IndexType offsets[], const IndexType n )
{
    SCAI_REGION( "CUDA.CSRUtils.offsets2sizes" )
    SCAI_LOG_INFO( logger, "offsets2sizes " << " #n = " << n )
    SCAI_CHECK_CUDA_ACCESS
    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( n, dimBlock.x );
    offsets2sizes_kernel <<< dimGrid, dimBlock>>>( sizes, offsets, n );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "offsets2sizes" )
}

/* --------------------------------------------------------------------------- */
/*     hasDiagonalProperty                                                     */
/* --------------------------------------------------------------------------- */

template<typename ValueType>
struct identic_functor
{
    __host__ __device__
    double operator()( thrust::tuple<ValueType, ValueType> x )
    {
        return thrust::get < 0 > ( x ) == thrust::get < 1 > ( x );
    }
};

//trivial kernel to check diagonal property
__global__ void hasDiagonalProperty_kernel(
    const IndexType numDiagonals,
    const IndexType ia[],
    const IndexType ja[],
    bool* hasProperty )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if ( i >= numDiagonals )
    {
        return;
    }

    if ( ! ( *hasProperty ) )
    {
        return;
    }

    if ( ia[i] == ia[i + 1] )
    {
        *hasProperty = false;
    }
    else if ( ja[ia[i]] != i )
    {
        *hasProperty = false;
    }
}

bool CUDACSRUtils::hasDiagonalProperty( const IndexType numDiagonals, const IndexType csrIA[], const IndexType csrJA[] )
{
    SCAI_REGION( "CUDA.CSRUtils.hasDiagonalProperty" )

    if ( numDiagonals == 0 )
    {
        return true;
    }

    SCAI_CHECK_CUDA_ACCESS
    //make grid
    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimGrid( ( numDiagonals - 1 ) / blockSize + 1, 1, 1 );// = makeGrid( numDiagonals, blockSize );
    dim3 dimBlock( blockSize, 1, 1 );
    bool* d_hasProperty;
    bool hasProperty;
    SCAI_CUDA_RT_CALL( hipMalloc( ( void** ) &d_hasProperty, sizeof( bool ) ),
                       "allocate 4 bytes on the device for the result of hasDiagonalProperty_kernel" )
    SCAI_CUDA_RT_CALL( hipMemset( d_hasProperty, 1, sizeof( bool ) ), "memset bool hasProperty = true" )
    hasDiagonalProperty_kernel <<< dimGrid, dimBlock>>>( numDiagonals, csrIA, csrJA, d_hasProperty );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hasDiagonalProperty failed: are ia and ja correct?" )
    SCAI_CUDA_RT_CALL( hipMemcpy( &hasProperty, d_hasProperty, sizeof( bool ), hipMemcpyDeviceToHost ),
                       "copy the result of hasDiagonalProperty_kernel to host" )
    return hasProperty;
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::convertCSR2CSC(
    IndexType cscIA[],
    IndexType cscJA[],
    ValueType cscValues[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[],
    IndexType numRows,
    IndexType numColumns,
    IndexType numValues )
{
    SCAI_REGION( "CUDA.CSRUtils.CSR2CSC" )
    SCAI_LOG_INFO( logger, "convertCSR2CSC of " << numRows << " x " << numColumns << ", nnz = " << numValues )
    // Sort the csrJA ( same as cooJA ), apply it to cooIA and cooValues
    IndexType* cooIA;
    SCAI_CUDA_RT_CALL( hipMalloc( &cooIA, sizeof( IndexType ) * numValues ),
                       "allocate temp for cooIA" )
    // Step 1 : build COO storage,  cooIA (to do), cooJA ( = csrJA ), cooValues ( = csrValues )
    //          -> translate the csrIA offset array to a cooIA array
    const IndexType numDiagonals = 0;// not supported yet
    CUDACOOUtils::offsets2ia( cscJA, numValues, csrIA, numRows, numDiagonals );
    // switch cooIA and cooJA, copy values and resort
    CUDAUtils::set( cooIA, csrJA, numValues, utilskernel::reduction::COPY );
    CUDAUtils::set( cscValues, csrValues, numValues, utilskernel::reduction::COPY );
    thrust::device_ptr<IndexType> ja_d( cooIA );
    thrust::device_ptr<ValueType> values_d( cscValues );
    thrust::device_ptr<IndexType> ia_d( cscJA );
    // sort by column indexes in ascending order
    // zip_iterator used to resort cscValues and cscJA in one step
    thrust::stable_sort_by_key( ja_d, ja_d + numValues,
                                thrust::make_zip_iterator( thrust::make_tuple( values_d, ia_d ) ) );
    // cscJA is now sorted, can become an offset array
    CUDACOOUtils::ia2offsets( cscIA, numColumns, 0, cooIA, numValues );
    SCAI_CUDA_RT_CALL( hipFree( cooIA ), "free tmp cooIA" )
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void scale_kernel(
    ValueType* result,
    const ValueType* y_d,
    const ValueType beta,
    IndexType numRows )
{
    // result = beta * y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        result[i] = beta * y_d[i];
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_beta_zero(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows )
{
    // result = alpha * A * x_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = alpha * value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_alpha_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType beta,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows )
{
    // result = A * x_d + beta * y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = beta * y_d[i];
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_beta_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows )
{
    // result = alpha * A * x_d + y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = y_d[i];
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = alpha * value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_alpha_one_beta_zero(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows )
{
    // result = A * x_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void assign_kernel(
    ValueType* result,
    const ValueType* y_d,
    IndexType numRows )
{
    // result = y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        result[i] = y_d[i];
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_alpha_one_beta_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows )
{
    // result = A * x_d + y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = y_d[i];
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType beta,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows )
{
    // result = alpha * A * x_d + beta * y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = beta * y_d[i];
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = alpha * value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_alpha_one_beta_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows,
    IndexType numColumns )
{
    // result = x_d * A + y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType summand = y_d[i];
        ValueType value = 0.0;

        for ( IndexType j = 0; j < numRows; ++j )
        {
            const IndexType rowStart = csrIA[j];
            const IndexType rowEnd = csrIA[j + 1];

            for ( IndexType k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_alpha_one_beta_zero(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows,
    IndexType numColumns )
{
    // result = x_d * A
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( IndexType j = 0; j < numRows; ++j )
        {
            const IndexType rowStart = csrIA[j];
            const IndexType rowEnd = csrIA[j + 1];

            for ( IndexType k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_alpha_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType beta,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows,
    IndexType numColumns )
{
    // result = x_d * A + beta * y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType summand = beta * y_d[i];
        ValueType value = 0.0;

        for ( IndexType j = 0; j < numRows; ++j )
        {
            const IndexType rowStart = csrIA[j];
            const IndexType rowEnd = csrIA[j + 1];

            for ( IndexType k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_beta_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows,
    IndexType numColumns )
{
    // result = alpha * x_d * A + y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( IndexType j = 0; j < numRows; ++j )
        {
            const IndexType rowStart = csrIA[j];
            const IndexType rowEnd = csrIA[j + 1];

            for ( IndexType k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = alpha * value + y_d[i];
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_beta_zero(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows,
    IndexType numColumns )
{
    // result = alpha * x_d * A
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( IndexType j = 0; j < numRows; ++j )
        {
            const IndexType rowStart = csrIA[j];
            const IndexType rowEnd = csrIA[j + 1];

            for ( IndexType k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = alpha * value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType beta,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    IndexType numRows,
    IndexType numColumns )
{
    // result = alpha * x_d * A + beta * y_d
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType summand = beta * y_d[i];
        ValueType value = 0.0;

        for ( IndexType j = 0; j < numRows; ++j )
        {
            const IndexType rowStart = csrIA[j];
            const IndexType rowEnd = csrIA[j + 1];

            for ( IndexType k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = alpha * value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void sparse_gemv_kernel_alpha_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    const IndexType* rowIndexes,
    IndexType numRows )
{
    // result = A * x_d
    const IndexType ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( ii < numRows )
    {
        IndexType i = rowIndexes[ii];
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] += value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void sparse_gemv_kernel(
    ValueType* result,
    const ValueType* x_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    const IndexType* rowIndexes,
    IndexType numRows )
{
    // result = alpha * A * x_d
    const IndexType ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( ii < numRows )
    {
        IndexType i = rowIndexes[ii];
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] += alpha * value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void sparse_gevm_kernel(
    ValueType* result,
    const ValueType* x_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const IndexType* csrIA,
    const IndexType* csrJA,
    const IndexType* rowIndexes,
    IndexType numColumns,
    IndexType numNonZeroRows )
{
    // result += alpha * x_d * A
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( IndexType jj = 0; jj < numNonZeroRows; ++jj )
        {
            IndexType j = rowIndexes[jj];
            const IndexType rowStart = csrIA[j];
            const IndexType rowEnd = csrIA[j + 1];

            for ( IndexType k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] += alpha * value;
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  scaleRows                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType, typename OtherValueType>
__global__
void scaleRowsKernel(
    ValueType* values,
    const IndexType* ia,
    const IndexType numRows,
    const OtherValueType* diagonal )
{
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType tmp = static_cast<OtherValueType>( diagonal[i] );

        for ( IndexType j = ia[i]; j < ia[i + 1]; ++j )
        {
            values[j] *= tmp;
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType1, typename ValueType2>
void CUDACSRUtils::scaleRows(
    ValueType1 csrValues[],
    const IndexType csrIA[],
    const IndexType numRows,
    const ValueType2 values[] )
{
    SCAI_REGION( "CUDA.CSRUtils.scaleRows" )
    SCAI_LOG_INFO( logger, "scaleRows<" << TypeTraits<ValueType1>::id() << ","
                   << TypeTraits<ValueType2>::id() << ">"
                   << ", numrows= " << numRows )
    SCAI_CHECK_CUDA_ACCESS
    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );
    scaleRowsKernel <<< dimGrid, dimBlock>>>( csrValues, csrIA, numRows, values );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "CSRUtils:scaleRowsKernel FAILED" )
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::normalGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType SCAI_UNUSED( numColumns ),
    const IndexType SCAI_UNUSED( nnz ),
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_REGION( "CUDA.CSRUtils.normalGEMV" )
    SCAI_LOG_INFO( logger, "normalGEMV<" << TypeTraits<ValueType>::id() << ">" <<
                   " result[ " << numRows << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )
    SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )
    SCAI_CHECK_CUDA_ACCESS
    hipStream_t stream = 0; // default stream if no syncToken is given
    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );
    bool useTexture = CUDASettings::useTexture();
    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        // asynchronous execution takes other stream and will not synchronize later
        stream = syncToken->getCUDAStream();
    }

    SCAI_LOG_INFO( logger, "Start normal_gemv_kernel<" << TypeTraits<ValueType>::id()
                   << ", useTexture = " << useTexture << ">" );

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == constants::ONE && beta == constants::ONE )
        {
            // result = A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one_beta_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_alpha_one_beta_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == constants::ONE && beta == constants::ZERO )
        {
            // result = A * x_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one_beta_zero<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_alpha_one_beta_zero<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == constants::ZERO && beta == constants::ONE )
        {
            // result = y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( assign_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            assign_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>( result, y, numRows );
        }
        else if ( alpha == constants::ONE )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, beta, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == constants::ZERO )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( scale_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            scale_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>( result, y, beta, numRows );
        }
        else if ( beta == constants::ONE )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_beta_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_beta_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows );
        }
        else if ( beta == constants::ZERO )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_beta_zero<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_beta_zero<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows );
        }
        else
        {
            // result = alpha * A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, beta, csrValues, csrIA, csrJA, numRows );
        }
    }
    else
    {
        if ( alpha == constants::ONE && beta == constants::ONE )
        {
            // result = A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one_beta_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_alpha_one_beta_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == constants::ONE && beta == constants::ZERO )
        {
            // result = A * x_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one_beta_zero<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_alpha_one_beta_zero<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == constants::ZERO && beta == constants::ONE )
        {
            // result = y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( assign_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            assign_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>( result, y, numRows );
        }
        else if ( alpha == constants::ONE )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, beta, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == constants::ZERO )
        {
            // result = beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( scale_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            scale_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>( result, y, beta, numRows );
        }
        else if ( beta == constants::ONE )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_beta_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_beta_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows );
        }
        else if ( beta == constants::ZERO )
        {
            // result = alpha * A * x_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_beta_zero<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel_beta_zero<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows );
        }
        else
        {
            // result = alpha * A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gemv_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, beta, csrValues, csrIA, csrJA, numRows );
        }
    }

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "normalGEMV, stream = " << stream )
        SCAI_LOG_DEBUG( logger, "normalGEMV<" << TypeTraits<ValueType>::id() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorUnbindTexture( x );
        }
        else
        {
            // get routine with the right signature
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;
            // delay unbind until synchroniziaton
            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::normalGEVM(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_LOG_INFO( logger, "normalGEVM<" << TypeTraits<ValueType>::id() << ">" <<
                   " result[ " << numColumns << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )
    SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )
    SCAI_CHECK_CUDA_ACCESS
    hipStream_t stream = 0; // default stream if no syncToken is given
    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numColumns, dimBlock.x );
    bool useTexture = CUDASettings::useTexture();
    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_LOG_INFO( logger, "Start normal_gevm_kernel<" << TypeTraits<ValueType>::id()
                   << ", useTexture = " << useTexture << ">" );

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == constants::ONE && beta == constants::ONE )
        {
            // result = A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one_beta_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_alpha_one_beta_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == constants::ONE && beta == constants::ZERO )
        {
            // result = A * x_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one_beta_zero<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_alpha_one_beta_zero<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == constants::ZERO && beta == constants::ONE )
        {
            // result = y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( assign_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            assign_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>( result, y, numColumns );
        }
        else if ( alpha == constants::ONE )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, beta, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == constants::ZERO )
        {
            // result = beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( scale_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            scale_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>( result, y, beta, numColumns );
        }
        else if ( beta == constants::ONE )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_beta_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_beta_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( beta == constants::ZERO )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_beta_zero<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_beta_zero<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else
        {
            // result = alpha * A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, beta, csrValues, csrIA, csrJA, numRows, numColumns );
        }
    }
    else
    {
        if ( alpha == constants::ONE && beta == constants::ONE )
        {
            // result = A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one_beta_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_alpha_one_beta_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == constants::ONE && beta == constants::ZERO )
        {
            // result = A * x_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one_beta_zero<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_alpha_one_beta_zero<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == constants::ZERO && beta == constants::ONE )
        {
            // result = y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( assign_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            assign_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>( result, y, numColumns );
        }
        else if ( alpha == constants::ONE )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, beta, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == constants::ZERO )
        {
            // result = beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( scale_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            scale_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>( result, y, beta, numColumns );
        }
        else if ( beta == constants::ONE )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_beta_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_beta_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( beta == constants::ZERO )
        {
            // result = alpha * A * x_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_beta_zero<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel_beta_zero<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else
        {
            // result = alpha * A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
            normal_gevm_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, beta, csrValues, csrIA, csrJA, numRows, numColumns );
        }
    }

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "normalGEVM, stream = " << stream )
        SCAI_LOG_DEBUG( logger, "normalGEVM<" << TypeTraits<ValueType>::id() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorUnbindTexture( x );
        }
        else
        {
            // get routine with the right signature
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;
            // delay unbind until synchroniziaton
            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::sparseGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const IndexType numNonZeroRows,
    const IndexType rowIndexes[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_REGION( "CUDA.CSRUtils.sparseGEMV" )
    SCAI_LOG_INFO( logger,
                   "sparseGEMV<" << TypeTraits<ValueType>::id() << ">" << ", #non-zero rows = " << numNonZeroRows )
    SCAI_CHECK_CUDA_ACCESS
    hipStream_t stream = 0;
    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    const int blockSize = CUDASettings::getBlockSize( numNonZeroRows );
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numNonZeroRows, dimBlock.x );
    bool useTexture = CUDASettings::useTexture();

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == constants::ONE )
        {
            sparse_gemv_kernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numNonZeroRows );
        }
        else
        {
            sparse_gemv_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numNonZeroRows );
        }
    }
    else
    {
        if ( alpha == constants::ONE )
        {
            sparse_gemv_kernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numNonZeroRows );
        }
        else
        {
            sparse_gemv_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numNonZeroRows );
        }
    }

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "sparseGEMV, stream = " << stream )
        SCAI_LOG_INFO( logger, "sparseGEMV<" << TypeTraits<ValueType>::id() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorUnbindTexture( x );
        }
        else
        {
            // get routine with the right signature
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;
            // delay unbind until synchroniziaton
            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::sparseGEVM(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const IndexType numColumns,
    const IndexType numNonZeroRows,
    const IndexType rowIndexes[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_LOG_INFO( logger,
                   "sparseGEVM<" << TypeTraits<ValueType>::id() << ">" << ", #non-zero rows = " << numNonZeroRows )
    SCAI_CHECK_CUDA_ACCESS
    hipStream_t stream = 0;
    // check if asynchronous execution is wanted
    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    const int blockSize = CUDASettings::getBlockSize( numNonZeroRows );
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numNonZeroRows, dimBlock.x );
    bool useTexture = CUDASettings::useTexture();

    if ( useTexture )
    {
        vectorBindTexture( x );
        SCAI_LOG_DEBUG( logger, "sparse_gevm_kernel<useTexture=true>" )
        sparse_gevm_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
        ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numColumns, numNonZeroRows );
    }
    else
    {
        SCAI_LOG_DEBUG( logger, "sparse_gevm_kernel<useTexture=false>" )
        sparse_gevm_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
        ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numColumns, numNonZeroRows );
    }

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "sparseGEVM, stream = " << stream )
        SCAI_LOG_INFO( logger, "sparseGEVM<" << TypeTraits<ValueType>::id() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorUnbindTexture( x );
        }
        else
        {
            // get routine with the right signature
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;
            // delay unbind until synchroniziaton
            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */
/*                          Jacobi                                             */
/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void csr_jacobi_kernel(
    const IndexType* const csrIA,
    const IndexType* const csrJA,
    const ValueType* const csrValues,
    const IndexType numRows,
    const ValueType* const rhs,
    ValueType* const solution,
    const ValueType* const oldSolution,
    const ValueType omega )
{
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType temp = rhs[i];
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        const ValueType diag = csrValues[rowStart];

        for ( IndexType jj = rowStart + 1; jj < rowEnd; ++jj )
        {
            temp -= csrValues[jj] * fetchVectorX<ValueType, useTexture>( oldSolution, csrJA[jj] );
        }

        if ( omega == 0.5 )
        {
            solution[i] = omega * ( fetchVectorX<ValueType, useTexture>( oldSolution, i ) + temp / diag );
        }
        else if ( omega == 1.0 )
        {
            solution[i] = temp / diag;
        }
        else
        {
            solution[i] = omega * ( temp / diag ) + ( 1.0 - omega ) * fetchVectorX<ValueType, useTexture>( oldSolution, i );
        }
    }
}

template<typename ValueType>
__inline__ __device__ ValueType getSharedValue( ValueType* shared, const ValueType* const value, const IndexType index )
{
    if ( index / blockDim.x == blockIdx.x )
    {
        return shared[index % blockDim.x];
    }
    else
    {
        return value[index];
    }
}

//these templates allow to combine dynamic shared memory with templates
template<typename ValueType>
struct SharedMemory
{
    //! @brief Return a pointer to the runtime-sized shared memory array.
    //! @returns Pointer to runtime-sized shared memory array
    __device__
    ValueType* getPointer()
    {
        extern __device__ void Error_UnsupportedType(); // Ensure that we won't compile any un-specialized types
        Error_UnsupportedType();
        return ( ValueType* ) 0;
    }

};

template<>
struct SharedMemory<float>
{
    __device__
    float* getPointer()
    {
        extern __shared__ float s_float[];
        return s_float;
    }
};

template<>
struct SharedMemory<double>
{
    __device__
    double* getPointer()
    {
        extern __shared__ double s_double[];
        return s_double;
    }
};

//this is just like the other jacobi kernel, but it performs a coalesced prefetch of the old solution
//instead of using the texture memory
template<typename ValueType>
__global__ void csr_alternate_jacobi_kernel(
    const IndexType* const csrIA,
    const IndexType* const csrJA,
    const ValueType* const csrValues,
    const IndexType numRows,
    const ValueType* const rhs,
    ValueType* const solution,
    const ValueType* const oldSolution,
    const ValueType omega )
{
    const IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );
    SharedMemory<ValueType> smem;
    ValueType* shared = smem.getPointer();

    if ( i < numRows )
    {
        //this is the prefetch
        shared[threadIdx.x] = oldSolution[i];
        __syncthreads();
        ValueType temp = rhs[i];
        const IndexType rowStart = csrIA[i];
        const IndexType rowEnd = csrIA[i + 1];
        const ValueType diag = csrValues[rowStart];

        for ( IndexType jj = rowStart + 1; jj < rowEnd; ++jj )
        {
            temp -= csrValues[jj] * getSharedValue<ValueType>( shared, oldSolution, csrJA[jj] );
        }

        if ( omega == 0.5 )
        {
            solution[i] = omega * ( getSharedValue<ValueType>( shared, oldSolution, i ) + temp / diag );
        }
        else if ( omega == 1.0 )
        {
            solution[i] = temp / diag;
        }
        else
        {
            solution[i] = omega * ( temp / diag ) + ( 1.0 - omega ) * getSharedValue<ValueType>( shared, oldSolution, i );
        }
    }
}

template<typename ValueType>
void CUDACSRUtils::jacobi(
    ValueType* const solution,
    const IndexType* const csrIA,
    const IndexType* const csrJA,
    const ValueType* const csrValues,
    const ValueType* const oldSolution,
    const ValueType* const rhs,
    const ValueType omega,
    const IndexType numRows )
{
    SCAI_LOG_INFO( logger, "jacobi, #rows = " << numRows )
    SCAI_CHECK_CUDA_ACCESS
    hipStream_t stream = 0;
    bool useTexture = CUDASettings::useTexture();
    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
        useTexture = false; // not yet supported
    }

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );
    SCAI_LOG_INFO( logger, "Start csr_jacobi_kernel<" << TypeTraits<ValueType>::id()
                   << ", useTexture = " << useTexture << ">" );

    if ( useTexture )
    {
        vectorBindTexture( oldSolution );
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobi_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
        csr_jacobi_kernel <ValueType, true> <<< dimGrid, dimBlock, 0, stream>>>( csrIA, csrJA, csrValues, numRows,
                rhs, solution, oldSolution, omega );
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobi_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
        csr_jacobi_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream>>>( csrIA, csrJA, csrValues, numRows, rhs,
                solution, oldSolution, omega );
    }

    if ( !syncToken )
    {
        hipStreamSynchronize( stream );
    }

    if ( useTexture )
    {
        vectorUnbindTexture( oldSolution );
    }
}

/* --------------------------------------------------------------------------- */
/*                          Jacobi halo                                        */
/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void csr_jacobiHalo_kernel(
    ValueType* const solution,
    const IndexType* const localIA,
    const ValueType* const localValues,
    const IndexType* const haloIA,
    const IndexType* const haloJA,
    const ValueType* const haloValues,
    const IndexType* const rowIndexes,
    const IndexType numNonEmptyRows,
    const ValueType* const oldSolution,
    const ValueType omega )
{
    const IndexType ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( ii < numNonEmptyRows )
    {
        IndexType i = ii; // default: rowIndexes is identity

        if ( rowIndexes )
        {
            i = rowIndexes[ii];
        }

        ValueType temp = 0.0;
        const IndexType rowStart = haloIA[i];
        const IndexType rowEnd = haloIA[i + 1];

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            temp += haloValues[jj] * fetchVectorX<ValueType, useTexture>( oldSolution, haloJA[jj] );
        }

        const ValueType diag = localValues[localIA[i]];
        solution[i] -= temp * ( omega / diag );
    }
}

template<typename ValueType>
void CUDACSRUtils::jacobiHalo(
    ValueType solution[],
    const IndexType localIA[],
    const ValueType localValues[],
    const IndexType haloIA[],
    const IndexType haloJA[],
    const ValueType haloValues[],
    const IndexType haloRowIndexes[],
    const ValueType oldSolution[],
    const ValueType omega,
    const IndexType numNonEmptyRows )
{
    SCAI_LOG_INFO( logger, "jacobiHalo, #non-empty rows = " << numNonEmptyRows )
    SCAI_CHECK_CUDA_ACCESS
    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numNonEmptyRows, dimBlock.x );
    bool useTexture = CUDASettings::useTexture();
    useTexture = false;

    if ( useTexture )
    {
        vectorBindTexture( oldSolution );
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobiHalo_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
        csr_jacobiHalo_kernel <ValueType, true> <<< dimGrid, dimBlock>>>( solution, localIA, localValues, haloIA,
                haloJA, haloValues, haloRowIndexes,
                numNonEmptyRows, oldSolution, omega );
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobiHalo_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
        csr_jacobiHalo_kernel<ValueType, false> <<< dimGrid, dimBlock>>>( solution, localIA, localValues, haloIA,
                haloJA, haloValues, haloRowIndexes, numNonEmptyRows,
                oldSolution, omega );
    }

    SCAI_CUDA_RT_CALL( hipGetLastError(), "LAMA_STATUS_CSRJACOBIHALO_CUDAKERNEL_FAILED" )
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "LAMA_STATUS_CSRJACOBIHALO_CUDAKERNEL_FAILED" )

    if ( useTexture )
    {
        vectorUnbindTexture( oldSolution );
    }
}

/* --------------------------------------------------------------------------- */
/*                          Jacobi halo with diagonal array                    */
/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void csr_jacobiHaloWithDiag_kernel(
    ValueType* const solution,
    const ValueType* const localDiagValues,
    const IndexType* const haloIA,
    const IndexType* const haloJA,
    const ValueType* const haloValues,
    const IndexType* const rowIndexes,
    const IndexType numNonEmptyRows,
    const ValueType* const oldSolution,
    const ValueType omega )
{
    const IndexType ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( ii < numNonEmptyRows )
    {
        IndexType i = ii; // default: rowIndexes is identity

        if ( rowIndexes )
        {
            i = rowIndexes[ii];
        }

        ValueType temp = 0.0;
        const IndexType rowStart = haloIA[i];
        const IndexType rowEnd = haloIA[i + 1];

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            temp += haloValues[jj] * fetchVectorX<ValueType, useTexture>( oldSolution, haloJA[jj] );
        }

        const ValueType diag = localDiagValues[i];
        solution[i] -= temp * ( omega / diag );
    }
}

template<typename ValueType>
void CUDACSRUtils::jacobiHaloWithDiag(
    ValueType solution[],
    const ValueType localDiagValues[],
    const IndexType haloIA[],
    const IndexType haloJA[],
    const ValueType haloValues[],
    const IndexType haloRowIndexes[],
    const ValueType oldSolution[],
    const ValueType omega,
    const IndexType numNonEmptyRows )
{
    SCAI_LOG_INFO( logger, "jacobiHaloWithDiag, #non-empty rows = " << numNonEmptyRows )
    SCAI_CHECK_CUDA_ACCESS
    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numNonEmptyRows, dimBlock.x );
    bool useTexture = CUDASettings::useTexture();
    useTexture = false;

    if ( useTexture )
    {
        vectorBindTexture( oldSolution );
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobiHaloWithDiag_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobiHaloWithDiag_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
    }

    if ( useTexture )
    {
        csr_jacobiHaloWithDiag_kernel <ValueType, true> <<< dimGrid, dimBlock>>>( solution, localDiagValues, haloIA,
                haloJA, haloValues, haloRowIndexes,
                numNonEmptyRows, oldSolution, omega );
    }
    else
    {
        csr_jacobiHaloWithDiag_kernel<ValueType, false> <<< dimGrid, dimBlock>>>( solution, localDiagValues, haloIA,
                haloJA, haloValues, haloRowIndexes, numNonEmptyRows,
                oldSolution, omega );
    }

    SCAI_CUDA_RT_CALL( hipGetLastError(), "LAMA_STATUS_CSRJACOBIHALOWITHDIAG_CUDAKERNEL_FAILED" )
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "LAMA_STATUS_CSRJACOBIHALOWITHDIAG_CUDAKERNEL_FAILED" )

    if ( useTexture )
    {
        vectorUnbindTexture( oldSolution );
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             helper                                                                 */
/* ------------------------------------------------------------------------------------------------------------------ */

__device__ __inline__ IndexType multHlp_getNumActiveThreads(
    IndexType aColIt,
    IndexType aColEnd,
    const IndexType* aIA,
    IndexType aRowIt,
    IndexType offset )
{
#ifdef CUDA_CAP_20
    IndexType end = __popc ( __ballot ( aColIt < aColEnd ) );
#else
    IndexType aColStart = aIA[aRowIt] + offset;
    IndexType end = ( aColEnd - aColStart );
#endif
    return end;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAddSizes                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template<int nWarps>
__global__ void matrixAddSizesKernel(
    IndexType* cIa,
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const IndexType* aIa,
    const IndexType* aJa,
    const IndexType* bIa,
    const IndexType* bJa )
{
    __shared__ volatile IndexType sColA[nWarps];
    __shared__ volatile bool sFound[nWarps];
    IndexType localWarpId = threadIdx.x / warpSize;
    IndexType globalWarpId = ( blockIdx.x * blockDim.x + threadIdx.x ) / warpSize;
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
//IndexType numWarpsLocal  = blockDim.x / warpSize;
    IndexType numWarpsGlobal = ( blockDim.x * gridDim.x ) / warpSize;
    IndexType rowIt = globalWarpId;

    for ( ; __any( rowIt < numRows ); rowIt += numWarpsGlobal )
    {
        if ( rowIt < numRows )
        {
            if ( diagonalProperty && rowIt >= numColumns )
            {
                diagonalProperty = false;
            }

            IndexType aColIt = aIa[rowIt] + laneId;
            IndexType aColEnd = aIa[rowIt + 1];
            IndexType bColIt = bIa[rowIt] + laneId;
            IndexType bColEnd = bIa[rowIt + 1];

            if ( laneId == 0 )
            {
                cIa[rowIt] = bColEnd - bColIt;
            }

            for ( IndexType aColItOffset = 0; __any( aColIt < aColEnd ); aColIt += warpSize, aColItOffset += warpSize )
            {
                IndexType colA = aColIt < aColEnd ? aJa[aColIt] : cudaNIndex;
                IndexType end = multHlp_getNumActiveThreads( aColIt, aColEnd, aIa, rowIt, aColItOffset );

                for ( IndexType k = 0; k < end && k < warpSize; k++ )
                {
                    if ( laneId == k )
                    {
                        sColA[localWarpId] = colA;
                    }

                    sFound[localWarpId] = false;

                    for ( IndexType bColItOffset = 0; !sFound[localWarpId] && __any( ( bColIt + bColItOffset ) < bColEnd );
                            bColItOffset += warpSize )
                    {
                        IndexType colB = ( bColIt + bColItOffset ) < bColEnd ? bJa[bColIt + bColItOffset] : cudaNIndex;

                        if ( sColA[localWarpId] == colB )
                        {
                            sFound[localWarpId] = true;
                        }
                    }

                    if ( laneId == 0 && !sFound[localWarpId] )
                    {
                        cIa[rowIt]++;
                    }
                }
            }
        }
    }
}

IndexType CUDACSRUtils::matrixAddSizes(
    IndexType cIa[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const IndexType aIa[],
    const IndexType aJa[],
    const IndexType bIa[],
    const IndexType bJa[] )
{
    SCAI_REGION( "CUDA.CSRUtils.matrixAddSizes" )
    SCAI_LOG_INFO(
        logger,
        "matrixAddSizes for " << numRows << " x " << numColumns << " matrix" << ", diagonalProperty = " << diagonalProperty )
    SCAI_CHECK_CUDA_ACCESS
// Reset cIa
    thrust::device_ptr<IndexType> cIaPtr( cIa );
    thrust::fill( cIaPtr, cIaPtr + numRows, 0 );
// TODO: Check if diagonal property needs special attention
    matrixAddSizesKernel<NUM_WARPS> <<< NUM_BLOCKS, NUM_THREADS>>>( cIa, numRows, numColumns, diagonalProperty,
            aIa, aJa, bIa, bJa );
    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR
// Convert sizes array to offset array
    thrust::exclusive_scan( cIaPtr, cIaPtr + numRows + 1, cIaPtr );
// Copy numValues from cIa to Host
// TODO: use cuMem cpy
    thrust::device_ptr<IndexType> iaPtr( cIa );
    thrust::host_vector<IndexType> numValues( iaPtr + numRows, iaPtr + numRows + 1 );
    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR
// TODO: write it!
    return numValues[0];
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             hashTable Methods                                                      */
/* ------------------------------------------------------------------------------------------------------------------ */

__device__
inline bool multHlp_insertIndexex( IndexType colB,
                                   IndexType sHashTableIndexes[],
                                   IndexType aRowIt,
                                   IndexType* chunkPtr,
                                   volatile IndexType chunkList[],
                                   IndexType numReservedChunks,
                                   IndexType* cIA )
{
    const IndexType one = 1;

    IndexType fx = HASH_A * colB;
    IndexType gx = ( fx + HASH_B ) % HASH_P;

    if ( numReservedChunks == 0 )
    {
        for ( IndexType i = 0; i < NUM_HASH_RETRIES; i++ )
        {
            IndexType hash = ( gx + HASH_C0 * i + HASH_C1 *  i * i ) % NUM_ELEMENTS_IN_SHARED;
            IndexType val = common::CUDAUtils::atomicCAS( &sHashTableIndexes[hash], cudaNIndex, colB );

            if ( val == cudaNIndex )
            {
                common::CUDAUtils::atomicAdd( &cIA[aRowIt], one );
                return true;
            }

            if ( val == colB )
            {
                return true;
            }
        }

        return false;
    }

    for ( IndexType i = 0; i < NUM_HASH_RETRIES; i++ )
    {
        IndexType globalHash = ( gx + HASH_C0 * i + HASH_C1 * ( IndexType ) i * i ) % ( NUM_ELEMENTS_PER_CHUNK * numReservedChunks );
        IndexType localHash = globalHash % NUM_ELEMENTS_PER_CHUNK;
        IndexType chunk = globalHash / NUM_ELEMENTS_PER_CHUNK;
        IndexType val = common::CUDAUtils::atomicCAS( &chunkPtr[chunkList[chunk] * NUM_ELEMENTS_PER_CHUNK + localHash], cudaNIndex, colB );

        if ( val == cudaNIndex )
        {
            common::CUDAUtils::atomicAdd( &cIA[aRowIt], one );
            return true;
        }

        if ( val == colB )
        {
            return true;
        }
    }

    return false;
}

template <typename ValueType>
__device__
inline bool multHlp_insertValues( IndexType colB,
                                  IndexType* sHashTableIndexes,
                                  ValueType* sHashTableValues,
                                  IndexType* indexChunks,
                                  ValueType* valueChunks,
                                  volatile IndexType chunkList[],
                                  IndexType numReservedChunks,
                                  ValueType valB,
                                  ValueType sValA )
{
    IndexType fx = HASH_A * colB;
    IndexType gx = ( fx + HASH_B ) % HASH_P;

    if ( numReservedChunks == 0 )
    {
        for ( IndexType i = 0; i < NUM_HASH_RETRIES; i++ )
        {
            IndexType hash = ( gx + HASH_C0 * i + HASH_C1 * i * i ) % NUM_ELEMENTS_IN_SHARED;
            IndexType val = common::CUDAUtils::atomicCAS( &sHashTableIndexes[hash], cudaNIndex, colB );

            if ( val == cudaNIndex )
            {
                sHashTableValues[hash] = valB * sValA;
                return true;
            }

            if ( val == colB )
            {
                sHashTableValues[hash] += valB * sValA;
                return true;
            }
        }

        return false;
    }

    for ( IndexType i = 0; i < NUM_HASH_RETRIES; i++ )
    {
        IndexType globalHash = ( gx + HASH_C0 * i + HASH_C1 * ( IndexType ) i * i ) % ( NUM_ELEMENTS_PER_CHUNK * numReservedChunks );
        IndexType localHash = globalHash % NUM_ELEMENTS_PER_CHUNK;
        IndexType chunk = globalHash / NUM_ELEMENTS_PER_CHUNK;
        IndexType val = common::CUDAUtils::atomicCAS( &indexChunks[chunkList[chunk] * NUM_ELEMENTS_PER_CHUNK + localHash], cudaNIndex, colB );

        if ( val == cudaNIndex )
        {
            valueChunks[chunkList[chunk] * NUM_ELEMENTS_PER_CHUNK + localHash] = sValA * valB;
            return true;
        }

        if ( val == colB )
        {
            valueChunks[chunkList[chunk] * NUM_ELEMENTS_PER_CHUNK + localHash] += sValA * valB;
            return true;
        }
    }

    return false;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiplySizes                                                    */
/* ------------------------------------------------------------------------------------------------------------------ */

__device__
inline bool multHlp_nextRow( IndexType* row,
                             IndexType numRows
#ifdef USE_LOAD_BALANCING
                             , IndexType* rowCounter
#endif
                           )
{
#ifdef USE_LOAD_BALANCING
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
    IndexType localWarpId = threadIdx.x / warpSize;
    __shared__ volatile IndexType sRowIt[NUM_WARPS];

    if ( laneId == 0 )
    {
        IndexType one = 1;
        sRowIt[localWarpId] = common::CUDAUtils::atomicAdd( rowCounter, one );
    }

    *row = sRowIt[localWarpId];

    if ( *row < numRows )
    {
        return true;
    }
    else
    {
        return false;
    }

#else
    IndexType numWarpsGlobal = ( blockDim.x * gridDim.x ) / warpSize;
    *row += numWarpsGlobal;

    if ( *row < numRows )
    {
        return true;
    }
    else
    {
        return false;
    }

#endif
}

__device__
inline void multHlp_releaseChunks ( IndexType* chunkList,
                                    volatile IndexType* sChunkList,
                                    volatile IndexType* sReservedChunks,
                                    IndexType chunkCount )
{
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;

    if ( laneId == 0 )
    {
        for ( IndexType i = *sReservedChunks - 1; i >= *sReservedChunks - chunkCount; --i )
        {
            IndexType headItem;
            IndexType old;

            do
            {
                headItem = chunkList[0];
                chunkList[sChunkList[i] + 1] = headItem;
                old = common::CUDAUtils::atomicCAS( const_cast<IndexType*>( &chunkList[0] ), headItem, sChunkList[i] );
            }
            while ( old != headItem );
        }
    }

    *sReservedChunks = *sReservedChunks - chunkCount;
}

__device__
inline bool multHlp_reserveChunks( IndexType* chunkList,
                                   volatile IndexType* sChunkList,
                                   volatile IndexType* sReservedChunks,
                                   IndexType chunkCount )
{
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;

    if ( chunkCount > NUM_CHUNKS_PER_WARP )
    {
//        printf("to many chunks %i\n", chunkCount);
        return false;
    }

    if ( laneId == 0 && chunkCount > 0 && *sReservedChunks != chunkCount )
    {
        if ( *sReservedChunks < chunkCount )
        {
            for ( IndexType i = *sReservedChunks; i < chunkCount; ++i )
            {
                IndexType headItem;
                IndexType nextItem;
                IndexType old;

                do
                {
                    headItem = chunkList[0];

                    if ( headItem != cudaNIndex )
                    {
                        __threadfence();
                        nextItem = chunkList[headItem + 1];

                        old = common::CUDAUtils::atomicCAS( const_cast<IndexType*>( &chunkList[0] ), headItem, nextItem );

                        if ( old == headItem )
                        {
                            sChunkList[i] = headItem;
                        }
                    }
                    else
                    {
//                        printf("no more chunks!\n");
                        return false;
                    }
                }
                while ( old != headItem );
            }

            *sReservedChunks = chunkCount;
            return true;
        }
        else
        {
            multHlp_releaseChunks ( chunkList, sChunkList, sReservedChunks, *sReservedChunks - chunkCount );
            return true;
        }
    }
    else
    {
        return true;
    }
}

__device__
inline void multHlp_initializeChunks ( IndexType* sHashTable,
                                       IndexType* chunks,
                                       const IndexType numElementsPerChunk,
                                       volatile IndexType* sChunkList,
                                       volatile IndexType sReservedChunks )
{
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;

    if ( sReservedChunks == 0 )
    {
        for ( IndexType i = 0; i < NUM_ELEMENTS_IN_SHARED; i += warpSize )
        {
            if ( i + laneId < NUM_ELEMENTS_IN_SHARED )
            {
                sHashTable[i + laneId] = cudaNIndex;
            }
        }

        return;
    }

    for ( IndexType i = 0; i < sReservedChunks; ++i )
    {
        IndexType chunkId = sChunkList[i];

        for ( IndexType j = laneId; j < numElementsPerChunk; j += warpSize )
        {
            chunks[chunkId * numElementsPerChunk + j] = cudaNIndex;
        }
    }
}

__device__
inline IndexType multHlp_growth ( IndexType numChunks )
{
    if ( numChunks == 0 )
    {
        return 2;
    }
    else
    {
        return numChunks * 2;
    }
}

__device__
inline IndexType multHlp_calcOptChunkCount ( IndexType row,
        const IndexType* cIA,
        const IndexType numElementsPerChunk )
{
    IndexType numElements = cIA[row + 1] - cIA[row];

    if ( numElements * 2 < NUM_ELEMENTS_IN_SHARED )
    {
        return 0;
    }
    else
    {
        return ( ( ( cIA[row + 1] - cIA[row] ) * 2 ) / numElementsPerChunk ) + 1;
    }
}

__global__
void matrixMultiplySizesKernel(
    const IndexType* aIA,
    const IndexType* aJA,
    const IndexType* bIA,
    const IndexType* bJA,
    IndexType* cIA,
    const IndexType numRows,
    const IndexType numColumns,
    IndexType* chunkPtr,
    IndexType* chunkList,
    IndexType numChunks,
    bool* hashError,
    bool diagonalProperty )
{
    __shared__ IndexType sHashTable[NUM_ELEMENTS_IN_SHARED];
    __shared__ volatile IndexType sReservedChunks;
    __shared__ volatile IndexType sChunkList[NUM_CHUNKS_PER_WARP];
    __shared__ volatile IndexType sColA;
    __shared__ volatile IndexType sRowIt;
    __shared__ volatile bool sInsertMiss;
    IndexType globalWarpId = ( blockIdx.x * blockDim.x + threadIdx.x ) / warpSize;
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
    IndexType colB;
    IndexType aRowIt = globalWarpId;
    bool localSystemError = false;
    sReservedChunks = 0;

    if ( aRowIt < numRows )
    {
        do
        {
            do
            {
                sInsertMiss = false;
                IndexType aColIt = aIA[aRowIt] + laneId;
                IndexType aColEnd = aIA[aRowIt + 1];

                if ( laneId == 0 && diagonalProperty )
                {
                    cIA[aRowIt]++;
                }

                multHlp_initializeChunks( sHashTable,
                                          chunkPtr,
                                          NUM_ELEMENTS_PER_CHUNK,
                                          sChunkList,
                                          sReservedChunks );

                for ( IndexType offset = 0; __any( aColIt < aColEnd ); aColIt += warpSize, offset += warpSize )
                {
                    IndexType colA = aColIt < aColEnd ? aJA[aColIt] : cudaNIndex;
                    IndexType end = multHlp_getNumActiveThreads( aColIt, aColEnd, aIA, aRowIt, offset );

                    for ( IndexType k = 0; k < end && k < warpSize; k++ )
                    {
                        if ( laneId == k )
                        {
                            sColA = colA;
                        }

                        IndexType bColIt = bIA[sColA] + laneId;
                        IndexType bColEnd = bIA[sColA + 1];

                        for ( ; __any( bColIt < bColEnd ); bColIt += warpSize )
                        {
                            colB = bColIt < bColEnd ? bJA[bColIt] : cudaNIndex;

                            if ( colB != cudaNIndex && ( !diagonalProperty || colB != aRowIt ) )
                            {
                                bool inserted = multHlp_insertIndexex( colB,
                                                                       sHashTable,
                                                                       aRowIt,
                                                                       chunkPtr,
                                                                       sChunkList,
                                                                       sReservedChunks,
                                                                       cIA );

                                if ( !inserted )
                                {
                                    sInsertMiss = true;
                                }
                            }
                        }
                    }
                }

                // only release if insertion was ok, otherwire reserve some more
                // STEP x: release reserved chunks
                if ( laneId == 0 )
                {
                    if ( sInsertMiss )
                    {
                        cIA[aRowIt] = 0;

                        if ( !multHlp_reserveChunks( chunkList, sChunkList, &sReservedChunks, multHlp_growth( sReservedChunks ) ) )
                        {
                            // ABORT KERNEL HERE;
                            localSystemError = true;
                        }
                    }
                }

                if ( __any( localSystemError ) )
                {
                    *hashError = true;
                    return;
                }
            }
            while ( sInsertMiss );
        }
        while ( multHlp_nextRow( &aRowIt, numRows ) );
    }

    // release all remaining chunks
    multHlp_releaseChunks( chunkList, sChunkList, &sReservedChunks, sReservedChunks );
}

struct multHlp_chunkFill
{
    const IndexType n;
    multHlp_chunkFill( IndexType _n )
        : n( _n )
    {
    }
    __device__
    IndexType operator()( IndexType i )
    {
        if ( i == ( n - 1 ) )
        {
            return cudaNIndex;
        }

        return i;
    }
};

IndexType CUDACSRUtils::matrixMultiplySizes(
    IndexType cIa[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType /* k */,
    bool diagonalProperty,
    const IndexType aIa[],
    const IndexType aJa[],
    const IndexType bIa[],
    const IndexType bJa[] )
{
    SCAI_REGION( "CUDA.CSR.matrixMultiplySizes" )
    SCAI_LOG_INFO(
        logger,
        "matrixMutliplySizes for " << numRows << " x " << numColumns << " matrix" << ", diagonalProperty = " << diagonalProperty )
    SCAI_CHECK_CUDA_ACCESS
    // Reset cIa
    thrust::device_ptr<IndexType> cIaPtr( cIa );
    thrust::fill( cIaPtr, cIaPtr + numRows, 0 );
    ContextPtr loc = Context::getContextPtr( context::CUDA );
    MemoryPtr mem = loc->getMemoryPtr();
    bool hashErrorHost = false;
    bool* hashError = ( bool* ) mem->allocate( sizeof( bool ) );
    hipMemcpy( hashError, &hashErrorHost, sizeof( bool ), hipMemcpyHostToDevice );
    size_t free;
    size_t total;
    hipMemGetInfo( &free, &total );
    int nnz_a;
    int nnz_b;
    hipMemcpy( &nnz_a, &aIa[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnz_b, &bIa[numColumns], sizeof( IndexType ), hipMemcpyDeviceToHost );
    int avgDensity = ( nnz_a / numRows + nnz_b / numColumns ) / 2;
    int numChunks;
    int maxNumChunks = ( free - ( 100 * 1024 * 1024 ) ) / ( NUM_ELEMENTS_PER_CHUNK * sizeof ( IndexType ) * 2 );
    int chunksPerWarp = NUM_BLOCKS * ( ( avgDensity * 8 ) / NUM_ELEMENTS_PER_CHUNK + 1 );

    if ( chunksPerWarp > maxNumChunks )
    {
        numChunks = maxNumChunks;
    }
    else
    {
        numChunks = chunksPerWarp;
    }

    unsigned int hashTableAllocatedBytes = numChunks * NUM_ELEMENTS_PER_CHUNK * sizeof( IndexType );
    IndexType* hashTable = ( IndexType* ) mem->allocate( hashTableAllocatedBytes );
    // chunkList table needs one integers per chunk plus 1 start pointer
    unsigned int chunkListAllocatedBytes = numChunks * sizeof( IndexType ) + sizeof( IndexType );
    IndexType* chunkList = ( IndexType* ) mem->allocate( chunkListAllocatedBytes );
    thrust::device_ptr<IndexType> chunkListPtr( chunkList );
    thrust::transform( thrust::make_counting_iterator( 0 ),
                       thrust::make_counting_iterator( numChunks + 1 ),
                       chunkListPtr,
                       multHlp_chunkFill( numChunks + 1 ) );
    matrixMultiplySizesKernel <<< NUM_BLOCKS, NUM_THREADS>>>( aIa,
            aJa,
            bIa,
            bJa,
            cIa,
            numRows,
            numColumns,
            hashTable,
            chunkList,
            numChunks,
            hashError,
            diagonalProperty );
    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR
    hipMemcpy( &hashErrorHost, hashError, sizeof( bool ), hipMemcpyDeviceToHost );

    if ( hashErrorHost )
    {
        COMMON_THROWEXCEPTION( "Multiplication failed!" );
    }

    // Free hashTable and hashError
    mem->free( ( void* ) hashError, sizeof( bool ) );
    mem->free( ( void* ) hashTable, hashTableAllocatedBytes );
    mem->free( ( void* ) chunkList, chunkListAllocatedBytes );
    // Convert sizes array to offset array
    thrust::exclusive_scan( cIaPtr, cIaPtr + numRows + 1, cIaPtr );
    IndexType numValues;
    hipMemcpy( &numValues, &cIa[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    return numValues;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAdd                                                              */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType, int nWarps>
__global__
void matrixAddKernel(
    IndexType* cJA,
    ValueType* cValues,
    const IndexType* cIA,
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const ValueType alpha,
    const IndexType* aIA,
    const IndexType* aJA,
    const ValueType* aValues,
    const ValueType beta,
    const IndexType* bIA,
    const IndexType* bJA,
    const ValueType* bValues )
{
// TODO: Just naive implementation, could be done faster, but works!
// TODO: Check if diagonal property needs special attention
    __shared__ volatile IndexType sColA[nWarps];
    __shared__ volatile ValueType sValA[nWarps];
    __shared__ volatile IndexType sFoundJa[nWarps];
    IndexType localWarpId = threadIdx.x / warpSize;
    IndexType globalWarpId = ( blockIdx.x * blockDim.x + threadIdx.x ) / warpSize;
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
//IndexType numWarpsLocal  = blockDim.x / warpSize;
    IndexType numWarpsGlobal = ( blockDim.x * gridDim.x ) / warpSize;
    IndexType rowIt = globalWarpId;

    for ( ; __any( rowIt < numRows ); rowIt += numWarpsGlobal )
    {
        if ( rowIt < numRows )
        {
            if ( diagonalProperty && rowIt >= numColumns )
            {
                diagonalProperty = false;
            }

            IndexType aColIt = aIA[rowIt] + laneId;
            IndexType aColEnd = aIA[rowIt + 1];
            IndexType bColIt = bIA[rowIt] + laneId;
            IndexType bColEnd = bIA[rowIt + 1];
            IndexType cColIt = cIA[rowIt] + laneId;

// Copy values of b to C
            for ( IndexType bColOffset = 0; __any( ( bColIt + bColOffset ) < bColEnd ); bColOffset += warpSize )
            {
                IndexType colB = ( bColIt + bColOffset ) < bColEnd ? bJA[bColIt + bColOffset] : cudaNIndex;
                ValueType valB = ( bColIt + bColOffset ) < bColEnd ? bValues[bColIt + bColOffset] : static_cast<ValueType>( 0 );

                if ( colB != cudaNIndex )
                {
                    cJA[cColIt + bColOffset] = colB;
                    cValues[cColIt + bColOffset] = valB * beta;
                }
            }

// Offset in c after coping b to c
            IndexType cColOffset = bIA[rowIt + 1] - bIA[rowIt];

// Add values of a to c
            for ( IndexType aColItOffset = 0; __any( aColIt < aColEnd ); aColIt += warpSize, aColItOffset += warpSize )
            {
                IndexType colA = aColIt < aColEnd ? aJA[aColIt] : cudaNIndex;
                ValueType valA = aColIt < aColEnd ? aValues[aColIt] : static_cast<ValueType>( 0 );
                IndexType end = multHlp_getNumActiveThreads( aColIt, aColEnd, aIA, rowIt, aColItOffset );

                for ( IndexType k = 0; k < end && k < warpSize; k++ )
                {
                    if ( laneId == k )
                    {
                        sColA[localWarpId] = colA;
                        sValA[localWarpId] = valA;
                        sFoundJa[localWarpId] = cudaNIndex;
                    }

                    for ( IndexType bColItOffset = 0; ( sFoundJa[localWarpId] == cudaNIndex ) && __any( ( bColIt + bColItOffset ) < bColEnd );
                            bColItOffset += warpSize )
                    {
                        IndexType colB = ( bColIt + bColItOffset ) < bColEnd ? bJA[bColIt + bColItOffset] : cudaNIndex;

                        if ( sColA[localWarpId] == colB )
                        {
                            sFoundJa[localWarpId] = laneId + bColItOffset;
                        }
                    }

                    if ( laneId == 0 )
                    {
                        if ( sFoundJa[localWarpId] == cudaNIndex )
                        {
                            // Element is new element, add new element
                            cJA[cColIt + cColOffset] = colA;
                            cValues[cColIt + cColOffset] = sValA[localWarpId] * alpha;
                            cColOffset++;
                        }
                        else
                        {
                            // Element exists, add values
                            // We can use cColIt, because this is thread with laneId = 0!
                            cValues[cColIt + sFoundJa[localWarpId]] += sValA[localWarpId] * alpha;
                        }
                    }
                }
            }
        }
    }
}

template<typename ValueType>
void CUDACSRUtils::matrixAdd(
    IndexType cJA[],
    ValueType cValues[],
    const IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const ValueType alpha,
    const IndexType aIA[],
    const IndexType aJA[],
    const ValueType aValues[],
    const ValueType beta,
    const IndexType bIA[],
    const IndexType bJA[],
    const ValueType bValues[] )
{
    SCAI_REGION( "CUDA.CSRUtils.matrixAdd" )
    SCAI_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )
    SCAI_CHECK_CUDA_ACCESS
    matrixAddKernel<ValueType, NUM_WARPS> <<< NUM_BLOCKS, NUM_THREADS>>>( cJA, cValues, cIA, numRows, numColumns,
            diagonalProperty, alpha, aIA, aJA, aValues, beta, bIA, bJA, bValues );
    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR
}
/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiply                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template <typename ValueType>
__device__
inline void multHlp_copyHashtable ( volatile IndexType* sColA,
                                    const IndexType* cIA,
                                    IndexType laneId,
                                    IndexType aRowIt,
                                    const ValueType alpha,
                                    IndexType* cJA,
                                    ValueType* cValues,
                                    IndexType* sHashTableIndexes,
                                    ValueType* sHashTableValues,
                                    IndexType* indexChunks,
                                    ValueType* valueChunks,
                                    volatile IndexType chunkList[],
                                    IndexType numReservedChunks,
                                    bool diagonalProperty,
                                    ValueType diagonalElement )

{
    // TODO: rename sColA => destinationOffset!

    *sColA = 0;
    IndexType rowOffset = cIA[aRowIt];
    IndexType one = 1;
    IndexType hashCol;
    ValueType hashVal;

    if ( diagonalProperty && laneId == 0 )
    {
        cJA[rowOffset] = aRowIt;
        cValues[rowOffset] = diagonalElement * alpha;
        *sColA = 1;
    }

    if ( numReservedChunks == 0 )
    {
        for ( int j = laneId; j < NUM_ELEMENTS_IN_SHARED; j += warpSize )
        {
            hashCol = sHashTableIndexes[j];
            hashVal = sHashTableValues[j];
#if SCAI_CUDA_COMPUTE_CAPABILITY >= 20
            IndexType localOffset;
            // TODO: be carefull here, ballot is warpsize Bit's long!
            IndexType ballot = __ballot ( hashCol != cudaNIndex );
            localOffset = __popc( ballot << ( warpSize - laneId ) );

            if ( hashCol != cudaNIndex )
            {
                cJA[rowOffset + *sColA + localOffset] = hashCol;
                cValues[rowOffset + *sColA + localOffset] = hashVal * alpha;
            }

            *sColA += __popc( ballot );
#else

            if ( hashCol != cudaNIndex )
            {
                // the volatile attribute must be cast away
                IndexType offset = common::CUDAUtils::atomicAdd( const_cast<IndexType*>( sColA ), one );
                cJA[rowOffset + offset] = hashCol;
                cValues[rowOffset + offset] = hashVal * alpha;
            }

#endif
        }

        return;
    }

    for ( int i = 0; i < numReservedChunks; ++i )
    {
        for ( int j = laneId; j < NUM_ELEMENTS_PER_CHUNK; j += warpSize )
        {
            hashCol = indexChunks[chunkList[i] * NUM_ELEMENTS_PER_CHUNK + j];
            hashVal = valueChunks[chunkList[i] * NUM_ELEMENTS_PER_CHUNK + j];
#if SCAI_CUDA_COMPUTE_CAPABILITY >= 20
            IndexType localOffset;
            // TODO: be carefull here, ballot is warpsize Bit's long!
            IndexType ballot = __ballot ( hashCol != cudaNIndex );
            localOffset = __popc( ballot << ( warpSize - laneId ) );

            if ( hashCol != cudaNIndex )
            {
                cJA[rowOffset + *sColA + localOffset] = hashCol;
                cValues[rowOffset + *sColA + localOffset] = hashVal * alpha;
            }

            if ( laneId == 0 )
            {
                *sColA += __popc( ballot );
            }

#else

            if ( hashCol != cudaNIndex )
            {
                IndexType offset = common::CUDAUtils::atomicAdd( const_cast<IndexType*>( sColA ), IndexType( 1 ) );
                cJA[rowOffset + offset] = hashCol;
                cValues[rowOffset + offset] = hashVal * alpha;
            }

#endif
        }
    }
}

template<typename ValueType>
__global__
void matrixMultiplyKernel(
    const IndexType* aIA,
    const IndexType* aJA,
    const ValueType* aValues,
    const IndexType* bIA,
    const IndexType* bJA,
    const ValueType* bValues,
    const IndexType* cIA,
    const ValueType alpha,
    IndexType* cJA,
    ValueType* cValues,
    const IndexType numRows,
    const IndexType numColumns,
    IndexType* indexChunks,
    ValueType* valueChunks,
    IndexType* chunkList,
    const IndexType numChunks,
    bool* hashError,
    bool diagonalProperty )
{
    __shared__ IndexType sHashTableIndexes[NUM_ELEMENTS_IN_SHARED];
    __shared__ ValueType sHashTableValues[NUM_ELEMENTS_IN_SHARED];
    __shared__ volatile IndexType sReservedChunks;
    __shared__ volatile IndexType sChunkList[NUM_CHUNKS_PER_WARP];
    __shared__ volatile IndexType sColA;
    __shared__ volatile ValueType sValA;
    __shared__ volatile IndexType sRowIt;
    __shared__ volatile bool sInsertMiss;
    __shared__ volatile ValueType diagonalElement;
    IndexType globalWarpId = ( blockIdx.x * blockDim.x + threadIdx.x ) / warpSize;
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
    IndexType colB;
    IndexType aRowIt = globalWarpId;
    bool localSystemError = false;
    sReservedChunks = 0;

    if ( aRowIt < numRows )
    {
        do
        {
            IndexType optimalChunkCount = multHlp_calcOptChunkCount ( aRowIt, cIA, NUM_ELEMENTS_PER_CHUNK );

            // reserve Chunks
            if ( !multHlp_reserveChunks( chunkList, sChunkList, &sReservedChunks, optimalChunkCount ) )
            {
                // ABORT KERNEL HERE;
                localSystemError = true;
            }

            if ( __any( localSystemError ) )
            {
                *hashError = true;
                return;
            }

            do
            {
                sInsertMiss = false;
                IndexType aColIt = aIA[aRowIt] + laneId;
                IndexType aColEnd = aIA[aRowIt + 1];

                if ( laneId == 0 && diagonalProperty )
                {
                    diagonalElement = 0.0;
                }

                multHlp_initializeChunks( sHashTableIndexes,
                                          indexChunks,
                                          NUM_ELEMENTS_PER_CHUNK,
                                          sChunkList,
                                          sReservedChunks );

                for ( IndexType offset = 0; __any( aColIt < aColEnd ); aColIt += warpSize, offset += warpSize )
                {
                    IndexType colA = aColIt < aColEnd ? aJA[aColIt] : cudaNIndex;
                    ValueType valA = aColIt < aColEnd ? aValues[aColIt] : static_cast<ValueType>( 0 );
                    IndexType end = multHlp_getNumActiveThreads( aColIt, aColEnd, aIA, aRowIt, offset );

                    for ( IndexType k = 0; k < end && k < warpSize; k++ )
                    {
                        if ( laneId == k )
                        {
                            sColA = colA;
                            sValA = valA;
                        }

                        IndexType bColIt = bIA[sColA] + laneId;
                        IndexType bColEnd = bIA[sColA + 1];

                        for ( ; __any( bColIt < bColEnd ); bColIt += warpSize )
                        {
                            colB = bColIt < bColEnd ? bJA[bColIt] : cudaNIndex;
                            ValueType valB = bColIt < bColEnd ? bValues[bColIt] : static_cast<ValueType>( 0 );

                            if ( diagonalProperty && colB == aRowIt )
                            {
                                diagonalElement += sValA * valB;
                            }
                            else
                            {
                                if ( colB != cudaNIndex && ( !diagonalProperty || colB != aRowIt ) )
                                {
                                    bool inserted = multHlp_insertValues( colB,
                                                                          sHashTableIndexes,
                                                                          sHashTableValues,
                                                                          indexChunks,
                                                                          valueChunks,
                                                                          sChunkList,
                                                                          sReservedChunks,
                                                                          valB,
                                                                          sValA );

                                    if ( !inserted )
                                    {
                                        sInsertMiss = true;
                                    }
                                }
                            }
                        }
                    }
                }

                if ( !sInsertMiss )
                {
                    multHlp_copyHashtable ( &sColA,
                                            cIA,
                                            laneId,
                                            aRowIt,
                                            alpha,
                                            cJA,
                                            cValues,
                                            sHashTableIndexes,
                                            sHashTableValues,
                                            indexChunks,
                                            valueChunks,
                                            sChunkList,
                                            sReservedChunks,
                                            diagonalProperty,
                                            diagonalElement );
                }
                else
                {
                    if ( !multHlp_reserveChunks( chunkList, sChunkList, &sReservedChunks, multHlp_growth( sReservedChunks ) ) )
                    {
                        // ABORT KERNEL HERE;
                        localSystemError = true;
                    }

                    if ( __any( localSystemError ) )
                    {
                        *hashError = true;
                        return;
                    }
                }
            }
            while ( sInsertMiss );
        }
        while ( multHlp_nextRow( &aRowIt, numRows ) );
    }

    // release all remaining chunks
    multHlp_releaseChunks( chunkList, sChunkList, &sReservedChunks, sReservedChunks );
}

template<typename ValueType>
void CUDACSRUtils::matrixMultiply(
    const IndexType cIa[],
    IndexType cJa[],
    ValueType cValues[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType /* k */,
    const ValueType alpha,
    bool diagonalProperty,
    const IndexType aIa[],
    const IndexType aJa[],
    const ValueType aValues[],
    const IndexType bIa[],
    const IndexType bJa[],
    const ValueType bValues[] )
{
    SCAI_REGION( "CUDA.CSRUtils.matrixMultiply" )
    SCAI_LOG_INFO( logger, "matrixMultiply for " << numRows << "x" << numColumns << " matrix" )
    SCAI_CHECK_CUDA_ACCESS
    ContextPtr loc = Context::getContextPtr( context::CUDA );
    MemoryPtr mem = loc->getMemoryPtr();
    bool hashErrorHost = false;
    bool* hashError = ( bool* ) mem->allocate( sizeof( bool ) );
    hipMemcpy( hashError, &hashErrorHost, sizeof( bool ), hipMemcpyHostToDevice );
    size_t free;
    size_t total;
    hipMemGetInfo( &free, &total );
    IndexType nnz_a;
    IndexType nnz_b;
    hipMemcpy( &nnz_a, &aIa[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnz_b, &bIa[numColumns], sizeof( IndexType ), hipMemcpyDeviceToHost );
    IndexType avgDensity = ( nnz_a / numRows + nnz_b / numColumns ) / 2;
    IndexType numChunks;
    IndexType maxNumChunks = ( free - ( 100 * 1024 * 1024 ) ) / ( NUM_ELEMENTS_PER_CHUNK * sizeof ( IndexType ) * 2 );
    IndexType chunksPerWarp = NUM_BLOCKS * ( ( avgDensity * 8 ) / NUM_ELEMENTS_PER_CHUNK + 1 );

    if ( chunksPerWarp > maxNumChunks )
    {
        numChunks = maxNumChunks;
    }
    else
    {
        numChunks = chunksPerWarp;
    }

    unsigned int hashTableAllocatedBytes = numChunks * NUM_ELEMENTS_PER_CHUNK * ( sizeof( IndexType ) + sizeof( ValueType ) );
    void* chunks = ( void* ) mem->allocate( hashTableAllocatedBytes );
    IndexType* indexChunks = ( IndexType* ) chunks;
    ValueType* valueChunks = ( ValueType* ) ( indexChunks + numChunks * NUM_ELEMENTS_PER_CHUNK );
    // chunkList table needs one integers per chunk plus 1 start pointer
    unsigned int chunkListAllocatedBytes = numChunks * sizeof( IndexType ) + sizeof( IndexType );
    IndexType* chunkList = ( IndexType* ) mem->allocate( chunkListAllocatedBytes );
    thrust::device_ptr<IndexType> chunkListPtr( chunkList );
    IndexType zero = 0;
    thrust::transform( thrust::make_counting_iterator( zero ),
                       thrust::make_counting_iterator( numChunks + 1 ),
                       chunkListPtr,
                       multHlp_chunkFill( numChunks + 1 ) );
    matrixMultiplyKernel <<< NUM_BLOCKS, NUM_THREADS>>>( aIa,
            aJa,
            aValues,
            bIa,
            bJa,
            bValues,
            cIa,
            alpha,
            cJa,
            cValues,
            numRows,
            numColumns,
            indexChunks,
            valueChunks,
            chunkList,
            numChunks,
            hashError,
            diagonalProperty );
    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR
    hipMemcpy( &hashErrorHost, hashError, sizeof( bool ), hipMemcpyDeviceToHost );

    if ( hashErrorHost )
    {
        COMMON_THROWEXCEPTION( "Multiplication failed!" );
    }

    // Free hashTable and hashError
    mem->free( ( void* ) hashError, sizeof( bool ) );
    mem->free( ( void* ) chunks, hashTableAllocatedBytes );
    mem->free( ( void* ) chunkList, chunkListAllocatedBytes );
    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR
}

/* ------------------------------------------------------------------------------------------------------------------ */

/* --------------------------------------------------------------------------- */
/*     Template instantiations via registration routine                        */
/* --------------------------------------------------------------------------- */

void CUDACSRUtils::Registrator::registerKernels( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;
    const common::context::ContextType ctx = common::context::CUDA;
    SCAI_LOG_DEBUG( logger, "set CSR routines for CUDA in Interface" )
    KernelRegistry::set<CSRKernelTrait::sizes2offsets>( sizes2offsets, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::offsets2sizes>( offsets2sizes, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::hasDiagonalProperty>( hasDiagonalProperty, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::matrixAddSizes>( matrixAddSizes, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::matrixMultiplySizes>( matrixMultiplySizes, ctx, flag );
}

template<typename ValueType>
void CUDACSRUtils::RegistratorV<ValueType>::registerKernels( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;
    const common::context::ContextType ctx = common::context::CUDA;
    SCAI_LOG_DEBUG( logger, "register CSRUtils CUDA-routines for CUDA at kernel registry [" << flag
                     << " --> " << common::getScalarType<ValueType>() << "]" )
    KernelRegistry::set<CSRKernelTrait::convertCSR2CSC<ValueType> >( convertCSR2CSC, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::normalGEMV<ValueType> >( normalGEMV, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::sparseGEMV<ValueType> >( sparseGEMV, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::normalGEVM<ValueType> >( normalGEVM, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::sparseGEVM<ValueType> >( sparseGEVM, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::matrixAdd<ValueType> >( matrixAdd, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::matrixMultiply<ValueType> >( matrixMultiply, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::jacobi<ValueType> >( jacobi, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::jacobiHalo<ValueType> >( jacobiHalo, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::jacobiHaloWithDiag<ValueType> >( jacobiHaloWithDiag, ctx, flag );
}

template<typename ValueType, typename OtherValueType>
void CUDACSRUtils::RegistratorVO<ValueType, OtherValueType>::registerKernels( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;
    const common::context::ContextType ctx = common::context::CUDA;
    SCAI_LOG_DEBUG( logger, "register CSRUtils CUDA-routines for CUDA at kernel registry [" << flag
                   << " --> " << common::getScalarType<ValueType>() << ", " << common::getScalarType<OtherValueType>() << "]" )
    KernelRegistry::set<CSRKernelTrait::scaleRows<ValueType, OtherValueType> >( scaleRows, ctx, flag );
}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUDACSRUtils::CUDACSRUtils()
{
    SCAI_LOG_INFO( logger, "register CSRUtilsKernel CUDA version" )

    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ADD;
    Registrator::registerKernels( flag );
    kregistry::mepr::RegistratorV<RegistratorV, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
    kregistry::mepr::RegistratorVO<RegistratorVO, SCAI_NUMERIC_TYPES_CUDA_LIST, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
}

CUDACSRUtils::~CUDACSRUtils()
{
    SCAI_LOG_INFO( logger, "unregister CSRUtilsKernel CUDA version" )

    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ERASE;
    Registrator::registerKernels( flag );
    kregistry::mepr::RegistratorV<RegistratorV, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
    kregistry::mepr::RegistratorVO<RegistratorVO, SCAI_NUMERIC_TYPES_CUDA_LIST, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
}

CUDACSRUtils CUDACSRUtils::guard;    // guard variable for registration

/* --------------------------------------------------------------------------- */
/*    Static initialiazion at program start                                    */
/* --------------------------------------------------------------------------- */

unsigned int CUDACSRUtils::lastHashTableSize = 1024;

} /* end namespace sparsekernel */

} /* end namespace scai */
