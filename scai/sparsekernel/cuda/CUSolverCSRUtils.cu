/**
 * @file CUSolverCSRUtils.cu
 *
 * @license
 * Copyright (c) 2009-2016
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * This file is part of the SCAI framework LAMA.
 *
 * LAMA is free software: you can redistribute it and/or modify it under the
 * terms of the GNU Affero General Public License as published by the Free
 * Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * LAMA is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU Affero General Public License for
 * more details.
 *
 * You should have received a copy of the GNU Affero General Public License
 * along with LAMA. If not, see <http://www.gnu.org/licenses/>.
 *
 * Other Usage
 * Alternatively, this file may be used in accordance with the terms and
 * conditions contained in a signed written agreement between you and
 * Fraunhofer SCAI. Please contact our distributor via info[at]scapos.com.
 * @endlicense
 *
 * @brief Implementation of some CSR routines with CUSolver library
 * @author Thomas Brandes
 * @date 11.06.2013
 */

// hpp
#include <scai/sparsekernel/cuda/CUSolverCSRUtils.hpp>

// local library
#include <scai/sparsekernel/cuda/CUSOLVERWrapper.hpp>
#include <scai/sparsekernel/CSRKernelTrait.hpp>

// internal scai libraries
#include <scai/utilskernel/UtilKernelTrait.hpp>
#include <scai/tasking/cuda/CUDAStreamSyncToken.hpp>
#include <scai/kregistry/KernelRegistry.hpp>

#include <scai/tracing.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/CUDAAccess.hpp>
#include <scai/common/Settings.hpp>
#include <scai/common/Constants.hpp>

// CUDA
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#if ( CUDART_VERSION >= 7050 )

namespace scai
{

using tasking::CUDAStreamSyncToken;

namespace sparsekernel
{

SCAI_LOG_DEF_LOGGER( CUSolverCSRUtils::logger, "CUDA.CUSolverCSRUtils" )

/* --------------------------------------------------------------------------- */
/*                          LUFactorization (cuSolver)                         */
/* --------------------------------------------------------------------------- */


template<typename ValueType>
void CUSolverCSRUtils::LUfactorization(
    ValueType* const solution,
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[],
    const ValueType rhs[],
    const IndexType numRows,
    const IndexType nnz )
{
    SCAI_LOG_INFO( logger,
                   "LUfactorization<" << common::getScalarType<ValueType>() << ", matrix numRows = "
                   << numRows << ", nnz = " << nnz )

    typedef CUSOLVERTrait::BLASIndexType BLASIndexType;

    if ( common::TypeTraits<IndexType>::stype
            != common::TypeTraits<BLASIndexType>::stype )
    {
        COMMON_THROWEXCEPTION( "indextype mismatch" );
    }

    // note: SCAI_CHECK_CUDA_ACCESS not required due to getCurrentCUDACtx
    hipsolverSpHandle_t handle = common::CUDAAccess::getCurrentCUDACtx().getcuSolverSpHandle();

	hipsparseMatDescr_t descrA;
    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrA ), "hipsparseCreateMatDescr" )
    hipsparseSetMatType( descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrA, HIPSPARSE_INDEX_BASE_ZERO );

    ValueType tol = 1e-10;
    IndexType reorder = 0;
    IndexType singularity = 0;

    CUSOLVERWrapper<ValueType>::csrQR( handle, numRows, nnz, descrA, csrValues, csrIA, csrJA,
                                       rhs, tol, reorder, solution, &singularity );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "LUfactorization" )
}

/* ------------------------------------------------------------------------------------------------------------------ */

/* --------------------------------------------------------------------------- */
/*     Template instantiations via registration routine                        */
/* --------------------------------------------------------------------------- */

void CUSolverCSRUtils::Registrator::initAndReg( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    /*
    using kregistry::KernelRegistry;
    const common::context::ContextType ctx = common::context::CUDA;
    SCAI_LOG_INFO( logger, "register CUSparseCSRUtils CUSparse-routines for CUDA at kernel registry [" << flag << "]" )
    KernelRegistry::set<CSRKernelTrait::matrixAddSizes>( matrixAddSizes, ctx, flag );
    */
}

template<typename ValueType>
void CUSolverCSRUtils::RegistratorV<ValueType>::initAndReg( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;
    const common::context::ContextType ctx = common::context::CUDA;
    SCAI_LOG_INFO( logger, "register CUSolverCSRUtils CUSolver-routines for CUDA at kernel registry [" << flag << " --> " << common::getScalarType<ValueType>() << "]" )
    KernelRegistry::set<CSRKernelTrait::LUfactorization<ValueType> >( LUfactorization, ctx, flag );
}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUSolverCSRUtils::CUSolverCSRUtils()
{
    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ADD;

    Registrator::initAndReg( flag );
    kregistry::mepr::RegistratorV<RegistratorV, SCAI_ARITHMETIC_CUDA_LIST>::call( flag );
}

CUSolverCSRUtils::~CUSolverCSRUtils()
{
    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ERASE;
    Registrator::initAndReg( flag );
    kregistry::mepr::RegistratorV<RegistratorV, SCAI_ARITHMETIC_CUDA_LIST>::call( flag );
}

CUSolverCSRUtils CUSolverCSRUtils::guard;    // guard variable for registration

} /* end namespace sparsekernel */

} /* end namespace scai */

#endif
