#include "hip/hip_runtime.h"
/**
 * @file CUDADIAUtils.cu
 *
 * @license
 * Copyright (c) 2009-2018
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * This file is part of the SCAI framework LAMA.
 *
 * LAMA is free software: you can redistribute it and/or modify it under the
 * terms of the GNU Lesser General Public License as published by the Free
 * Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * LAMA is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License for
 * more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with LAMA. If not, see <http://www.gnu.org/licenses/>.
 * @endlicense
 *
 * @brief Implementation of DIA utilities with CUDA
 * @author Bea Hornef, Thomas Brandes
 * @date 04.07.2012
 */

// hpp
#include <scai/sparsekernel/cuda/CUDADIAUtils.hpp>

// local library
#include <scai/sparsekernel/DIAKernelTrait.hpp>
#include <scai/utilskernel/cuda/CUDAUtils.hpp>

// internal scai library
#include <scai/tasking/cuda/CUDAStreamSyncToken.hpp>

#include <scai/kregistry/KernelRegistry.hpp>
#include <scai/tracing.hpp>

#include <scai/common/cuda/CUDASettings.hpp>
#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/launchHelper.hpp>
#include <scai/common/cuda/CUDATexVector.hpp>
#include <scai/common/macros/assert.hpp>
#include <scai/common/Constants.hpp>
#include <scai/common/TypeTraits.hpp>

// thrust
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <functional>

using namespace scai::tasking;

namespace scai
{

using common::TypeTraits;
using common::CUDASettings;

namespace sparsekernel
{

SCAI_LOG_DEF_LOGGER( CUDADIAUtils::logger, "CUDA.DIAUtils" )

/* --------------------------------------------------------------------------- */

template<bool useTexture, bool useSharedMemory>
__inline__ __device__
IndexType fetchOffset( const IndexType* const offset_d, IndexType[], const IndexType i )
{
    return offset_d[i];
}

template<>
__inline__ __device__
IndexType fetchOffset<true, false>( const IndexType* const offset_d, IndexType[], const IndexType i )
{
    return fetchVectorX<IndexType, true>( offset_d, i );
}

template<>
__inline__ __device__
IndexType fetchOffset<true, true>( const IndexType* const, IndexType offset_sm[], const IndexType i )
{
    return offset_sm[i];
}

template<>
__inline__ __device__
IndexType fetchOffset<false, true>( const IndexType* const, IndexType offset_sm[], const IndexType i )
{
    return offset_sm[i];
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture, bool useSharedMem>
__global__ void normal_gemv_kernel(
    ValueType* result,
    const ValueType* x,
    const ValueType* y,
    const ValueType alpha,
    const ValueType beta,
    const ValueType* diagonalValues,
    const IndexType* offsets_d,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals )
{
    extern __shared__ IndexType offsets_sm[];

    if ( useSharedMem )
    {
        IndexType k = threadIdx.x;

        while ( k < numDiagonals )
        {
            offsets_sm[k] = offsets_d[k];
            k += blockDim.x;
        }

        __syncthreads();
    }

    IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = beta * y[i];
        ValueType temp = 0.0;

        for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
        {
            IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

            if ( common::Utils::validIndex( j, numColumns ) )
            {
                ValueType val = diagonalValues[ numRows * idiag + i ];
                temp += val * fetchVectorX<ValueType, useTexture>( x, j );
            }
        }

        result[i] = alpha * temp + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture, bool useSharedMem>
__global__ void normal_gemv_kernel_alpha_one_beta_one(
    ValueType* result,
    const ValueType* x,
    const ValueType* y,
    const ValueType* diagonalValues,
    const IndexType* offsets_d,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals )
{
    extern __shared__ IndexType offsets_sm[];

    if ( useSharedMem )
    {
        IndexType k = threadIdx.x;

        while ( k < numDiagonals )
        {
            offsets_sm[k] = offsets_d[k];
            k += blockDim.x;
        }

        __syncthreads();
    }

    IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = y[i];
        ValueType temp = 0.0;

        for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
        {
            IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

            if ( common::Utils::validIndex( j, numColumns ) )
            {
                ValueType val = diagonalValues[ numRows * idiag + i ];
                temp += val * fetchVectorX<ValueType, useTexture>( x, j );
            }
        }

        result[i] = temp + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture, bool useSharedMem>
__global__ void normal_gemv_kernel_alpha_one_beta_zero(
    ValueType* result,
    const ValueType* x,
    const ValueType* y,
    const ValueType* diagonalValues,
    const IndexType* offsets_d,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals )
{
    extern __shared__ IndexType offsets_sm[];

    if ( useSharedMem )
    {
        IndexType k = threadIdx.x;

        while ( k < numDiagonals )
        {
            offsets_sm[k] = offsets_d[k];
            k += blockDim.x;
        }

        __syncthreads();
    }

    IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType temp = 0.0;

        for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
        {
            IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

            if ( common::Utils::validIndex( j, numColumns ) )
            {
                ValueType val = diagonalValues[ numRows * idiag + i ];
                temp += val * fetchVectorX<ValueType, useTexture>( x, j );
            }
        }

        result[i] = temp;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture, bool useSharedMem>
__global__ void normal_gemv_kernel_alpha_one(
    ValueType* result,
    const ValueType* x,
    const ValueType* y,
    const ValueType beta,
    const ValueType* diagonalValues,
    const IndexType* offsets_d,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals )
{
    extern __shared__ IndexType offsets_sm[];

    if ( useSharedMem )
    {
        IndexType k = threadIdx.x;

        while ( k < numDiagonals )
        {
            offsets_sm[k] = offsets_d[k];
            k += blockDim.x;
        }

        __syncthreads();
    }

    IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = 0.0;

        if ( beta != 0.0 )
        {
            summand = beta * y[i];
        }

        ValueType temp = 0.0;

        for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
        {
            IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

            if ( common::Utils::validIndex( j, numColumns ) )
            {
                ValueType val = diagonalValues[ numRows * idiag + i ];
                temp += val * fetchVectorX<ValueType, useTexture>( x, j );
            }
        }

        result[i] = temp + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture, bool useSharedMem>
__global__ void normal_gemv_kernel_beta_one(
    ValueType* result,
    const ValueType* x,
    const ValueType* y,
    const ValueType alpha,
    const ValueType* diagonalValues,
    const IndexType* offsets_d,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals )
{
    extern __shared__ IndexType offsets_sm[];

    if ( useSharedMem )
    {
        IndexType k = threadIdx.x;

        while ( k < numDiagonals )
        {
            offsets_sm[k] = offsets_d[k];
            k += blockDim.x;
        }

        __syncthreads();
    }

    IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = y[i];
        ValueType temp = 0.0;

        for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
        {
            IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

            if ( common::Utils::validIndex( j, numColumns ) )
            {
                ValueType val = diagonalValues[ numRows * idiag + i ];
                temp += val * fetchVectorX<ValueType, useTexture>( x, j );
            }
        }

        result[i] = alpha * temp + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture, bool useSharedMem>
__global__ void normal_gemv_kernel_beta_zero(
    ValueType* result,
    const ValueType* x,
    const ValueType* y,
    const ValueType alpha,
    const ValueType* diagonalValues,
    const IndexType* offsets_d,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals )
{
    extern __shared__ IndexType offsets_sm[];

    if ( useSharedMem )
    {
        IndexType k = threadIdx.x;

        while ( k < numDiagonals )
        {
            offsets_sm[k] = offsets_d[k];
            k += blockDim.x;
        }

        __syncthreads();
    }

    IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType temp = 0.0;

        for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
        {
            IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

            if ( common::Utils::validIndex( j, numColumns ) )
            {
                ValueType val = diagonalValues[ numRows * idiag + i ];
                temp += val * fetchVectorX<ValueType, useTexture>( x, j );
            }
        }

        result[i] = alpha * temp;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture, bool useSharedMem>
__global__ void normal_gevm_kernel(
    ValueType* result,
    const ValueType* x,
    const ValueType* y,
    const ValueType alpha,
    const ValueType beta,
    const ValueType* diagonalValues,
    const IndexType* offsets_d,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals )
{
    extern __shared__ IndexType offsets_sm[];

    if ( useSharedMem )
    {
        IndexType k = threadIdx.x;

        while ( k < numDiagonals )
        {
            offsets_sm[k] = offsets_d[k];
            k += blockDim.x;
        }

        __syncthreads();
    }

    IndexType k = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( k < numColumns )
    {
        ValueType summand = 0;

        if ( beta != 0 )
        {
            summand = beta * y[k];
        }

        ValueType temp = 0.0;

        for ( IndexType ii = 0; ii < numDiagonals; ii++ )
        {
            IndexType i = k - fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, ii );

            if ( common::Utils::validIndex( i, numRows ) )
            {
                temp += diagonalValues[ numRows * ii + i ] * fetchVectorX<ValueType, useTexture>( x, i );
            }
        }

        result[k] = alpha * temp + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture, bool useSharedMem>
static inline void launchGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals,
    const IndexType diaOffsets[],
    const ValueType diaValues[],
    const common::MatrixOp op,
    hipStream_t stream )
{
    const IndexType blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );

    int sharedMemSize = useSharedMem ? numDiagonals * sizeof( IndexType ) : 0;

    if ( common::isTranspose( op ) )
    {
        dim3 dimGrid = makeGrid( numColumns, dimBlock.x );

        normal_gevm_kernel<ValueType, useTexture, useSharedMem> <<< dimGrid, dimBlock, sharedMemSize, stream >>>(
             result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
    }
    else
    {
        dim3 dimGrid = makeGrid( numRows, dimBlock.x );

        // Note: alpha == 0 has already been handled before

        if ( alpha == common::Constants::ONE )
        {
            if ( beta == common::Constants::ZERO )
            {
                normal_gemv_kernel_alpha_one_beta_zero<ValueType, useTexture, useSharedMem> <<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
            else if ( beta == common::Constants::ONE )
            {
                normal_gemv_kernel_alpha_one_beta_one<ValueType, useTexture, useSharedMem> <<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
            else
            {
                normal_gemv_kernel_alpha_one<ValueType, useTexture, useSharedMem> <<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
        }
        else
        {
            if ( beta == common::Constants::ONE )
            {
                normal_gemv_kernel_beta_one<ValueType, useTexture, useSharedMem> <<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
            else if ( beta == common::Constants::ZERO )
            {
                normal_gemv_kernel_beta_zero<ValueType, useTexture, useSharedMem> <<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
            else
            {
                normal_gemv_kernel<ValueType, useTexture, useSharedMem> <<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                    result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDADIAUtils::normalGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numDiagonals,
    const IndexType diaOffsets[],
    const ValueType diaValues[],
    const common::MatrixOp op )
{
    SCAI_REGION( "CUDA.DIA.normalGEMV" )

    IndexType nTarget = common::isTranspose( op ) ? numColumns : numRows;

    SCAI_LOG_INFO( logger, "normalGEMV<" << TypeTraits<ValueType>::id() << ">"
                   << " result[ " << nTarget << "] = " << alpha
                   << " * A( #diags = " << numDiagonals << " ), op = " << op << " * x + " << beta << " * y " )

    if ( alpha == common::Constants::ZERO )
    {
        // result = beta * y 


        utilskernel::CUDAUtils::binaryOpScalar( result, y, beta, nTarget, common::BinaryOp::MULT, false );

        return;
    }

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    const bool useSharedMem = CUDASettings::useSharedMem();

    const bool useTexture = CUDASettings::useTexture();

    SCAI_LOG_INFO( logger, "Start normal_gemv_kernel<" << TypeTraits<ValueType>::id()
                   << "> <<< stream = " << stream
                   << ", useTexture = " << useTexture << ", useSharedMem = " << useSharedMem << ">>>" );

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( !useSharedMem )
        {
            vectorBindTexture( diaOffsets );

            launchGEMV<ValueType, true, false>( result, alpha, x, beta, y, numRows, numColumns, 
                                               numDiagonals, diaOffsets, diaValues, op, stream );
        }
        else
        {
            launchGEMV<ValueType, true, true>( result, alpha, x, beta, y, numRows, numColumns, 
                                               numDiagonals, diaOffsets, diaValues, op, stream );
        }
    }
    else
    {
        if ( useSharedMem )
        {
            launchGEMV<ValueType, false, true>( result, alpha, x, beta, y, numRows, numColumns, 
                                                numDiagonals, diaOffsets, diaValues, op, stream );
        }
        else
        {
            launchGEMV<ValueType, false, false>( result, alpha, x, beta, y, numRows, numColumns, 
                                                 numDiagonals, diaOffsets, diaValues, op, stream );
        }
    }

    if ( !syncToken )
    {
        // synchronize now, unbind used texture
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "normalGEMV for DIA" )

        if ( useTexture )
        {
            vectorUnbindTexture( x );

            if ( !useSharedMem )
            {
                vectorUnbindTexture( diaOffsets );
            }
        }
    }
    else
    {
        // synchronize by syncToken, delay unbind texture
        if ( useTexture )
        {
            void ( *unbindV ) ( const ValueType* ) = &vectorUnbindTexture;
            void ( *unbindI ) ( const IndexType* ) = &vectorUnbindTexture;
            syncToken->pushRoutine( std::bind( unbindV, x ) );

            if ( !useSharedMem )
            {
                syncToken->pushRoutine( std::bind( unbindI, diaOffsets ) );
            }
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDADIAUtils::RegistratorV<ValueType>::registerKernels( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;
    SCAI_LOG_DEBUG( logger, "register DIAUtils CUDA-routines for CUDA at kernel registry [" << flag
                    << " --> " << common::getScalarType<ValueType>() << "]" )
    const common::ContextType ctx = common::ContextType::CUDA;
    KernelRegistry::set<DIAKernelTrait::normalGEMV<ValueType> >( normalGEMV, ctx, flag );
}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUDADIAUtils::CUDADIAUtils()
{
    SCAI_LOG_INFO( logger, "register DIAUtilsKernel CUDA version" )

    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ADD;
    kregistry::mepr::RegistratorV<RegistratorV, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
}

CUDADIAUtils::~CUDADIAUtils()
{
    SCAI_LOG_INFO( logger, "unregister DIAUtilsKernel CUDA version" )

    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ERASE;
    kregistry::mepr::RegistratorV<RegistratorV, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
}

CUDADIAUtils CUDADIAUtils::guard;    // guard variable for registration

} /* end namespace sparsekernel */

} /* end namespace scai */
