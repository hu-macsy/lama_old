#include "hip/hip_runtime.h"
/**
 * @file sparsekernel/cuda/CUDAStencilKernel.cu
 *
 * @license
 * Copyright (c) 2009-2018
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * This file is part of the SCAI framework LAMA.
 *
 * LAMA is free software: you can redistribute it and/or modify it under the
 * terms of the GNU Lesser General Public License as published by the Free
 * Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * LAMA is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License for
 * more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with LAMA. If not, see <http://www.gnu.org/licenses/>.
 * @endlicense
 *
 * @brief CUDA Implementations on GPU for stencil kernels.
 * @author Thomas Brandes
 * @date 04.05.2017
 */

#include <scai/hmemo/HArray.hpp>
#include <scai/tracing.hpp>

#include <scai/common/Grid.hpp>
#include <scai/sparsekernel/cuda/CUDAStencilKernel.hpp>
#include <scai/sparsekernel/StencilKernelTrait.hpp>
#include <scai/tasking/cuda/CUDAStreamSyncToken.hpp>

#include <scai/common/Settings.hpp>

#include <scai/common/cuda/CUDATexVector.hpp>
#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/CUDASettings.hpp>
#include <scai/common/cuda/CUDAUtils.hpp>
#include <scai/common/cuda/launchHelper.hpp>
#include <scai/common/Grid.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>

#include <functional>

#define CUDA_MAX_STENCIL_POINTS 128

namespace scai
{

using common::Grid;
using common::BorderType;

namespace sparsekernel
{

/* --------------------------------------------------------------------------- */

SCAI_LOG_DEF_LOGGER( CUDAStencilKernel::logger, "CUDA.StencilKernel" )

/* --------------------------------------------------------------------------- */

/** Help routine to determine the left position in a dimension with a certain boUnaryOp type. */

__inline__ __device__ 
bool getBorderPosL( IndexType& pos, const IndexType offset, const IndexType size, const IndexType border )
{
    bool valid = true;

    if ( pos >= offset )
    {
        pos = pos - offset;  // is a valid pos
    }
    else if ( border == 0 )
    {
        valid = false;
    }
    else if ( border == 1 )
    {
        pos = ( pos + size ) - offset;
    }

    return valid;
}

/** Help routine to determine the right position in a dimension with a certain boUnaryOp type. */

__inline__ __device__ 
bool getBorderPosR( IndexType& pos, const IndexType offset, const IndexType size, const IndexType border )
{
    bool valid = true;

    if ( pos + offset < size )
    {
        pos += offset;  // is a valid pos
    }
    else if ( border == static_cast<IndexType>( BorderType::ABSORBING ) )
    {
        valid = false;
    }
    else if ( border == static_cast<IndexType>( BorderType::PERIODIC ) )
    {
        pos = ( pos + offset ) - size; 
    }
    return valid;
}

/** Help routine to determine the correct stencil position depending on border types.
 *
 *  Note: gridSizes and gridBorders are available in constant memory.
 */

__inline__ __device__ 
bool getOffsetPos( 
    IndexType pos[],
    const int stencilPositions[],
    const IndexType p,
    const IndexType nDims,
    const IndexType gridSizes[],
    const IndexType gridBorders[] )
{
    bool valid = true;

    for ( IndexType iDim = 0; iDim < nDims; ++iDim )
    {
        int offs = stencilPositions[ nDims * p + iDim ];

        if ( offs < 0 )
        {
            valid = getBorderPosL( pos[iDim], static_cast<IndexType>( -offs ), gridSizes[iDim],  gridBorders[2 * iDim] );
        }
        else if ( offs > 0 )
        {
            valid = getBorderPosR( pos[iDim], static_cast<IndexType>( offs ), gridSizes[iDim], gridBorders[ 2 * iDim + 1] );
        }
        if ( !valid )
        {
            break;
        }
    }

    return valid;
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
__global__
void gemv1Kernel(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[] )
{
    __shared__ IndexType smGridInfo[6];
    __shared__ int smStencilOffset[CUDA_MAX_STENCIL_POINTS];
    __shared__ ValueType smStencilVal[CUDA_MAX_STENCIL_POINTS];

    IndexType* smGridSize = smGridInfo;
    IndexType* smGridDistance = smGridInfo + 1;
    IndexType* smGridBorders  = smGridInfo + 2;
    IndexType* smGridStencilWidth = smGridInfo + 4;

    IndexType tid = threadIdx.x;

    if ( tid < 1 )
    {
        smGridSize[tid] = gridSizes[tid];
        smGridDistance[tid] = gridDistances[tid];
    }

    if ( tid < 2 )
    {
        smGridBorders[tid] = gridBorders[tid];
        smGridStencilWidth[tid] = gridStencilWidth[tid];
    }

    if ( tid < nPoints )
    {
        smStencilOffset[tid] = stencilOffset[tid];
        smStencilVal[tid] = stencilVal[tid];
    }

    __syncthreads();

    const IndexType i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= smGridSize[0] )
    {
        return;   // might happen if gridSizes[0] is not multiple of blockDim.x
    }

    IndexType gridPos = i * smGridDistance[0];

    ValueType v = 0;

    if ( ( i >= smGridStencilWidth[0] ) && ( i < smGridSize[0] - smGridStencilWidth[1] ) )
    {
        // gridPoint ( i ) is inner point, we have not to check for valid stencil points

        for ( IndexType p = 0; p < nPoints; ++p )
        {
            v += smStencilVal[ p ] * x[ gridPos + smStencilOffset[ p ] ];
        }
    }
    else 
    {
        // gridPoint ( i, j) is border point, we have to check each stencil neighbor individually

        for ( IndexType p = 0; p < nPoints; ++p )
        {
            IndexType pos[] = { i };

            bool valid = getOffsetPos( pos, stencilPositions, p, 1, smGridSize, smGridBorders );

            if ( !valid )
            {
                continue;
            }

            IndexType stencilLinearPos = pos[0] * smGridDistance[0];

            v += smStencilVal[ p ] * x[ stencilLinearPos ];
        }
    }

    if ( beta == 0 )
    {
        result[ gridPos] = alpha * v;
    }
    else
    {
        result[ gridPos] = alpha * v + beta * y[ gridPos ];
    }
}   

template<typename ValueType>
void CUDAStencilKernel::stencilGEMV1(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType hostGridSizes[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[] )
{
    SCAI_REGION( "CUDA.Stencil.GEMV1" )

    SCAI_ASSERT_LE_ERROR( nPoints, CUDA_MAX_STENCIL_POINTS, "too many stencil points, increase CUDA_MAX_STENCIL_POINTS" )

    IndexType n0 = hostGridSizes[0];

    SCAI_LOG_INFO( logger,  "stencilGEMV1<" << common::TypeTraits<ValueType>::id() << "> on " << n0 << " grid" )

    dim3 threadsPerBlock( 256 );

    dim3 numBlocks( ( n0 + threadsPerBlock.x - 1 ) / threadsPerBlock.x );

    hipStream_t stream = 0; // default stream if no syncToken is given

    tasking::CUDAStreamSyncToken* syncToken = tasking::CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        // asynchronous execution takes other stream and will not synchronize later
        stream = syncToken->getCUDAStream();
    }

    gemv1Kernel<ValueType><<< numBlocks, threadsPerBlock, 0, stream>>>(
        result, alpha, x, beta, y, nPoints, stencilPositions, stencilVal, stencilOffset,
        gridSizes, gridDistances, gridBorders, gridStencilWidth );
    
    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "gemv1Kernel failed" ) ;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
__global__
void gemv2Kernel(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[] )
{
    __shared__ IndexType smGridInfo[6 * 2];
    __shared__ int smStencilOffset[CUDA_MAX_STENCIL_POINTS];
    __shared__ ValueType smStencilVal[CUDA_MAX_STENCIL_POINTS];

    IndexType* smGridSize = smGridInfo;
    IndexType* smGridDistance = smGridSize + 2;
    IndexType* smGridBorders  = smGridDistance + 2;
    IndexType* smGridStencilWidth = smGridBorders + 2 * 2;

    IndexType tid = threadIdx.x + threadIdx.y * blockDim.x;

    if ( tid < 2 )
    {
        smGridSize[tid] = gridSizes[tid];
        smGridDistance[tid] = gridDistances[tid];
    }

    if ( tid < 4 )
    {
        smGridBorders[tid] = gridBorders[tid];
        smGridStencilWidth[tid] = gridStencilWidth[tid];
    }

    if ( tid < nPoints )
    {
        smStencilOffset[tid] = stencilOffset[tid];
        smStencilVal[tid] = stencilVal[tid];
    }

    __syncthreads();

    const IndexType j = blockIdx.x * blockDim.x + threadIdx.x;
    const IndexType i = blockIdx.y * blockDim.y + threadIdx.y;

    if ( j >= smGridSize[1]  || i >= smGridSize[0] )
    {
        return;
    }

    IndexType gridPos = i * smGridDistance[0] + j * smGridDistance[1];

    ValueType v = 0;

    if (    ( i >= smGridStencilWidth[0] ) && ( i < smGridSize[0] - smGridStencilWidth[1] ) 
         && ( j >= smGridStencilWidth[2] ) && ( j < smGridSize[1] - smGridStencilWidth[3] ) )
    {
        // gridPoint(i,j) is inner point, all stencil points can be applied

        for ( IndexType p = 0; p < nPoints; ++p )
        {
            v += smStencilVal[ p ] * x[ gridPos + smStencilOffset[ p ] ];
        }
    }
    else 
    {
        // gridPoint(i,j) is border point, each stencil neighbor is checked individually

        for ( IndexType p = 0; p < nPoints; ++p )
        {
            IndexType pos[] = { i, j };

            bool valid = getOffsetPos( pos, stencilPositions, p, 2, smGridSize, smGridBorders );

            if ( !valid )
            {
                continue;
            }

            IndexType stencilLinearPos = pos[0] * smGridDistance[0] + pos[1] * smGridDistance[1];

            v += smStencilVal[ p ] * x[ stencilLinearPos ];
        }
    }

    if ( beta == 0 )
    {
        result[ gridPos] = alpha * v;
    }
    else
    {
        result[ gridPos] = alpha * v + beta * y[ gridPos ];
    }
}   

template<typename ValueType>
void CUDAStencilKernel::stencilGEMV2(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType hostGridSizes[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[] )
{
    SCAI_REGION( "CUDA.Stencil.GEMV2" )

    SCAI_ASSERT_LE_ERROR( nPoints, CUDA_MAX_STENCIL_POINTS, "too many stencil points, increase CUDA_MAX_STENCIL_POINTS" )

    IndexType n0 = hostGridSizes[0];
    IndexType n1 = hostGridSizes[1];

    SCAI_LOG_INFO( logger,  "stencilGEMV2<" << common::TypeTraits<ValueType>::id() << "> on " 
                             << n0 << " x " << n1 )

    dim3 threadsPerBlock( 16, 16 );

    dim3 numBlocks( ( n1 + threadsPerBlock.x - 1 ) / threadsPerBlock.x,
                    ( n0 + threadsPerBlock.y - 1 ) / threadsPerBlock.y );


    hipStream_t stream = 0; // default stream if no syncToken is given

    tasking::CUDAStreamSyncToken* syncToken = tasking::CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        // asynchronous execution takes other stream and will not synchronize later
        stream = syncToken->getCUDAStream();
    }

    gemv2Kernel<ValueType><<< numBlocks, threadsPerBlock, 0, stream>>>(
        result, alpha, x, beta, y, nPoints, stencilPositions, stencilVal, stencilOffset,
        gridSizes, gridDistances, gridBorders, gridStencilWidth );
    
    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "gemv2Kernel failed" ) ;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
__global__
void gemv3Kernel(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[] )
{
    __shared__ IndexType smGridInfo[18];
    __shared__ int smStencilOffset[CUDA_MAX_STENCIL_POINTS];
    __shared__ ValueType smStencilVal[CUDA_MAX_STENCIL_POINTS];

    IndexType* smGridSize = smGridInfo;
    IndexType* smGridDistance = smGridInfo + 3;
    IndexType* smGridBorders  = smGridInfo + 6;
    IndexType* smGridStencilWidth = smGridInfo + 12;

    IndexType tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    if ( tid < 3 )
    {
        smGridSize[tid] = gridSizes[tid];
        smGridDistance[tid] = gridDistances[tid];
    }

    if ( tid < 6 )
    {
        smGridBorders[tid] = gridBorders[tid];
        smGridStencilWidth[tid] = gridStencilWidth[tid];
    }

    if ( tid < nPoints )
    {
        smStencilOffset[tid] = stencilOffset[tid];
        smStencilVal[tid] = stencilVal[tid];
    }

    __syncthreads();

    const IndexType k = blockIdx.x * blockDim.x + threadIdx.x;
    const IndexType j = blockIdx.y * blockDim.y + threadIdx.y;
    const IndexType i = blockIdx.z * blockDim.z + threadIdx.z;

    if ( j >= smGridSize[1]  || k >= smGridSize[2] || i >= smGridSize[0] )
    {
        return;
    }

    IndexType gridPos = i * smGridDistance[0] + j * smGridDistance[1] + k * smGridDistance[2];

    ValueType v = 0;

    if (    ( i >= smGridStencilWidth[0] ) && ( i < smGridSize[0] - smGridStencilWidth[1] ) 
         && ( j >= smGridStencilWidth[2] ) && ( j < smGridSize[1] - smGridStencilWidth[3] ) 
         && ( k >= smGridStencilWidth[4] ) && ( k < smGridSize[2] - smGridStencilWidth[5] ) )
    {
        // gridPoint ( i, j, k ) is inner point, we have not to check for valid stencil points

        for ( IndexType p = 0; p < nPoints; ++p )
        {
            v += smStencilVal[ p ] * x[ gridPos + smStencilOffset[ p ] ];
        }
    }
    else 
    {
        // gridPoint ( i, j, k ) is border point, we have to check each stencil neighbor individually

        for ( IndexType p = 0; p < nPoints; ++p )
        {
            IndexType pos[] = { i, j, k };

            bool valid = getOffsetPos( pos, stencilPositions, p, 3, smGridSize, smGridBorders );

            if ( !valid )
            {
                continue;
            }

            IndexType stencilLinearPos =   pos[0] * smGridDistance[0] + pos[1] * smGridDistance[1] 
                                         + pos[2] * smGridDistance[2];

            v += smStencilVal[ p ] * x[ stencilLinearPos ];
        }
    }

    if ( beta == 0 )
    {
        result[ gridPos] = alpha * v;
    }
    else
    {
        result[ gridPos] = alpha * v + beta * y[ gridPos ];
    }
}   

template<typename ValueType>
void CUDAStencilKernel::stencilGEMV3(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType hostGridSizes[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[] )
{
    SCAI_REGION( "CUDA.Stencil.GEMV3" )

    SCAI_ASSERT_LE_ERROR( nPoints, CUDA_MAX_STENCIL_POINTS, "too many stencil points, increase CUDA_MAX_STENCIL_POINTS" )

    SCAI_CHECK_CUDA_ACCESS

    IndexType n0 = hostGridSizes[0];
    IndexType n1 = hostGridSizes[1];
    IndexType n2 = hostGridSizes[2];

    SCAI_LOG_INFO( logger,  "stencilGEMV3<" << common::TypeTraits<ValueType>::id() << "> on " 
                             << n0 << " x " << n1 << " x " << n2 )

    dim3 threadsPerBlock( 16, 4, 4 );

    dim3 numBlocks( ( n2 + threadsPerBlock.x - 1 ) / threadsPerBlock.x,
                    ( n1 + threadsPerBlock.y - 1 ) / threadsPerBlock.y, 
                    ( n0 + threadsPerBlock.y - 1 ) / threadsPerBlock.z );

    hipStream_t stream = 0; // default stream if no syncToken is given

    tasking::CUDAStreamSyncToken* syncToken = tasking::CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        // asynchronous execution takes other stream and will not synchronize later
        stream = syncToken->getCUDAStream();
    }

    gemv3Kernel<ValueType><<< numBlocks, threadsPerBlock, 0, stream>>>( 
        result, alpha, x, beta, y, nPoints, stencilPositions, stencilVal, stencilOffset,
        gridSizes, gridDistances, gridBorders, gridStencilWidth );

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "gemv3Kernel failed" ) ;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
__global__
void gemv4Kernel(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[] )
{
    __shared__ IndexType smGridInfo[24];
    __shared__ int smStencilOffset[CUDA_MAX_STENCIL_POINTS];
    __shared__ ValueType smStencilVal[CUDA_MAX_STENCIL_POINTS];

    IndexType* smGridSize = smGridInfo;
    IndexType* smGridDistance = smGridInfo + 4;
    IndexType* smGridBorders  = smGridInfo + 8;
    IndexType* smGridStencilWidth = smGridInfo + 16;

    IndexType tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    if ( tid < 4 )
    {
        smGridSize[tid] = gridSizes[tid];
        smGridDistance[tid] = gridDistances[tid];
    }

    if ( tid < 8 )
    {
        smGridBorders[tid] = gridBorders[tid];
        smGridStencilWidth[tid] = gridStencilWidth[tid];
    }

    if ( tid < nPoints )
    {
        smStencilOffset[tid] = stencilOffset[tid];
        smStencilVal[tid] = stencilVal[tid];
    }

    __syncthreads();

    const IndexType m = blockIdx.x * blockDim.x + threadIdx.x;
    const IndexType k = blockIdx.y * blockDim.y + threadIdx.y;
    const IndexType ij = ( blockIdx.z * blockDim.z ) + threadIdx.z;
    const IndexType i  = ij / smGridSize[1];
    const IndexType j  = ij - i * smGridSize[1];

    // as grid sizes are not always multiple of the corresponding threads, check for valid grid point

    if ( m >= smGridSize[3]  || k >= smGridSize[2] || j >= smGridSize[1] || i >= smGridSize[0] )
    {
        return;
    }

    IndexType gridPos = i * smGridDistance[0] + j * smGridDistance[1] + k * smGridDistance[2] + m * smGridDistance[3];

    ValueType v = 0;

    if (    ( i >= smGridStencilWidth[0] ) && ( i < smGridSize[0] - smGridStencilWidth[1] ) 
         && ( j >= smGridStencilWidth[2] ) && ( j < smGridSize[1] - smGridStencilWidth[3] ) 
         && ( k >= smGridStencilWidth[4] ) && ( k < smGridSize[2] - smGridStencilWidth[5] ) 
         && ( m >= smGridStencilWidth[6] ) && ( m < smGridSize[3] - smGridStencilWidth[7] ) )
    {
        // gridPoint(i, j, k, m) is inner point, we have not to check for valid stencil points

        for ( IndexType p = 0; p < nPoints; ++p )
        {
            v += smStencilVal[ p ] * x[ gridPos + smStencilOffset[ p ] ];
        }
    }
    else 
    {
        // gridPoint ( i, j, k ) is border point, we have to check each stencil neighbor individually

        for ( IndexType p = 0; p < nPoints; ++p )
        {
            IndexType pos[] = { i, j, k, m };

            bool valid = getOffsetPos( pos, stencilPositions, p, 4, smGridSize, smGridBorders );

            if ( !valid )
            {
                continue;
            }

            IndexType stencilLinearPos =   pos[0] * smGridDistance[0] + pos[1] * smGridDistance[1] 
                                         + pos[2] * smGridDistance[2] + pos[3] * smGridDistance[3];

            v += smStencilVal[ p ] * x[ stencilLinearPos ];
        }
    }

    if ( beta == 0 )
    {
        result[ gridPos] = alpha * v;
    }
    else
    {
        result[ gridPos] = alpha * v + beta * y[ gridPos ];
    }
}   

template<typename ValueType>
void CUDAStencilKernel::stencilGEMV4(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType hostGridSizes[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[] )
{
    SCAI_REGION( "CUDA.Stencil.GEMV4" )

    SCAI_ASSERT_LE_ERROR( nPoints, CUDA_MAX_STENCIL_POINTS, "too many stencil points, increase CUDA_MAX_STENCIL_POINTS" )

    IndexType n0 = hostGridSizes[0];
    IndexType n1 = hostGridSizes[1];
    IndexType n2 = hostGridSizes[2];
    IndexType n3 = hostGridSizes[3];

    SCAI_LOG_INFO( logger, "stencilGEMV4<" << common::TypeTraits<ValueType>::id() << "> on " 
                             << n0 << " x " << n1 << " x " << n2 << " x " << n3 )

    dim3 threadsPerBlock( 16, 4, 4 );

    dim3 numBlocks( ( n3 + threadsPerBlock.x - 1 ) / threadsPerBlock.x,
                    ( n2 + threadsPerBlock.y - 1 ) / threadsPerBlock.y,
                    ( n1 * n0 + threadsPerBlock.z - 1 ) / threadsPerBlock.z );

    hipStream_t stream = 0; // default stream if no syncToken is given

    tasking::CUDAStreamSyncToken* syncToken = tasking::CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        // asynchronous execution takes other stream and will not synchronize later
        stream = syncToken->getCUDAStream();
    }

    gemv4Kernel<ValueType><<< numBlocks, threadsPerBlock, 0, stream>>>(
        result, alpha, x, beta, y, nPoints, stencilPositions, stencilVal, stencilOffset,
        gridSizes, gridDistances, gridBorders, gridStencilWidth );

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "gemv4Kernel failed" ) ;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDAStencilKernel::normalGEMV( 
    ValueType result[], 
    const ValueType alpha,  
    const ValueType x[],
    const ValueType beta,  
    const ValueType y[],
    const IndexType nDims, 
    const IndexType hostGridSizes[],
    const IndexType gridSizes[],
    const IndexType gridDistances[],
    const IndexType gridBorders[],
    const IndexType gridStencilWidth[],
    const IndexType nPoints,
    const int stencilPositions[],
    const ValueType stencilVal[],
    const int stencilOffset[] )
{
    SCAI_LOG_INFO( logger, "normalGEMV" << nDims << ", #points = " << nPoints )

    switch ( nDims ) 
    {
        case 1 : stencilGEMV1( result, alpha, x, beta, y, hostGridSizes,
                               nPoints, stencilPositions, stencilVal, stencilOffset,
                               gridSizes, gridDistances, gridBorders, gridStencilWidth );
                 break;

        case 2 : stencilGEMV2( result, alpha, x, beta, y, hostGridSizes,
                               nPoints, stencilPositions, stencilVal, stencilOffset,
                               gridSizes, gridDistances, gridBorders, gridStencilWidth );
                 break;

        case 3 : stencilGEMV3( result, alpha, x, beta, y, hostGridSizes,
                               nPoints, stencilPositions, stencilVal, stencilOffset,
                               gridSizes, gridDistances, gridBorders, gridStencilWidth );
                 break;

        case 4 : stencilGEMV4( result, alpha, x, beta, y, hostGridSizes,
                               nPoints, stencilPositions, stencilVal, stencilOffset,
                               gridSizes, gridDistances, gridBorders, gridStencilWidth );
                 break;

        default: COMMON_THROWEXCEPTION( "stencilGEMV for nDims = " << nDims << " not supported yet" )
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDAStencilKernel::RegistratorV<ValueType>::registerKernels( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;

    const common::ContextType ctx = common::ContextType::CUDA;

    SCAI_LOG_DEBUG( logger,
                    "register StencilKernel CUDA-routines for Host at kernel registry [" << flag 
                    << " --> " << common::getScalarType<ValueType>() << "]" )

    KernelRegistry::set<StencilKernelTrait::normalGEMV<ValueType> >( normalGEMV, ctx, flag );
}

/* --------------------------------------------------------------------------- */

CUDAStencilKernel::CUDAStencilKernel()
{
    SCAI_LOG_INFO( logger, "register StencilKernel CUDA-routines for Host at kernel registry" )

    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ADD;

    // Registrator::registerKernels( flag );

    kregistry::mepr::RegistratorV<RegistratorV, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
}

/* --------------------------------------------------------------------------- */

CUDAStencilKernel::~CUDAStencilKernel()
{
    SCAI_LOG_INFO( logger, "unregister StencilKernel CUDA-routines for Host at kernel registry" )

    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ERASE;

    // Registrator::registerKernels( flag );

    kregistry::mepr::RegistratorV<RegistratorV, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
}

/* --------------------------------------------------------------------------- */

CUDAStencilKernel CUDAStencilKernel::guard;

/* --------------------------------------------------------------------------- */

}

}
