/**
 * @file CUSparseCSRUtils.cu
 *
 * @license
 * Copyright (c) 2009-2017
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * This file is part of the SCAI framework LAMA.
 *
 * LAMA is free software: you can redistribute it and/or modify it under the
 * terms of the GNU Affero General Public License as published by the Free
 * Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * LAMA is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU Affero General Public License for
 * more details.
 *
 * You should have received a copy of the GNU Affero General Public License
 * along with LAMA. If not, see <http://www.gnu.org/licenses/>.
 *
 * Other Usage
 * Alternatively, this file may be used in accordance with the terms and
 * conditions contained in a signed written agreement between you and
 * Fraunhofer SCAI. Please contact our distributor via info[at]scapos.com.
 * @endlicense
 *
 * @brief Implementation of some CSR routines with CUSparse library 5.0
 * @author Thomas Brandes
 * @date 11.06.2013
 */

// hpp
#include <scai/sparsekernel/cuda/cusparse/CUSparseCSRUtils.hpp>

// local library
#include <scai/sparsekernel/cuda/cusparse/CUSPARSEWrapper.hpp>
#include <scai/sparsekernel/CSRKernelTrait.hpp>

// internal scai libraries
#include <scai/utilskernel/UtilKernelTrait.hpp>
#include <scai/tasking/cuda/CUDAStreamSyncToken.hpp>
#include <scai/kregistry/KernelRegistry.hpp>

#include <scai/tracing.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/CUDAAccess.hpp>
#include <scai/common/Settings.hpp>
#include <scai/common/Constants.hpp>

// CUDA
#include <hip/hip_runtime.h>
#include <hipsparse.h>

namespace scai
{

using tasking::CUDAStreamSyncToken;

namespace sparsekernel
{

SCAI_LOG_DEF_LOGGER( CUSparseCSRUtils::logger, "CUDA.CSRUtilsSparse" )

/* --------------------------------------------------------------------------- */
/*     Template specialization convertCSR2CSC<float>                           */
/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUSparseCSRUtils::convertCSR2CSC(
    IndexType cscIA[],
    IndexType cscJA[],
    ValueType cscValues[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[],
    IndexType numRows,
    IndexType numColumns,
    IndexType numValues )
{
    SCAI_REGION( "CUSparse.CSR.convert2CSC" )

    SCAI_LOG_INFO( logger,
                   "convertCSR2CSC<" << common::getScalarType<ValueType>() << "> -> hipsparseScsr2csc" << ", matrix size = "
                   << numRows << " x " << numColumns << ", nnz = " << numValues )
    typedef CUSPARSETrait::BLASIndexType BLASIndexType;

    if ( common::TypeTraits<IndexType>::stype
            != common::TypeTraits<BLASIndexType>::stype )
    {
        COMMON_THROWEXCEPTION( "indextype mismatch" );
    }

    // note: SCAI_CHECK_CUDA_ACCESS not required due to getCurrentCUDACtx
    hipsparseHandle_t handle = common::CUDAAccess::getCurrentCUDACtx().getcuSparseHandle();
    CUSPARSEWrapper<ValueType>::csr2csc( handle,
                                         numRows, numColumns, numValues,
                                         csrValues, csrIA, csrJA,
                                         cscValues, cscJA, cscIA,
                                         HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             normalGEMV                                                             */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
void CUSparseCSRUtils::normalGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType nnz,
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_REGION( "CUSparse.CSR.normalGEMV" )

    SCAI_LOG_INFO( logger, "normalGEMV<" << common::getScalarType<ValueType>() << ">" <<
                   " result[ " << numRows << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )
    SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )
    SCAI_CHECK_CUDA_ACCESS
    hipStream_t stream = 0; // default stream if no syncToken is given
    hipsparseMatDescr_t descrCSR;
    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )
    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );
    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    if ( y != result && beta != 0.0f )
    {
        SCAI_CUDA_RT_CALL( hipMemcpy( result, y, numRows * sizeof( ValueType ), hipMemcpyDeviceToDevice ),
                           "hipMemcpy for result = y" )
    }

    // call result = alpha * op(A) * x + beta * result of cusparse
    // Note: alpha, beta are passed as pointers
    SCAI_LOG_INFO( logger, "Start cusparseXcsrmv, stream = " << stream )
    hipsparseHandle_t handle = common::CUDAAccess::getCurrentCUDACtx().getcuSparseHandle();
    CUSPARSEWrapper<ValueType>::csrmv( handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       numRows, numColumns, nnz, &alpha, descrCSR,
                                       csrValues, csrIA, csrJA, x, &beta, result );

    if ( syncToken )
    {
        // set back stream for cusparse
        SCAI_CUSPARSE_CALL( hipsparseSetStream( handle, 0 ),
                            "hipsparseSetStream" )
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "cusparseXcsrmv" )
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAddSizes                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

IndexType CUSparseCSRUtils::matrixAddSizes(
    IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const IndexType bIA[],
    const IndexType bJA[] )
{
    SCAI_REGION( "CUDA.cuCSR.matrixAddSizes" )
    SCAI_LOG_INFO(
        logger,
        "matrixAddSizes for " << numRows << " x " << numColumns << " matrix" << ", diagonalProperty = " << diagonalProperty )
    SCAI_CHECK_CUDA_ACCESS
    hipsparseMatDescr_t descrCSR;
    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )
    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );
    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ numColumns ]
    // we have not passed the values, so copy it from device to host
    hipMemcpy( &nnzA, &aIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    int nnzC;
    hipsparseHandle_t handle = common::CUDAAccess::getCurrentCUDACtx().getcuSparseHandle();
    SCAI_CUSPARSE_CALL(
        hipsparseXcsrgeamNnz( handle,
                             numRows, numColumns,
                             descrCSR, nnzA, aIA, aJA,
                             descrCSR, nnzB, bIA, bJA,
                             descrCSR, cIA, &nnzC ),
        "hipsparseXcsrgeamNnz" )
    // synchronization might be redundant due to the return value
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseXcsrgeamNnz" )
    return nnzC;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiplySizes                                                    */
/* ------------------------------------------------------------------------------------------------------------------ */

IndexType CUSparseCSRUtils::matrixMultiplySizes(
    IndexType cIA[],
    const IndexType m,
    const IndexType n,
    const IndexType k,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const IndexType bIA[],
    const IndexType bJA[] )
{
    SCAI_REGION( "CUDA.CSR.matrixMultiplySizes" )
    SCAI_LOG_INFO(
        logger,
        "matrixMutliplySizes for " << m << " x " << n << " matrix" << ", diagonalProperty = " << diagonalProperty )
    SCAI_CHECK_CUDA_ACCESS
    hipsparseMatDescr_t descrCSR;
    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )
    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );
    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ numColumns ]
    // we have not passed the values, so copy it
    hipMemcpy( &nnzA, &aIA[m], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[k], sizeof( IndexType ), hipMemcpyDeviceToHost );
    int nnzC;
    SCAI_LOG_DEBUG( logger, "multSizes, A is " << m << " x " << k << ", nnz = " << nnzA
                    << ", B is " << k << " x " << n << ", nnz = " << nnzB
                    << ", C = " << m << " x " << n )
    hipsparseHandle_t handle = common::CUDAAccess::getCurrentCUDACtx().getcuSparseHandle();
    SCAI_CUSPARSE_CALL(
        hipsparseXcsrgemmNnz( handle,
                             HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             m, n, k,
                             descrCSR, nnzA, aIA, aJA,
                             descrCSR, nnzB, bIA, bJA,
                             descrCSR, cIA, &nnzC ),
        "hipsparseXcsrgemmNnz" )
    // synchronization might be redundant due to the return value
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )
    SCAI_LOG_DEBUG( logger, "matrixMultiplySizes, nnzC = " << nnzC )
    return nnzC;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAdd                                                              */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
void CUSparseCSRUtils::matrixAdd(
    IndexType cJA[],
    ValueType cValues[],
    const IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const ValueType alpha,
    const IndexType aIA[],
    const IndexType aJA[],
    const ValueType aValues[],
    const ValueType beta,
    const IndexType bIA[],
    const IndexType bJA[],
    const ValueType bValues[] )
{
    SCAI_REGION( "CUDA.cuCSR.matrixAdd" )
    SCAI_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )
    SCAI_CHECK_CUDA_ACCESS
    hipsparseMatDescr_t descrCSR;
    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )
    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );
    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ numColumns ]
    // we have not passed the values, so copy it
    hipMemcpy( &nnzA, &aIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    // cIA requires const_cast, but will not be modified
    hipsparseHandle_t handle = common::CUDAAccess::getCurrentCUDACtx().getcuSparseHandle();
    CUSPARSEWrapper<ValueType>::csrgeam( handle,
                                         numRows, numColumns,
                                         &alpha, descrCSR, nnzA, aValues, aIA, aJA,
                                         &beta, descrCSR, nnzB, bValues, bIA, bJA,
                                         descrCSR, cValues, const_cast<IndexType*>( cIA ), cJA );
    // synchronization might be redundant
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "cusparseXcsrgeam" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiply                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
void CUSparseCSRUtils::matrixMultiply(
    const IndexType cIA[],
    IndexType cJA[],
    ValueType cValues[],
    const IndexType m,
    const IndexType n,
    const IndexType k,
    const ValueType alpha,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const ValueType aValues[],
    const IndexType bIA[],
    const IndexType bJA[],
    const ValueType bValues[] )
{
    SCAI_REGION( "CUDA.CSR.matrixMultiply" )
    SCAI_LOG_INFO( logger, "matrixMultiply, result is " << m << "x" << n << " CSR storage" )
    SCAI_CHECK_CUDA_ACCESS
    hipsparseMatDescr_t descrCSR;
    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )
    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );
    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ numColumns ]
    // we have not passed the number of non-zero values for A, B, so copy it
    hipMemcpy( &nnzA, &aIA[m], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[k], sizeof( IndexType ), hipMemcpyDeviceToHost );

    if ( alpha != common::constants::ONE )
    {
        COMMON_THROWEXCEPTION( "cusparseMatrixMultiply only supports alpha = 1, but alpha = " << alpha )
    }

    hipsparseHandle_t handle = common::CUDAAccess::getCurrentCUDACtx().getcuSparseHandle();
    CUSPARSEWrapper<ValueType>::csrgemm( handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         m, n, k,
                                         descrCSR, nnzA, aValues, aIA, aJA,
                                         descrCSR, nnzB, bValues, bIA, bJA,
                                         descrCSR, cValues, cIA, cJA );
    // synchronization might be redundant d
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csrSparseMatmulX" )
}

/* ------------------------------------------------------------------------------------------------------------------ */

/* --------------------------------------------------------------------------- */
/*     Template instantiations via registration routine                        */
/* --------------------------------------------------------------------------- */

void CUSparseCSRUtils::Registrator::registerKernels( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;
    const common::context::ContextType ctx = common::context::CUDA;
    SCAI_LOG_INFO( logger, "register CUSparseCSRUtils CUSparse-routines for CUDA at kernel registry [" << flag << "]" )
    KernelRegistry::set<CSRKernelTrait::matrixAddSizes>( matrixAddSizes, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::matrixMultiplySizes>( matrixMultiplySizes, ctx, flag );
}

template<typename ValueType>
void CUSparseCSRUtils::RegistratorV<ValueType>::registerKernels( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;
    const common::context::ContextType ctx = common::context::CUDA;
    SCAI_LOG_INFO( logger, "register CUSparseCSRUtils CUSparse-routines for CUDA at kernel registry [" << flag
                   << " --> " << common::getScalarType<ValueType>() << "]" )
    KernelRegistry::set<CSRKernelTrait::normalGEMV<ValueType> >( normalGEMV, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::convertCSR2CSC<ValueType> >( convertCSR2CSC, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::matrixAdd<ValueType> >( matrixAdd, ctx, flag );
    KernelRegistry::set<CSRKernelTrait::matrixMultiply<ValueType> >( matrixMultiply, ctx, flag );
}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUSparseCSRUtils::CUSparseCSRUtils()
{
    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_REPLACE;
    bool useCUSparse = true;
    common::Settings::getEnvironment( useCUSparse, "SCAI_CUDA_USE_CUSPARSE" );

    // replace the own CUDA kernels as cuSPARSE library might be more efficient 

    if ( useCUSparse )
    {
        Registrator::registerKernels( flag );
        kregistry::mepr::RegistratorV<RegistratorV, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
    }
}

CUSparseCSRUtils::~CUSparseCSRUtils()
{
    const kregistry::KernelRegistry::KernelRegistryFlag flag = kregistry::KernelRegistry::KERNEL_ERASE;
    bool useCUSparse = true;
    common::Settings::getEnvironment( useCUSparse, "SCAI_CUDA_USE_CUSPARSE" );

    if ( useCUSparse )
    {
        Registrator::registerKernels( flag );
        kregistry::mepr::RegistratorV<RegistratorV, SCAI_NUMERIC_TYPES_CUDA_LIST>::registerKernels( flag );
    }
}

CUSparseCSRUtils CUSparseCSRUtils::guard;    // guard variable for registration

} /* end namespace sparsekernel */

} /* end namespace scai */
