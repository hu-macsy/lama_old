#include "hip/hip_runtime.h"
/**
 * @file common/test/cuda/CUDAKernel.cpp
 *
 * @license
 * Copyright (c) 2009-2015
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Basic tests for LAMA arrays with context/memory at CUDA devices
 * @author: Thomas Brandes
 * @date 08.07.2015
 **/

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/launchHelper.hpp>
#include <scai/common/cuda/CUDASettings.hpp>

#include <scai/tasking/cuda/CUDAStreamSyncToken.hpp>

#include <thrust/device_vector.h>
#include <thrust/fill.h>

using namespace scai;
using namespace tasking;

/* --------------------------------------------------------------------- */

float sum( const float array[], const int n )
{
    thrust::device_ptr<float> data( const_cast<float*>( array ) );

    float zero = static_cast<float>( 0 );

    float result = thrust::reduce( data, data + n, zero, thrust::plus<float>() );

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipStreamSynchronize( 0 )" );

    return result;
}

__global__
void initKernel( float* out, const int n, const float value )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < n )
    {
        out[i] = value;
    }
}

void init( float array[], const int n, const float value )
{
    SCAI_CHECK_CUDA_ACCESS

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    hipStream_t stream = 0;

    if ( syncToken )
    {
        // asynchronous execution takes other stream and will not synchronize later

        stream = syncToken->getCUDAStream();
    }

    const int blockSize = common::CUDASettings::getBlockSize( n );

    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( n, dimBlock.x );

    initKernel <<< dimGrid, dimBlock, 0, stream>>>( array, n, value );
}

