#include "hip/hip_runtime.h"
/**
 * @file CUDABLAS1.cpp
 *
 * @license
 * Copyright (c) 2009-2015
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Wrapper implementations for BLAS1 routines in CUDA using cuBLAS
 * @author Lauretta Schubert, Thomas Brandes, Eric Stricker
 * @date 05.07.2012
 * @since 1.0.0
 */

// hpp
#include <scai/blaskernel/cuda/CUDABLAS1.hpp>

// local library
#include <scai/blaskernel/cuda/cublas_cast.hpp>
#include <scai/blaskernel/cuda/CUBLASWrapper.hpp>
#include <scai/blaskernel/BLASKernelTrait.hpp>

// internal scai libraries
#include <scai/hmemo/cuda/CUDAStreamSyncToken.hpp>
#include <scai/kregistry/KernelRegistry.hpp>

#include <scai/tracing.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/launchHelper.hpp>
#include <scai/common/macros/unused.hpp>
#include <scai/common/TypeTraits.hpp>

// boost
#include <boost/preprocessor.hpp>

using namespace scai::tasking;
using namespace scai::hmemo;
using scai::common::TypeTraits;

namespace scai
{

extern hipblasHandle_t CUDAContext_cublasHandle;

namespace blaskernel
{

SCAI_LOG_DEF_LOGGER( CUDABLAS1::logger, "CUDA.BLAS1" )

/* ---------------------------------------------------------------------------------------*/
/*    sum                                                                                 */
/* ---------------------------------------------------------------------------------------*/

template<typename T>
__global__
void sum_kernel( const int n, T alpha, const T* x, T beta, const T* y, T* z )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < n )
    {
        z[i] = alpha * x[i] + beta * y[i];
    }
}

/* ---------------------------------------------------------------------------------------*/
/*    sum                                                                                 */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
void CUDABLAS1::sum(
    const IndexType n,
    ValueType alpha,
    const ValueType* x,
    ValueType beta,
    const ValueType* y,
    ValueType* z )
{
    SCAI_REGION( "CUDA.BLAS1.sum" )

    if ( n <= 0 )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger,
                    "sum<" << TypeTraits<ValueType>::id() << ">, n = " << n << ", " << alpha << " * x + " << beta << " * y " )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0; // default stream if no syncToken is given

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    const int blockSize = 256;
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( n, dimBlock.x );

    sum_kernel <<< dimGrid, dimBlock, 0, stream>>> ( n, alpha, x, beta, y, z );

    if( !syncToken )
    {
        hipStreamSynchronize( stream );
        SCAI_CHECK_CUDA_ERROR
    }
}

/* ---------------------------------------------------------------------------------------*/
/*    scale                                                                               */
/* ---------------------------------------------------------------------------------------*/

// Note: the cublasWrapper routines could be static routines on its own. But using
//       a common template routine is helpful to guarantee correct syntax

template<typename ValueType>
void CUDABLAS1::scal( IndexType n, const ValueType alpha, ValueType* x_d, const IndexType incX )
{
    SCAI_REGION( "CUDA.BLAS1.scal" )

    if( incX == 0 )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger, "scal<" << TypeTraits<ValueType>::id() << "> of x[" << n << "], alpha = " << alpha )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    SyncToken* syncToken = SyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        SCAI_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::scal set stream" );

    CUBLASWrapper::scal(  static_cast<CUBLASWrapper::BLASIndexType>(n) , alpha, x_d,  static_cast<CUBLASWrapper::BLASIndexType>(incX)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::scal set stream" );
}

/* ---------------------------------------------------------------------------------------*/
/*    nrm2                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
ValueType CUDABLAS1::nrm2( IndexType n, const ValueType* x_d, IndexType incX )
{
    SCAI_REGION( "CUDA.BLAS1.nrm2" )

    if( incX <= 0 )
    {
        return static_cast<ValueType>(0.0);
    }

    SCAI_LOG_DEBUG( logger, "nrm2<" << TypeTraits<ValueType>::id() << "> of x[" << n << "]" )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    // Note: we have to switch cublas Stream, this might be done globally later

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::nrm2 set stream" );

    ValueType res = CUBLASWrapper::nrm2(  static_cast<CUBLASWrapper::BLASIndexType>(n) , x_d,  static_cast<CUBLASWrapper::BLASIndexType>(incX)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::nrm2 set stream null" );

    return res;
}

/* ---------------------------------------------------------------------------------------*/
/*    asum                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
ValueType CUDABLAS1::asum( const IndexType n, const ValueType* x_d, const IndexType incX )
{
    SCAI_REGION( "CUDA.BLAS1.asum" )

    if( incX <= 0 )
    {
        return static_cast<ValueType>(0.0);
    }

    SCAI_LOG_DEBUG( logger, "asum<" << TypeTraits<ValueType>::id() << "> of x[" << n << "]" )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::asum set stream" );

    ValueType res = CUBLASWrapper::asum(  static_cast<CUBLASWrapper::BLASIndexType>(n) , x_d,  static_cast<CUBLASWrapper::BLASIndexType>(incX)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::asum set stream NULL" );
    return res;
}

/* ---------------------------------------------------------------------------------------*/
/*    iamax                                                                               */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
IndexType CUDABLAS1::iamax( const IndexType n, const ValueType* x_d, const IndexType incX )
{
    SCAI_REGION( "CUDA.BLAS1.iamax" )

    SCAI_LOG_DEBUG( logger, "iamax<" << TypeTraits<ValueType>::id() << "> of x[" << n << "]" )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUABLAS1::iamax set stream" );

    IndexType iamax = CUBLASWrapper::iamax(  static_cast<CUBLASWrapper::BLASIndexType>(n) , x_d,  static_cast<CUBLASWrapper::BLASIndexType>(incX) );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::iamax set stream NULL" );
    return iamax ? iamax - 1 : 0;
}

/* ---------------------------------------------------------------------------------------*/
/*    swap                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
void CUDABLAS1::swap(
    const IndexType n,
    ValueType* x_d,
    const IndexType incX,
    ValueType* y_d,
    const IndexType incY )
{
    SCAI_REGION( "CUDA.BLAS1.swap" )

    if( ( incX <= 0 ) || ( incY <= 0 ) )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger, "swap<" << TypeTraits<ValueType>::id() << "> of x, y with size " << n )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS::swap set stream" );

    CUBLASWrapper::swap(  static_cast<CUBLASWrapper::BLASIndexType>(n) , x_d,  static_cast<CUBLASWrapper::BLASIndexType>(incX) , y_d,  static_cast<CUBLASWrapper::BLASIndexType>(incY)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUADABLAS1::swap set stream NULL" );
}

/* ---------------------------------------------------------------------------------------*/
/*    copy                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
void CUDABLAS1::copy(
    IndexType n,
    const ValueType* x_d,
    IndexType incX,
    ValueType* y_d,
    IndexType incY )
{
    SCAI_REGION( "CUDA.BLAS1.copy" )

    if( ( incX <= 0 ) || ( incY <= 0 ) )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger, "copy<" << TypeTraits<ValueType>::id() << "> of x, y, n = " << n )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::copy set stream" );

    CUBLASWrapper::copy(  static_cast<CUBLASWrapper::BLASIndexType>(n) , x_d,  static_cast<CUBLASWrapper::BLASIndexType>(incX) , y_d,  static_cast<CUBLASWrapper::BLASIndexType>(incY) );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::copy set stream NULL" );
}

/* ---------------------------------------------------------------------------------------*/
/*    axpy                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
void CUDABLAS1::axpy(
    int n,
    ValueType alpha,
    const ValueType* x_d,
    int incX,
    ValueType* y_d,
    const int incY )
{
    SCAI_REGION( "CUDA.BLAS1.axpy" )

    if( ( incX <= 0 ) || ( incY <= 0 ) )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger, "axpy<" << TypeTraits<ValueType>::id() << "> of x, y, n = " << n << ", alpha = " << alpha )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::axpy set stream" );

    CUBLASWrapper::axpy(  static_cast<CUBLASWrapper::BLASIndexType>(n) , alpha, x_d,  static_cast<CUBLASWrapper::BLASIndexType>(incX) , y_d,  static_cast<CUBLASWrapper::BLASIndexType>(incY) );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::axpy set stream NULL" );
}

/* ---------------------------------------------------------------------------------------*/
/*    dot                                                                                 */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
ValueType CUDABLAS1::dot(
    IndexType n,
    const ValueType* x_d,
    IndexType incX,
    const ValueType* y_d,
    IndexType incY )
{
    SCAI_REGION( "CUDA.BLAS1.dot" )

    SCAI_LOG_DEBUG( logger,
                    "dot<" << TypeTraits<ValueType>::id() << ">, n = " << n << ", incX = " << incX << ", incY = " << incY << ", x_d = " << x_d << ", y_d = " << y_d )

    if( ( incX <= 0 ) || ( incY <= 0 ) )
    {
        return static_cast<ValueType>(0.0);
    }

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::dot set stream" );

    ValueType res = CUBLASWrapper::dot(  static_cast<CUBLASWrapper::BLASIndexType>(n) , x_d,  static_cast<CUBLASWrapper::BLASIndexType>(incX) , y_d,
    		static_cast<CUBLASWrapper::BLASIndexType>(incY)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::dot set stream NULL" );
    return res;
}

/* --------------------------------------------------------------------------- */
/*     Template instantiations via registration routine                        */
/* --------------------------------------------------------------------------- */

void CUDABLAS1::registerKernels( bool deleteFlag )
{
    using kregistry::KernelRegistry;
    using common::context::CUDA;

    KernelRegistry::KernelRegistryFlag flag = KernelRegistry::KERNEL_ADD;

    if ( deleteFlag )
    {
        flag = KernelRegistry::KERNEL_ERASE;
    }

    SCAI_LOG_INFO( logger, "register BLAS1 routines implemented by CuBLAS in KernelRegistry" )

    // register for one CUDA type: ARITHMETIC_CUDA_TYPE_xxx

#define LAMA_BLAS1_REGISTER(z, I, _)                                                              \
    KernelRegistry::set<BLASKernelTrait::scal<ARITHMETIC_CUDA_TYPE_##I> >( scal, CUDA, flag );    \
    KernelRegistry::set<BLASKernelTrait::nrm2<ARITHMETIC_CUDA_TYPE_##I> >( nrm2, CUDA, flag );    \
    KernelRegistry::set<BLASKernelTrait::asum<ARITHMETIC_CUDA_TYPE_##I> >( asum, CUDA, flag );    \
    KernelRegistry::set<BLASKernelTrait::iamax<ARITHMETIC_CUDA_TYPE_##I> >( iamax, CUDA, flag );  \
    KernelRegistry::set<BLASKernelTrait::swap<ARITHMETIC_CUDA_TYPE_##I> >( swap, CUDA, flag );    \
    KernelRegistry::set<BLASKernelTrait::copy<ARITHMETIC_CUDA_TYPE_##I> >( copy, CUDA, flag );    \
    KernelRegistry::set<BLASKernelTrait::axpy<ARITHMETIC_CUDA_TYPE_##I> >( axpy, CUDA, flag );    \
    KernelRegistry::set<BLASKernelTrait::dot<ARITHMETIC_CUDA_TYPE_##I> >( dot, CUDA, flag );      \
    KernelRegistry::set<BLASKernelTrait::sum<ARITHMETIC_CUDA_TYPE_##I> >( sum, CUDA, flag );      \

    // loop over all supported CUDA types

    BOOST_PP_REPEAT( ARITHMETIC_CUDA_TYPE_CNT, LAMA_BLAS1_REGISTER, _ )

#undef LAMA_BLAS1_REGISTER
}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUDABLAS1::CUDABLAS1()
{
    bool deleteFlag = false;
    registerKernels( deleteFlag );
}

CUDABLAS1::~CUDABLAS1()
{
    bool deleteFlag = true;
    registerKernels( deleteFlag );
}

CUDABLAS1 CUDABLAS1::guard;    // guard variable for registration

} /* end namespace blaskernel */

} /* end namespace scai */
