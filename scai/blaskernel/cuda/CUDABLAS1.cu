#include "hip/hip_runtime.h"
/**
 * @file CUDABLAS1.cpp
 *
 * @license
 * Copyright (c) 2009-2015
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Wrapper implementations for BLAS1 routines in CUDA using cuBLAS
 * @author Lauretta Schubert, Thomas Brandes, Eric Stricker
 * @date 05.07.2012
 * @since 1.0.0
 */

// hpp
#include <scai/blaskernel/cuda/CUDABLAS1.hpp>

// local library
#include <scai/blaskernel/cuda/CUBLASTrait.hpp>
#include <scai/blaskernel/cuda/CUBLASWrapper.hpp>
#include <scai/blaskernel/BLASKernelTrait.hpp>

// internal scai libraries
#include <scai/hmemo/cuda/CUDAStreamSyncToken.hpp>
#include <scai/kregistry/KernelRegistry.hpp>

#include <scai/tracing.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/launchHelper.hpp>
#include <scai/common/macros/unused.hpp>
#include <scai/common/TypeTraits.hpp>
#include <scai/common/mepr/Container.hpp>

using namespace scai::tasking;
using namespace scai::hmemo;
using scai::common::TypeTraits;

namespace scai
{

extern hipblasHandle_t CUDAContext_cublasHandle;

namespace blaskernel
{

SCAI_LOG_DEF_LOGGER( CUDABLAS1::logger, "CUDA.BLAS1" )

/* ---------------------------------------------------------------------------------------*/
/*    sum                                                                                 */
/* ---------------------------------------------------------------------------------------*/

template<typename T>
__global__
void sum_kernel( const int n, T alpha, const T* x, T beta, const T* y, T* z )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < n )
    {
        z[i] = alpha * x[i] + beta * y[i];
    }
}

/* ---------------------------------------------------------------------------------------*/
/*    sum                                                                                 */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
void CUDABLAS1::sum(
    const IndexType n,
    ValueType alpha,
    const ValueType* x,
    ValueType beta,
    const ValueType* y,
    ValueType* z )
{
    SCAI_REGION( "CUDA.BLAS1.sum" )

    if ( n <= 0 )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger,
                    "sum<" << TypeTraits<ValueType>::id() << ">, n = " << n << ", " << alpha << " * x + " << beta << " * y " )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0; // default stream if no syncToken is given

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    const int blockSize = 256;
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( n, dimBlock.x );

    sum_kernel <<< dimGrid, dimBlock, 0, stream>>> ( n, alpha, x, beta, y, z );

    if( !syncToken )
    {
        hipStreamSynchronize( stream );
        SCAI_CHECK_CUDA_ERROR
    }
}

/* ---------------------------------------------------------------------------------------*/
/*    scale                                                                               */
/* ---------------------------------------------------------------------------------------*/

// Note: the cublasWrapper routines could be static routines on its own. But using
//       a common template routine is helpful to guarantee correct syntax

template<typename ValueType>
void CUDABLAS1::scal( IndexType n, const ValueType alpha, ValueType* x_d, const IndexType incX )
{
    SCAI_REGION( "CUDA.BLAS1.scal" )

	typedef CUBLASTrait::BLASIndexType BLASIndexType;

    if( incX == 0 )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger, "scal<" << TypeTraits<ValueType>::id() << "> of x[" << n << "], alpha = " << alpha )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    SyncToken* syncToken = SyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        SCAI_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::scal set stream" );

    CUBLASWrapper<ValueType>::scal(  static_cast<BLASIndexType>(n), alpha, x_d,  static_cast<BLASIndexType>(incX)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::scal set stream" );
}

/* ---------------------------------------------------------------------------------------*/
/*    nrm2                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
ValueType CUDABLAS1::nrm2( IndexType n, const ValueType* x_d, IndexType incX )
{
    SCAI_REGION( "CUDA.BLAS1.nrm2" )

	typedef CUBLASTrait::BLASIndexType BLASIndexType;

    if( incX <= 0 )
    {
        return static_cast<ValueType>(0.0);
    }

    SCAI_LOG_DEBUG( logger, "nrm2<" << TypeTraits<ValueType>::id() << "> of x[" << n << "]" )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    // Note: we have to switch cublas Stream, this might be done globally later

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::nrm2 set stream" );

    ValueType res = CUBLASWrapper<ValueType>::nrm2(  static_cast<BLASIndexType>(n) , x_d,  static_cast<BLASIndexType>(incX)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::nrm2 set stream null" );

    return res;
}

/* ---------------------------------------------------------------------------------------*/
/*    asum                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
ValueType CUDABLAS1::asum( const IndexType n, const ValueType* x_d, const IndexType incX )
{
    SCAI_REGION( "CUDA.BLAS1.asum" )

	typedef CUBLASTrait::BLASIndexType BLASIndexType;

    if( incX <= 0 )
    {
        return static_cast<ValueType>(0.0);
    }

    SCAI_LOG_DEBUG( logger, "asum<" << TypeTraits<ValueType>::id() << "> of x[" << n << "]" )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::asum set stream" );

    ValueType res = CUBLASWrapper<ValueType>::asum(  static_cast<BLASIndexType>(n) , x_d,  static_cast<BLASIndexType>(incX)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::asum set stream NULL" );
    return res;
}

/* ---------------------------------------------------------------------------------------*/
/*    iamax                                                                               */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
IndexType CUDABLAS1::iamax( const IndexType n, const ValueType* x_d, const IndexType incX )
{
    SCAI_REGION( "CUDA.BLAS1.iamax" )

	typedef CUBLASTrait::BLASIndexType BLASIndexType;

    SCAI_LOG_DEBUG( logger, "iamax<" << TypeTraits<ValueType>::id() << "> of x[" << n << "]" )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUABLAS1::iamax set stream" );

    IndexType iamax = CUBLASWrapper<ValueType>::iamax(  static_cast<BLASIndexType>(n) , x_d,  static_cast<BLASIndexType>(incX) );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::iamax set stream NULL" );
    return iamax ? iamax - 1 : 0;
}

/* ---------------------------------------------------------------------------------------*/
/*    swap                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
void CUDABLAS1::swap(
    const IndexType n,
    ValueType* x_d,
    const IndexType incX,
    ValueType* y_d,
    const IndexType incY )
{
    SCAI_REGION( "CUDA.BLAS1.swap" )

	typedef CUBLASTrait::BLASIndexType BLASIndexType;

    if( ( incX <= 0 ) || ( incY <= 0 ) )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger, "swap<" << TypeTraits<ValueType>::id() << "> of x, y with size " << n )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS::swap set stream" );

    CUBLASWrapper<ValueType>::swap(  static_cast<BLASIndexType>(n) , x_d,  static_cast<BLASIndexType>(incX) , y_d,  static_cast<BLASIndexType>(incY)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUADABLAS1::swap set stream NULL" );
}

/* ---------------------------------------------------------------------------------------*/
/*    copy                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
void CUDABLAS1::copy(
    IndexType n,
    const ValueType* x_d,
    IndexType incX,
    ValueType* y_d,
    IndexType incY )
{
    SCAI_REGION( "CUDA.BLAS1.copy" )

	typedef CUBLASTrait::BLASIndexType BLASIndexType;

    if( ( incX <= 0 ) || ( incY <= 0 ) )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger, "copy<" << TypeTraits<ValueType>::id() << "> of x, y, n = " << n )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::copy set stream" );

    CUBLASWrapper<ValueType>::copy(  static_cast<BLASIndexType>(n) , x_d,  static_cast<BLASIndexType>(incX) , y_d,  static_cast<BLASIndexType>(incY) );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::copy set stream NULL" );
}

/* ---------------------------------------------------------------------------------------*/
/*    axpy                                                                                */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
void CUDABLAS1::axpy(
    int n,
    ValueType alpha,
    const ValueType* x_d,
    int incX,
    ValueType* y_d,
    const int incY )
{
    SCAI_REGION( "CUDA.BLAS1.axpy" )

	typedef CUBLASTrait::BLASIndexType BLASIndexType;

    if( ( incX <= 0 ) || ( incY <= 0 ) )
    {
        return;
    }

    SCAI_LOG_DEBUG( logger, "axpy<" << TypeTraits<ValueType>::id() << "> of x, y, n = " << n << ", alpha = " << alpha )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::axpy set stream" );

    CUBLASWrapper<ValueType>::axpy(  static_cast<BLASIndexType>(n) , alpha, x_d,  static_cast<BLASIndexType>(incX) , y_d,  static_cast<BLASIndexType>(incY) );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::axpy set stream NULL" );
}

/* ---------------------------------------------------------------------------------------*/
/*    dot                                                                                 */
/* ---------------------------------------------------------------------------------------*/

template<typename ValueType>
ValueType CUDABLAS1::dot(
    IndexType n,
    const ValueType* x_d,
    IndexType incX,
    const ValueType* y_d,
    IndexType incY )
{
    SCAI_REGION( "CUDA.BLAS1.dot" )

	typedef CUBLASTrait::BLASIndexType BLASIndexType;

    SCAI_LOG_DEBUG( logger,
                    "dot<" << TypeTraits<ValueType>::id() << ">, n = " << n << ", incX = " << incX << ", incY = " << incY << ", x_d = " << x_d << ", y_d = " << y_d )

    if( ( incX <= 0 ) || ( incY <= 0 ) )
    {
        return static_cast<ValueType>(0.0);
    }

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = NULL;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, stream ), "CUDABLAS1::dot set stream" );

    ValueType res = CUBLASWrapper<ValueType>::dot(  static_cast<BLASIndexType>(n) , x_d,  static_cast<BLASIndexType>(incX) , y_d,
    		static_cast<BLASIndexType>(incY)  );

    // No error check here possible as kernel is started asynchronously

    if( !syncToken )
    {
        hipStreamSynchronize( 0 );
        SCAI_CHECK_CUDA_ERROR
    }

    SCAI_CUBLAS_CALL( hipblasSetStream( CUDAContext_cublasHandle, NULL ), "CUDABLAS1::dot set stream NULL" );
    return res;
}

/* --------------------------------------------------------------------------- */
/*     Template instantiations via registration routine                        */
/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDABLAS1::RegistratorV<ValueType>::initAndReg( kregistry::KernelRegistry::KernelRegistryFlag flag )
{
    using kregistry::KernelRegistry;

    const common::context::ContextType ctx = common::context::CUDA;

    SCAI_LOG_INFO( logger, "register BLAS1 routines implemented by CuBLAS in KernelRegistry [" << flag << "]" )

    KernelRegistry::set<BLASKernelTrait::sum<ValueType> >( CUDABLAS1::sum, ctx, flag );
    KernelRegistry::set<BLASKernelTrait::scal<ValueType> >( CUDABLAS1::scal, ctx, flag );
    KernelRegistry::set<BLASKernelTrait::nrm2<ValueType> >( CUDABLAS1::nrm2, ctx, flag );
    KernelRegistry::set<BLASKernelTrait::asum<ValueType> >( CUDABLAS1::asum, ctx, flag );
    KernelRegistry::set<BLASKernelTrait::iamax<ValueType> >( CUDABLAS1::iamax, ctx, flag );
    KernelRegistry::set<BLASKernelTrait::swap<ValueType> >( CUDABLAS1::swap, ctx, flag );
    KernelRegistry::set<BLASKernelTrait::copy<ValueType> >( CUDABLAS1::copy, ctx, flag );
    KernelRegistry::set<BLASKernelTrait::axpy<ValueType> >( CUDABLAS1::axpy, ctx, flag );
    KernelRegistry::set<BLASKernelTrait::dot<ValueType> >( CUDABLAS1::dot, ctx, flag );
}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUDABLAS1::CUDABLAS1()
{
    kregistry::mepr::RegistratorV<RegistratorV, ARITHMETIC_CUDA_LIST>::call(
                    kregistry::KernelRegistry::KERNEL_ADD );
}

CUDABLAS1::~CUDABLAS1()
{
    kregistry::mepr::RegistratorV<RegistratorV, ARITHMETIC_CUDA_LIST>::call(
                    kregistry::KernelRegistry::KERNEL_ERASE );
}

CUDABLAS1 CUDABLAS1::guard;    // guard variable for registration

} /* end namespace blaskernel */

} /* end namespace scai */
