#include "hip/hip_runtime.h"
/**
 * @file CUDACOOUtils.cpp
 *
 * @license
 * Copyright (c) 2009-2013
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Implementation of COO utilities with CUDA
 * @author Bea Hornef, Thomas Brandes
 * @date 04.07.2012
 */

// hpp
#include <scai/lama/cuda/CUDACOOUtils.hpp>

// local library
#include <scai/lama/cuda/CUDAUtils.hpp>
#include <scai/lama/cuda/CUDASettings.hpp>

#include <scai/lama/COOKernelTrait.hpp>

// internal scai library
#include <scai/hmemo/cuda/CUDAStreamSyncToken.hpp>
#include <scai/kregistry/KernelRegistry.hpp>

#include <scai/tracing.hpp>

#include <scai/common/SCAITypes.hpp>
#include <scai/common/bind.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/launchHelper.hpp>
#include <scai/common/Constants.hpp>

// thrust
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

// boost
#include <boost/preprocessor.hpp>

using namespace scai::tasking;

namespace scai
{

using common::getScalarType;

namespace lama
{

SCAI_LOG_DEF_LOGGER( CUDACOOUtils::logger, "CUDA.COOUtils" )

/* --------------------------------------------------------------------------- */

#include <scai/lama/cuda/CUDATexVector.hpp>

/* --------------------------------------------------------------------------- */

__device__ inline void cooAtomicAdd( double* address, double val )
{
    unsigned long long int* address_as_ull =
        ( unsigned long long int* ) address;

    unsigned long long int old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS( address_as_ull, assumed,
                         __double_as_longlong( val +
                                               __longlong_as_double( assumed ) ) );
    }
    while ( assumed != old );
}

__device__ inline void cooAtomicAdd( float* address, float val )

{
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 200

    // CUDA runtime offers faster solution for capability >= 2.0

    atomicAdd( address, val );

#else

    // old slow solution

    int i_val = __float_as_int( val );

    int tmp0 = 0;

    int tmp1;

    while ( ( tmp1 = atomicCAS( ( int* ) address, tmp0, i_val ) ) != tmp0 )

    {
        tmp0 = tmp1;
        i_val = __float_as_int( val + __int_as_float( tmp1 ) );
    }

#endif
}

__device__ inline void cooAtomicAdd( ComplexFloat* address, ComplexFloat val )
{
    cooAtomicAdd( ( float* )&address[0], val.real() );
    cooAtomicAdd( ( float* )&address[1], val.imag() );
}

__device__ inline void cooAtomicAdd( ComplexDouble* address, ComplexDouble val )
{
    cooAtomicAdd( ( double* )&address[0], val.real() );
    cooAtomicAdd( ( double* )&address[1], val.imag() );
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__ void cooGemvKernel(
    ValueType* result,
    const ValueType alpha,
    const ValueType* x,
    const IndexType numValues,
    const IndexType* cooIA,
    const IndexType* cooJA,
    const ValueType* cooValues )
{
    const int k = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( k < numValues )
    {
        IndexType i = cooIA[k];
        IndexType j = cooJA[k];

        // we must use atomic updates as different threads might update same row i

        const ValueType resultUpdate = alpha * cooValues[k] * fetchVectorX<ValueType, useTexture>( x, j );

        // atomic add required, solution above

        cooAtomicAdd( &result[i], resultUpdate );
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__ void cooGemvKernel_alpha_one(
    ValueType* result,
    const ValueType* x,
    const IndexType numValues,
    const IndexType* cooIA,
    const IndexType* cooJA,
    const ValueType* cooValues )
{
    const int k = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( k < numValues )
    {
        IndexType i = cooIA[k];
        IndexType j = cooJA[k];

        // we must use atomic updates as different threads might update same row i

        const ValueType resultUpdate = cooValues[k] * fetchVectorX<ValueType, useTexture>( x, j );

        // atomic add required, solution above

        cooAtomicAdd( &result[i], resultUpdate );
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__ void cooGevmKernel(
    ValueType* result,
    const ValueType alpha,
    const ValueType* x,
    const IndexType numValues,
    const IndexType* cooIA,
    const IndexType* cooJA,
    const ValueType* cooValues )
{
    const int k = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( k < numValues )
    {
        IndexType i = cooIA[k];
        IndexType j = cooJA[k];

        // we must use atomic updates as different threads might update same row i

        const ValueType resultUpdate = alpha * cooValues[k] * fetchVectorX<ValueType, useTexture>( x, i );

        // atomic add required, solution above

        cooAtomicAdd( &result[j], resultUpdate );
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__ void cooGevmKernel_alpha_one(
    ValueType* result,
    const ValueType* x,
    const IndexType numValues,
    const IndexType* cooIA,
    const IndexType* cooJA,
    const ValueType* cooValues )
{
    const int k = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( k < numValues )
    {
        IndexType i = cooIA[k];
        IndexType j = cooJA[k];

        // we must use atomic updates as different threads might update same row i

        const ValueType resultUpdate = cooValues[k] * fetchVectorX<ValueType, useTexture>( x, i );

        // atomic add required, solution above

        cooAtomicAdd( &result[j], resultUpdate );
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACOOUtils::normalGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numValues,
    const IndexType cooIA[],
    const IndexType cooJA[],
    const ValueType cooValues[] )
{
    SCAI_REGION( "CUDA.COO.normalGEMV" )

    SCAI_LOG_INFO( logger, "normalGEMV<" << getScalarType<ValueType>() << ">, "
                   << "result[ " << numRows << "] = " << alpha
                   << " COO( #vals = " << numValues << " ) * x + " << beta << " * y" )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
        SCAI_LOG_INFO( logger, "asyncronous execution on stream " << stream );
    }

    bool useTexture = CUDASettings::useTexture();

    IndexType blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numValues, dimBlock.x );

    // set result = beta * y, not needed if beta == 1 and y == result

    if ( beta == scai::common::constants::ONE && result == y )
    {
        SCAI_LOG_DEBUG( logger, "normalGEMV is sparse, no init of result needed" )
    }
    else
    {
        SCAI_LOG_DEBUG( logger, "normalGEMV, set result = " << beta << " * y " )
        // setScale also deals with y undefined for beta == 0
        CUDAUtils::setScale( result, beta, y, numRows );
    }

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "COO: initGemvKernel FAILED" )

    if ( numValues == 0 )
    {
        return;
    }

    blockSize = CUDASettings::getBlockSize( numValues );
    dimBlock = dim3( blockSize, 1, 1 );
    dimGrid = makeGrid( numValues, dimBlock.x );

    SCAI_LOG_INFO( logger, "Start cooGemvKernel<" << getScalarType<ValueType>()
                   << "> <<< blockSize = " << blockSize << ", stream = " << stream
                   << ", alpha = " << alpha
                   << ", useTexture = " << useTexture << ">>>" )

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == scai::common::constants::ONE )
        {
            cooGemvKernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock>>>
            ( result, x, numValues, cooIA, cooJA, cooValues );
        }
        else
        {
            cooGemvKernel<ValueType, true> <<< dimGrid, dimBlock>>>
            ( result, alpha, x, numValues, cooIA, cooJA, cooValues );
        }
    }
    else
    {
        if ( alpha == scai::common::constants::ONE )
        {
            cooGemvKernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock>>>
            ( result, x, numValues, cooIA, cooJA, cooValues );
        }
        else
        {
            cooGemvKernel<ValueType, false> <<< dimGrid, dimBlock>>>
            ( result, alpha, x, numValues, cooIA, cooJA, cooValues );
        }
    }

    if ( !syncToken )
    {
        // synchronization now, unbind texture if it has been used

        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "COO: gemvKernel FAILED" )

        if ( useTexture )
        {
            vectorUnbindTexture( x );
        }
    }
    else
    {
        // synchronization at SyncToken, delay unbind

        if ( useTexture )
        {
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;

            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACOOUtils::normalGEVM(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numValues,
    const IndexType cooIA[],
    const IndexType cooJA[],
    const ValueType cooValues[] )
{
    SCAI_REGION( "CUDA.COO.normalGEVM" )

    SCAI_LOG_INFO( logger, "normalGEVM, #rows = " << numRows << ", #vals = " << numValues )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
        SCAI_LOG_INFO( logger, "asyncronous execution on stream " << stream );
    }

    bool useTexture = CUDASettings::useTexture();

    IndexType blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numValues, dimBlock.x );

    // set result = beta * y, not needed if beta == 1 and y == result

    if ( beta == scai::common::constants::ONE && result == y )
    {
        SCAI_LOG_DEBUG( logger, "normalGEVM is sparse, no init of result needed" )
    }
    else
    {
        SCAI_LOG_DEBUG( logger, "normalGEMV, set result = " << beta << " * y " )
        CUDAUtils::setScale( result, beta, y, numRows );
    }

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "COO: initGevmKernel FAILED" )

    if ( numValues == 0 )
    {
        return;
    }

    blockSize = CUDASettings::getBlockSize( numValues );
    dimBlock = dim3( blockSize, 1, 1 );
    dimGrid = makeGrid( numValues, dimBlock.x );

    SCAI_LOG_INFO( logger, "Start cooGevmKernel<" << getScalarType<ValueType>()
                   << "> <<< blockSize = " << blockSize << ", stream = " << stream
                   << ", useTexture = " << useTexture << ">>>" )

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == scai::common::constants::ONE )
        {
            cooGevmKernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock>>>
            ( result, x, numValues, cooIA, cooJA, cooValues );
        }
        else
        {
            cooGevmKernel<ValueType, true> <<< dimGrid, dimBlock>>>
            ( result, alpha, x, numValues, cooIA, cooJA, cooValues );
        }
    }
    else
    {
        if ( alpha == scai::common::constants::ONE )
        {
            cooGevmKernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock>>>
            ( result, x, numValues, cooIA, cooJA, cooValues );
        }
        else
        {
            cooGevmKernel<ValueType, false> <<< dimGrid, dimBlock>>>
            ( result, alpha, x, numValues, cooIA, cooJA, cooValues );
        }
    }

    if ( !syncToken )
    {
        // synchronization now, unbind texture if it has been used

        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "COO: gevmKernel FAILED" )

        if ( useTexture )
        {
            vectorUnbindTexture( x );
        }
    }
    else
    {
        // synchronization at SyncToken, delay unbind

        if ( useTexture )
        {
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;

            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */

__global__
static void offsets2ia_kernel( IndexType* cooIA, const IndexType* csrIA, const IndexType numRows, const IndexType numDiagonals )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        IndexType csrOffset = csrIA[i];
        IndexType cooOffset = 0; // additional offset due to diagonals

        if ( i < numDiagonals )
        {
            // diagonal elements will be the first nrows entries

            cooIA[i] = i;
            csrOffset += 1;// do not fill diagonal element again
            cooOffset = numDiagonals - i - 1;// offset in coo moves
        }

        // now fill remaining part of row i

        for ( IndexType jj = csrOffset; jj < csrIA[i + 1]; ++jj )
        {
            cooIA[ jj + cooOffset] = i;
        }
    }
}

/* --------------------------------------------------------------------------- */

void CUDACOOUtils::offsets2ia(
    IndexType cooIA[],
    const IndexType numValues,
    const IndexType csrIA[],
    const IndexType numRows,
    const IndexType numDiagonals )
{
    SCAI_LOG_INFO( logger,
                   "build cooIA( " << numValues << " ) from csrIA( " << ( numRows + 1 )
                   << " ), #diagonals = " << numDiagonals )

    SCAI_CHECK_CUDA_ACCESS

    // make grid

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    offsets2ia_kernel <<< dimGrid, dimBlock>>>( cooIA, csrIA, numRows, numDiagonals );

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "sync for offsets2ia_kernel" )
}

/* --------------------------------------------------------------------------- */

__global__
static void build_offset_kernel(
    IndexType* offsets,
    const IndexType n,
    const IndexType* ia,
    const IndexType nz )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    // Entries in offset filled every time there is a change in values of consecutive elements
    //   i:     0  1  2  3  4  5
    //  ia:     0  0  1  1  1  3
    // nd1:     0  0  1  1  1  3
    // nd2:     0  1  1  1  3  4
    //             x        x  x
    //             |        |  |->                6
    //             |        |---->          5  5
    //             |------------->       2
    // offset:                        0  2  5  5  6

    if ( i < nz )
    {
        IndexType nd1 = ia[i];
        IndexType nd2 = n;

        if ( i + 1 < nz )
        {
            nd2 = ia[i + 1];
        }

        for ( IndexType j = nd1; j < nd2; j++ )
        {
            offsets[j + 1] = i + 1;
        }

        if ( i == 0 )
        {
            for ( IndexType i = 0; i <= nd1; i++ )
            {
                offsets[i] = 0;
            }
        }
    }
}

__global__
static void add_diagonals_kernel(
    IndexType* offsets,
    const IndexType numRows,
    const IndexType numDiagonals )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    //  #diagonals = 3               |
    //  offsets( in ) :  0  3  4   7   9  10
    //  offsets( out ):  0  4  6  10  12  13
    //  i                   0  1   2   3   4

    if ( i < numRows )
    {
        if ( i < numDiagonals )
        {
            offsets[i + 1] += i + 1;
        }
        else
        {
            offsets[i + 1] += numDiagonals;
        }
    }
}

/* --------------------------------------------------------------------------- */

void CUDACOOUtils::ia2offsets(
    IndexType csrIA[],
    const IndexType numRows,
    const IndexType numDiagonals,
    const IndexType cooIA[],
    const IndexType numValues )
{
    SCAI_LOG_INFO( logger,
                   "build csrIA( " << numRows + 1 << " ) from cooIA( " << ( numValues )
                   << " ), #diagonals = " << numDiagonals )

    // Note: the array cooIA is assumed to be sorted after the diagonal elements

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;// default stream, asynchronous execution not supported here

    const int blockSize = CUDASettings::getBlockSize();
    const dim3 dimBlock( blockSize, 1, 1 );
    const dim3 dimGrid = makeGrid( numValues, dimBlock.x );

    build_offset_kernel <<< dimGrid, dimBlock>>>( csrIA, numRows,
            cooIA + numDiagonals, numValues - numDiagonals );

    // increment offsets for the diagonal elements

    if ( numDiagonals > 0 )
    {
        const dim3 dimGrid = makeGrid( numRows, dimBlock.x );

        add_diagonals_kernel <<< dimGrid, dimBlock>>>( csrIA, numRows, numDiagonals );
    }

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "normalGEMV, stream = " << stream )
}

/* --------------------------------------------------------------------------- */

template<typename COOValueType, typename CSRValueType>
__global__
static void csr2coo_kernel( COOValueType* cooValues, const CSRValueType* csrValues,
                            const IndexType* csrIA, const IndexType numRows, const IndexType numDiagonals )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        IndexType csrOffset = csrIA[i];
        IndexType cooOffset = 0; // additional offset due to diagonals

        if ( i < numDiagonals )
        {
            // diagonal elements will be the first nrows entries

            cooValues[i] = csrValues[csrOffset];

            csrOffset += 1;// do not fill diagonal element again
            cooOffset = numDiagonals - i - 1;// offset in coo moves
        }

        // now fill remaining part of row i

        for ( IndexType jj = csrOffset; jj < csrIA[i + 1]; ++jj )
        {
            cooValues[ jj + cooOffset] = static_cast<COOValueType>( csrValues[ jj ] );
        }
    }
}

template<typename COOValueType, typename CSRValueType>
void CUDACOOUtils::setCSRData(
    COOValueType cooValues[],
    const CSRValueType csrValues[],
    const IndexType numValues,
    const IndexType csrIA[],
    const IndexType numRows,
    const IndexType numDiagonals )
{
    SCAI_LOG_INFO( logger,
                   "build cooValues( << " << numValues << " from csrValues + csrIA( " << ( numRows + 1 )
                   << " ), #diagonals = " << numDiagonals )

    SCAI_CHECK_CUDA_ACCESS

    // make grid

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    csr2coo_kernel <<< dimGrid, dimBlock>>>( cooValues, csrValues, csrIA, numRows, numDiagonals );

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "sync for csr2coo_kernel" )
}

/* --------------------------------------------------------------------------- */

void CUDACOOUtils::registerKernels( bool deleteFlag )
{
    SCAI_LOG_INFO( logger, "set COO routines for CUDA in Interface" )

    using namespace scai::kregistry;
    using common::context::CUDA;

    KernelRegistry::KernelRegistryFlag flag = KernelRegistry::KERNEL_ADD ;   // lower priority

    if ( deleteFlag )
    {
        flag = KernelRegistry::KERNEL_ERASE;
    }

    KernelRegistry::set<COOKernelTrait::offsets2ia>( offsets2ia, CUDA, flag );
    KernelRegistry::set<COOKernelTrait::setCSRData<IndexType, IndexType> >( setCSRData, CUDA, flag );

#define LAMA_COO_UTILS2_REGISTER(z, J, TYPE )                                                                    \
    KernelRegistry::set<COOKernelTrait::setCSRData<TYPE, ARITHMETIC_CUDA_TYPE_##J> >( setCSRData, CUDA, flag );  \
     
#define LAMA_COO_UTILS_REGISTER(z, I, _)                                                                  \
    KernelRegistry::set<COOKernelTrait::normalGEMV<ARITHMETIC_CUDA_TYPE_##I> >( normalGEMV, CUDA, flag ); \
    KernelRegistry::set<COOKernelTrait::normalGEVM<ARITHMETIC_CUDA_TYPE_##I> >( normalGEVM, CUDA, flag ); \
                                                                                                          \
    BOOST_PP_REPEAT( ARITHMETIC_CUDA_TYPE_CNT,                                                            \
                     LAMA_COO_UTILS2_REGISTER,                                                            \
                     ARITHMETIC_CUDA_TYPE_##I )                                                           \
     
    BOOST_PP_REPEAT( ARITHMETIC_CUDA_TYPE_CNT, LAMA_COO_UTILS_REGISTER, _ )

#undef LAMA_COO_UTILS_REGISTER
#undef LAMA_COO_UTILS2_REGISTER

}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUDACOOUtils::CUDACOOUtils()
{
    bool deleteFlag = false;
    registerKernels( deleteFlag );
}

CUDACOOUtils::~CUDACOOUtils()
{
    bool deleteFlag = true;
    registerKernels( deleteFlag );
}

CUDACOOUtils CUDACOOUtils::guard;    // guard variable for registration

} /* end namespace lama */

} /* end namespace scai */
