#include "hip/hip_runtime.h"
/**
 * @file CUDACSRUtils.cu
 *
 * @license
 * Copyright (c) 2009-2013
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Implementation of CSR utilities with CUDA
 * @author Bea Hornef, Thomas Brandes, Jiri Kraus
 * @date 04.07.2012
 * @since 1.0.0
 */

// hpp
#include <scai/lama/cuda/CUDACSRUtils.hpp>

// local library
#include <scai/lama/cuda/CUDAUtils.hpp>
#include <scai/lama/cuda/CUDACSRUtils.hpp>
#include <scai/lama/cuda/CUDACOOUtils.hpp>
#include <scai/lama/cuda/CUDATexture.hpp>
#include <scai/lama/cuda/CUDASettings.hpp>

#include <scai/lama/CSRKernelTrait.hpp>

// internal scai library
#include <scai/hmemo/Memory.hpp>
#include <scai/kregistry/KernelRegistry.hpp>

#include <scai/hmemo/cuda/CUDAStreamSyncToken.hpp>

#include <scai/tracing.hpp>

#include <scai/common/SCAITypes.hpp>
#include <scai/common/bind.hpp>
#include <scai/common/Constants.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/cuda/launchHelper.hpp>

#include <scai/common/macros/unused.hpp>

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// thrust
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>

// boost
#include <boost/preprocessor.hpp>

// Parameters for Matrix Multiplication
#define NUM_HASH_RETRIES 16
#define NUM_ELEMENTS_PER_CHUNK 512
#define NUM_ELEMENTS_IN_SHARED 512
#define NUM_BLOCKS 9216
#define NUM_THREADS 32
#define NUM_WARPS NUM_THREADS/32
#define HASH_A 684
#define HASH_B 46165
#define HASH_P 88651
#define HASH_C0 1
#define HASH_C1 1
#define NUM_CHUNKS_PER_WARP 128

using namespace scai::common;
using namespace scai::hmemo;

namespace scai
{

using tasking::SyncToken;
using tasking::CUDAStreamSyncToken;

namespace lama
{

SCAI_LOG_DEF_LOGGER( CUDACSRUtils::logger, "CUDA.CSRUtils" )

IndexType CUDACSRUtils::sizes2offsets( IndexType array[], const IndexType n )
{
    SCAI_LOG_INFO( logger, "sizes2offsets " << " #n = " << n )

    SCAI_CHECK_CUDA_ACCESS

    thrust::device_ptr<IndexType> array_ptr( array );
    thrust::exclusive_scan( array_ptr, array_ptr + n + 1, array_ptr );
    thrust::host_vector<IndexType> numValues( array_ptr + n, array_ptr + n + 1 );

    return numValues[0];
}

/* --------------------------------------------------------------------------- */
/*     CUDA Kernels                                                            */
/* --------------------------------------------------------------------------- */

__global__
static void offsets2sizes_kernel( IndexType sizes[], const IndexType offsets[], const IndexType n )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < n )
    {
        sizes[i] = offsets[i + 1] - offsets[i];
    }
}

/* --------------------------------------------------------------------------- */
/*     offsets2sizes                                                           */
/* --------------------------------------------------------------------------- */

void CUDACSRUtils::offsets2sizes( IndexType sizes[], const IndexType offsets[], const IndexType n )
{
    SCAI_REGION( "CUDA.CSRUtils.offsets2sizes" )

    SCAI_LOG_INFO( logger, "offsets2sizes " << " #n = " << n )

    SCAI_CHECK_CUDA_ACCESS

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( n, dimBlock.x );

    offsets2sizes_kernel <<< dimGrid, dimBlock>>>( sizes, offsets, n );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "offsets2sizes" )
}

/* --------------------------------------------------------------------------- */
/*     hasDiagonalProperty                                                     */
/* --------------------------------------------------------------------------- */

template<typename ValueType>
struct identic_functor
{
    __host__ __device__
    double operator()( thrust::tuple<ValueType, ValueType> x )
    {
        return thrust::get < 0 > ( x ) == thrust::get < 1 > ( x );
    }
};

//trivial kernel to check diagonal property
__global__ void hasDiagonalProperty_kernel(
    const IndexType numDiagonals,
    const IndexType ia[],
    const IndexType ja[],
    bool* hasProperty )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if ( i >= numDiagonals )
    {
        return;
    }

    if ( ! ( *hasProperty ) )
    {
        return;
    }

    if ( ia[i] == ia[i + 1] )
    {
        *hasProperty = false;
    }
    else if ( ja[ia[i]] != i )
    {
        *hasProperty = false;
    }
}

bool CUDACSRUtils::hasDiagonalProperty( const IndexType numDiagonals, const IndexType csrIA[], const IndexType csrJA[] )
{
    SCAI_REGION( "CUDA.CSRUtils.hasDiagonalProperty" )

    if ( numDiagonals == 0 )
    {
        return true;
    }

    SCAI_CHECK_CUDA_ACCESS

    //make grid
    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimGrid( ( numDiagonals - 1 ) / blockSize + 1, 1, 1 );// = makeGrid( numDiagonals, blockSize );
    dim3 dimBlock( blockSize, 1, 1 );

    bool* d_hasProperty;
    bool hasProperty;

    SCAI_CUDA_RT_CALL( hipMalloc( ( void** ) &d_hasProperty, sizeof( bool ) ),
                       "allocate 4 bytes on the device for the result of hasDiagonalProperty_kernel" )
    SCAI_CUDA_RT_CALL( hipMemset( d_hasProperty, 1, sizeof( bool ) ), "memset bool hasProperty = true" )

    hasDiagonalProperty_kernel <<< dimGrid, dimBlock>>>( numDiagonals, csrIA, csrJA, d_hasProperty );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hasDiagonalProperty failed: are ia and ja correct?" )

    SCAI_CUDA_RT_CALL( hipMemcpy( &hasProperty, d_hasProperty, sizeof( bool ), hipMemcpyDeviceToHost ),
                       "copy the result of hasDiagonalProperty_kernel to host" )

    return hasProperty;
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::convertCSR2CSC(
    IndexType cscIA[],
    IndexType cscJA[],
    ValueType cscValues[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[],
    int numRows,
    int numColumns,
    int numValues )
{
    SCAI_REGION( "CUDA.CSRUtils.CSR2CSC" )

    SCAI_LOG_INFO( logger, "convertCSR2CSC of " << numRows << " x " << numColumns << ", nnz = " << numValues )

    // Sort the csrJA ( same as cooJA ), apply it to cooIA and cooValues

    IndexType* cooIA;

    SCAI_CUDA_RT_CALL( hipMalloc( &cooIA, sizeof( IndexType ) * numValues ),
                       "allocate temp for cooIA" )

    // Step 1 : build COO storage,  cooIA (to do), cooJA ( = csrJA ), cooValues ( = csrValues )
    //          -> translate the csrIA offset array to a cooIA array

    const IndexType numDiagonals = 0;// not supported yet

    CUDACOOUtils::offsets2ia( cscJA, numValues, csrIA, numRows, numDiagonals );

    // switch cooIA and cooJA, copy values and resort

    CUDAUtils::set( cooIA, csrJA, numValues );
    CUDAUtils::set( cscValues, csrValues, numValues );

    thrust::device_ptr<IndexType> ja_d( cooIA );
    thrust::device_ptr<ValueType> values_d( cscValues );
    thrust::device_ptr<IndexType> ia_d( cscJA );

    // sort by column indexes in ascending order
    // zip_iterator used to resort cscValues and cscJA in one step

    thrust::stable_sort_by_key( ja_d, ja_d + numValues,
                                thrust::make_zip_iterator( thrust::make_tuple( values_d, ia_d ) ) );

    // cscJA is now sorted, can become an offset array

    CUDACOOUtils::ia2offsets( cscIA, numColumns, 0, cooIA, numValues );

    SCAI_CUDA_RT_CALL( hipFree( cooIA ), "free tmp cooIA" )
}

/* --------------------------------------------------------------------------- */

#include <scai/lama/cuda/CUDATexVector.hpp>

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void scale_kernel(
    ValueType* result,
    const ValueType* y_d,
    const ValueType beta,
    int numRows )
{
    // result = beta * y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        result[i] = beta * y_d[i];
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_beta_zero(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows )
{
    // result = alpha * A * x_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( int jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = alpha * value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_alpha_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType beta,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows )
{
    // result = A * x_d + beta * y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = beta * y_d[i];
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( int jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_beta_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows )
{
    // result = alpha * A * x_d + y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = y_d[i];
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( int jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = alpha * value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_alpha_one_beta_zero(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows )
{
    // result = A * x_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( int jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void assign_kernel(
    ValueType* result,
    const ValueType* y_d,
    int numRows )
{
    // result = y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        result[i] = y_d[i];
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel_alpha_one_beta_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows )
{
    // result = A * x_d + y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = y_d[i];
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( int jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gemv_kernel(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType beta,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows )
{
    // result = alpha * A * x_d + beta * y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType summand = beta * y_d[i];
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( int jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] = alpha * value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_alpha_one_beta_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows,
    int numColumns )
{
    // result = x_d * A + y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType summand = y_d[i];
        ValueType value = 0.0;

        for ( int j = 0; j < numRows; ++j )
        {
            const int rowStart = csrIA[j];
            const int rowEnd = csrIA[j + 1];

            for ( int k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_alpha_one_beta_zero(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows,
    int numColumns )
{
    // result = x_d * A

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( int j = 0; j < numRows; ++j )
        {
            const int rowStart = csrIA[j];
            const int rowEnd = csrIA[j + 1];

            for ( int k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_alpha_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType beta,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows,
    int numColumns )
{
    // result = x_d * A + beta * y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType summand = beta * y_d[i];
        ValueType value = 0.0;

        for ( int j = 0; j < numRows; ++j )
        {
            const int rowStart = csrIA[j];
            const int rowEnd = csrIA[j + 1];

            for ( int k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_beta_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows,
    int numColumns )
{
    // result = alpha * x_d * A + y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( int j = 0; j < numRows; ++j )
        {
            const int rowStart = csrIA[j];
            const int rowEnd = csrIA[j + 1];

            for ( int k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = alpha * value + y_d[i];
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel_beta_zero(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows,
    int numColumns )
{
    // result = alpha * x_d * A

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( int j = 0; j < numRows; ++j )
        {
            const int rowStart = csrIA[j];
            const int rowEnd = csrIA[j + 1];

            for ( int k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = alpha * value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void normal_gevm_kernel(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* y_d,
    const ValueType alpha,
    const ValueType beta,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    int numRows,
    int numColumns )
{
    // result = alpha * x_d * A + beta * y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType summand = beta * y_d[i];
        ValueType value = 0.0;

        for ( int j = 0; j < numRows; ++j )
        {
            const int rowStart = csrIA[j];
            const int rowEnd = csrIA[j + 1];

            for ( int k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, j );
                }
            }
        }

        result[i] = alpha * value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void sparse_gemv_kernel_alpha_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    const IndexType* rowIndexes,
    int numRows )
{
    // result = A * x_d

    const int ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( ii < numRows )
    {
        IndexType i = rowIndexes[ii];
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( int jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] += value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void sparse_gemv_kernel(
    ValueType* result,
    const ValueType* x_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    const IndexType* rowIndexes,
    int numRows )
{
    // result = alpha * A * x_d

    const int ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( ii < numRows )
    {
        IndexType i = rowIndexes[ii];
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        ValueType value = 0.0;

        for ( int jj = rowStart; jj < rowEnd; ++jj )
        {
            value += csrValues[jj] * fetchVectorX<ValueType, useTexture>( x_d, csrJA[jj] );
        }

        result[i] += alpha * value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void sparse_gevm_kernel_alpha_one(
    ValueType* result,
    const ValueType* x_d,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    const IndexType* rowIndexes,
    int numColumns,
    int numNonZeroRows )
{
    // result = x_d * A

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( int jj = 0; jj < numNonZeroRows; ++jj )
        {
            int j = rowIndexes[jj];
            const int rowStart = csrIA[j];
            const int rowEnd = csrIA[j + 1];

            for ( int k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, i );
                }
            }
        }

        result[i] = value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void sparse_gevm_kernel(
    ValueType* result,
    const ValueType* x_d,
    const ValueType alpha,
    const ValueType* csrValues,
    const int* csrIA,
    const int* csrJA,
    const IndexType* rowIndexes,
    int numColumns,
    int numNonZeroRows )
{
    // TODO
    // result = alpha * x_d * A + beta * y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        ValueType value = 0.0;

        for ( int jj = 0; jj < numNonZeroRows; ++jj )
        {
            int j = rowIndexes[jj];
            const int rowStart = csrIA[j];
            const int rowEnd = csrIA[j + 1];

            for ( int k = rowStart; k < rowEnd; ++k )
            {
                if ( csrJA[k] == i )
                {
                    value += csrValues[k] * fetchVectorX<ValueType, useTexture>( x_d, i );
                }
            }
        }

        result[i] = alpha * value;
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  scaleRows                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType, typename OtherValueType>
__global__
void scaleRowsKernel(
    ValueType* values,
    const IndexType* ia,
    const IndexType numRows,
    const OtherValueType* diagonal )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType tmp = static_cast<OtherValueType>( diagonal[i] );

        for ( IndexType j = ia[i]; j < ia[i + 1]; ++j )
        {
            values[j] *= tmp;
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType1, typename ValueType2>
void CUDACSRUtils::scaleRows(
    ValueType1 csrValues[],
    const IndexType csrIA[],
    const IndexType numRows,
    const ValueType2 values[] )
{
    SCAI_REGION( "CUDA.CSRUtils.scaleRows" )

    SCAI_LOG_INFO( logger, "scaleRows<" << getScalarType<ValueType1>() << ","
                   << getScalarType<ValueType2>() << ">"
                   << ", numrows= " << numRows )

    SCAI_CHECK_CUDA_ACCESS

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    scaleRowsKernel <<< dimGrid, dimBlock>>>( csrValues, csrIA, numRows, values );

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "CSRUtils:scaleRowsKernel FAILED" )
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::normalGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType UNUSED( numColumns ),
    const IndexType UNUSED( nnz ),
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_REGION( "CUDA.CSRUtils.normalGEMV" )

    SCAI_LOG_INFO( logger, "normalGEMV<" << getScalarType<ValueType>() << ">" <<
                   " result[ " << numRows << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )

    SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0; // default stream if no syncToken is given

    const int blockSize = CUDASettings::getBlockSize();

    dim3 dimBlock( blockSize, 1, 1 );

    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        // asynchronous execution takes other stream and will not synchronize later

        stream = syncToken->getCUDAStream();
    }

    SCAI_LOG_INFO( logger, "Start normal_gemv_kernel<" << getScalarType<ValueType>()
                   << ", useTexture = " << useTexture << ">" );

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == scai::common::constants::ONE && beta == scai::common::constants::ONE )
        {
            // result = A * x_d + y_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one_beta_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_alpha_one_beta_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == scai::common::constants::ONE && beta == scai::common::constants::ZERO )
        {
            // result = A * x_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one_beta_zero<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_alpha_one_beta_zero<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == scai::common::constants::ZERO && beta == scai::common::constants::ONE )
        {
            // result = y_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( assign_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            assign_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>( result, y, numRows );
        }
        else if ( alpha == scai::common::constants::ONE )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, beta, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == scai::common::constants::ZERO )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( scale_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            scale_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>( result, y, beta, numRows );
        }
        else if ( beta == scai::common::constants::ONE )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_beta_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_beta_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows );
        }
        else if ( beta == scai::common::constants::ZERO )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_beta_zero<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_beta_zero<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows );
        }
        else
        {
            // result = alpha * A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, beta, csrValues, csrIA, csrJA, numRows );
        }
    }
    else
    {
        if ( alpha == scai::common::constants::ONE && beta == scai::common::constants::ONE )
        {
            // result = A * x_d + y_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one_beta_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_alpha_one_beta_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == scai::common::constants::ONE && beta == scai::common::constants::ZERO )
        {
            // result = A * x_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one_beta_zero<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_alpha_one_beta_zero<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == scai::common::constants::ZERO && beta == scai::common::constants::ONE )
        {
            // result = y_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( assign_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            assign_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>( result, y, numRows );
        }
        else if ( alpha == scai::common::constants::ONE )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_alpha_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, beta, csrValues, csrIA, csrJA, numRows );
        }
        else if ( alpha == scai::common::constants::ZERO )
        {
            // result = beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( scale_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            scale_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>( result, y, beta, numRows );
        }
        else if ( beta == scai::common::constants::ONE )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_beta_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_beta_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows );
        }
        else if ( beta == scai::common::constants::ZERO )
        {
            // result = alpha * A * x_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel_beta_zero<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel_beta_zero<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows );
        }
        else
        {
            // result = alpha * A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gemv_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, beta, csrValues, csrIA, csrJA, numRows );
        }
    }

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "normalGEMV, stream = " << stream )
        SCAI_LOG_DEBUG( logger, "normalGEMV<" << getScalarType<ValueType>() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorUnbindTexture( x );
        }
        else
        {
            // get routine with the right signature
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;

            // delay unbind until synchroniziaton
            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::normalGEVM(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_LOG_INFO( logger, "normalGEVM<" << getScalarType<ValueType>() << ">" <<
                   " result[ " << numColumns << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )

    SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0; // default stream if no syncToken is given

    const int blockSize = CUDASettings::getBlockSize();

    dim3 dimBlock( blockSize, 1, 1 );

    dim3 dimGrid = makeGrid( numColumns, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    SCAI_LOG_INFO( logger, "Start normal_gevm_kernel<" << getScalarType<ValueType>()
                   << ", useTexture = " << useTexture << ">" );

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == scai::common::constants::ONE && beta == scai::common::constants::ONE )
        {
            // result = A * x_d + y_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one_beta_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_alpha_one_beta_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == scai::common::constants::ONE && beta == scai::common::constants::ZERO )
        {
            // result = A * x_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one_beta_zero<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_alpha_one_beta_zero<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == scai::common::constants::ZERO && beta == scai::common::constants::ONE )
        {
            // result = y_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( assign_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            assign_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>( result, y, numColumns );
        }
        else if ( alpha == scai::common::constants::ONE )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, beta, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == scai::common::constants::ZERO )
        {
            // result = beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( scale_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            scale_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>( result, y, beta, numColumns );
        }
        else if ( beta == scai::common::constants::ONE )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_beta_one<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_beta_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( beta == scai::common::constants::ZERO )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_beta_zero<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_beta_zero<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else
        {
            // result = alpha * A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, beta, csrValues, csrIA, csrJA, numRows, numColumns );
        }
    }
    else
    {
        if ( alpha == scai::common::constants::ONE && beta == scai::common::constants::ONE )
        {
            // result = A * x_d + y_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one_beta_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_alpha_one_beta_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == scai::common::constants::ONE && beta == scai::common::constants::ZERO )
        {
            // result = A * x_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one_beta_zero<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_alpha_one_beta_zero<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == scai::common::constants::ZERO && beta == scai::common::constants::ONE )
        {
            // result = y_d

            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( assign_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            assign_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>( result, y, numColumns );
        }
        else if ( alpha == scai::common::constants::ONE )
        {
            // result = A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_alpha_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, beta, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( alpha == scai::common::constants::ZERO )
        {
            // result = beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( scale_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            scale_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>( result, y, beta, numColumns );
        }
        else if ( beta == scai::common::constants::ONE )
        {
            // result = alpha * A * x_d + y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_beta_one<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_beta_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else if ( beta == scai::common::constants::ZERO )
        {
            // result = alpha * A * x_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel_beta_zero<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel_beta_zero<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, csrValues, csrIA, csrJA, numRows, numColumns );
        }
        else
        {
            // result = alpha * A * x_d + beta * y_d
            SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

            normal_gevm_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, y, alpha, beta, csrValues, csrIA, csrJA, numRows, numColumns );
        }
    }

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "normalGEVM, stream = " << stream )
        SCAI_LOG_DEBUG( logger, "normalGEVM<" << getScalarType<ValueType>() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorUnbindTexture( x );
        }
        else
        {
            // get routine with the right signature
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;

            // delay unbind until synchroniziaton
            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::sparseGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const IndexType numNonZeroRows,
    const IndexType rowIndexes[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_REGION( "CUDA.CSRUtils.sparseGEMV" )

    SCAI_LOG_INFO( logger,
                   "sparseGEMV<" << getScalarType<ValueType>() << ">" << ", #non-zero rows = " << numNonZeroRows )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    const int blockSize = CUDASettings::getBlockSize( numNonZeroRows );

    dim3 dimBlock( blockSize, 1, 1 );

    dim3 dimGrid = makeGrid( numNonZeroRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == scai::common::constants::ONE )
        {
            sparse_gemv_kernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numNonZeroRows );
        }
        else
        {
            sparse_gemv_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numNonZeroRows );
        }
    }
    else
    {
        if ( alpha == scai::common::constants::ONE )
        {
            sparse_gemv_kernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numNonZeroRows );
        }
        else
        {
            sparse_gemv_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numNonZeroRows );
        }
    }

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "sparseGEMV, stream = " << stream )
        SCAI_LOG_INFO( logger, "sparseGEMV<" << getScalarType<ValueType>() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorUnbindTexture( x );
        }
        else
        {
            // get routine with the right signature
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;

            // delay unbind until synchroniziaton
            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDACSRUtils::sparseGEVM(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const IndexType numColumns,
    const IndexType numNonZeroRows,
    const IndexType rowIndexes[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const ValueType csrValues[] )
{
    SCAI_LOG_INFO( logger,
                   "sparseGEVM<" << getScalarType<ValueType>() << ">" << ", #non-zero rows = " << numNonZeroRows )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    // check if asynchronous execution is wanted

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    const int blockSize = CUDASettings::getBlockSize( numNonZeroRows );

    dim3 dimBlock( blockSize, 1, 1 );

    dim3 dimGrid = makeGrid( numNonZeroRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    if ( useTexture )
    {
        vectorBindTexture( x );

        if ( alpha == scai::common::constants::ONE )
        {
            sparse_gevm_kernel_alpha_one<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, csrValues, csrIA, csrJA, rowIndexes, numColumns, numNonZeroRows );
        }
        else
        {
            sparse_gevm_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numColumns, numNonZeroRows );
        }
    }
    else
    {
        if ( alpha == scai::common::constants::ONE )
        {
            sparse_gevm_kernel_alpha_one<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, csrValues, csrIA, csrJA, rowIndexes, numColumns, numNonZeroRows );
        }
        else
        {
            sparse_gevm_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
            ( result, x, alpha, csrValues, csrIA, csrJA, rowIndexes, numColumns, numNonZeroRows );
        }
    }

    if ( !syncToken )
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( stream ), "sparseGEVM, stream = " << stream )
        SCAI_LOG_INFO( logger, "sparseGEVM<" << getScalarType<ValueType>() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorUnbindTexture( x );
        }
        else
        {
            // get routine with the right signature
            void ( *unbind ) ( const ValueType* ) = &vectorUnbindTexture;

            // delay unbind until synchroniziaton
            syncToken->pushRoutine( common::bind( unbind, x ) );
        }
    }
}

/* --------------------------------------------------------------------------- */
/*                          Jacobi                                             */
/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void csr_jacobi_kernel(
    const int* const csrIA,
    const int* const csrJA,
    const ValueType* const csrValues,
    const int numRows,
    const ValueType* const rhs,
    ValueType* const solution,
    const ValueType* const oldSolution,
    const ValueType omega )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        ValueType temp = rhs[i];
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        const ValueType diag = csrValues[rowStart];

        for ( int jj = rowStart + 1; jj < rowEnd; ++jj )
        {
            temp -= csrValues[jj] * fetchVectorX<ValueType, useTexture>( oldSolution, csrJA[jj] );
        }

        if ( omega == 0.5 )
        {
            solution[i] = omega * ( fetchVectorX<ValueType, useTexture>( oldSolution, i ) + temp / diag );
        }
        else if ( omega == 1.0 )
        {
            solution[i] = temp / diag;
        }
        else
        {
            solution[i] = omega * ( temp / diag ) + ( 1.0 - omega ) * fetchVectorX<ValueType, useTexture>( oldSolution, i );
        }
    }
}

template<typename ValueType>
__inline__ __device__ ValueType getSharedValue( ValueType* shared, const ValueType* const value, const int index )
{
    if ( index / blockDim.x == blockIdx.x )
    {
        return shared[index % blockDim.x];
    }
    else
    {
        return value[index];
    }
}

//these templates allow to combine dynamic shared memory with templates
template<typename ValueType>
struct SharedMemory
{
    //! @brief Return a pointer to the runtime-sized shared memory array.
    //! @returns Pointer to runtime-sized shared memory array
    __device__
    ValueType* getPointer()
    {
        extern __device__ void Error_UnsupportedType(); // Ensure that we won't compile any un-specialized types
        Error_UnsupportedType();
        return ( ValueType* ) 0;
    }

};

template<>
struct SharedMemory<float>
{
    __device__
    float* getPointer()
    {
        extern __shared__ float s_float[];
        return s_float;
    }
};

template<>
struct SharedMemory<double>
{
    __device__
    double* getPointer()
    {
        extern __shared__ double s_double[];
        return s_double;
    }
};

//this is just like the other jacobi kernel, but it performs a coalesced prefetch of the old solution
//instead of using the texture memory
template<typename ValueType>
__global__ void csr_alternate_jacobi_kernel(
    const int* const csrIA,
    const int* const csrJA,
    const ValueType* const csrValues,
    const int numRows,
    const ValueType* const rhs,
    ValueType* const solution,
    const ValueType* const oldSolution,
    const ValueType omega )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    SharedMemory<ValueType> smem;
    ValueType* shared = smem.getPointer();

    if ( i < numRows )
    {
        //this is the prefetch
        shared[threadIdx.x] = oldSolution[i];
        __syncthreads();

        ValueType temp = rhs[i];
        const int rowStart = csrIA[i];
        const int rowEnd = csrIA[i + 1];
        const ValueType diag = csrValues[rowStart];

        for ( int jj = rowStart + 1; jj < rowEnd; ++jj )
        {
            temp -= csrValues[jj] * getSharedValue<ValueType>( shared, oldSolution, csrJA[jj] );
        }

        if ( omega == 0.5 )
        {
            solution[i] = omega * ( getSharedValue<ValueType>( shared, oldSolution, i ) + temp / diag );
        }
        else if ( omega == 1.0 )
        {
            solution[i] = temp / diag;
        }
        else
        {
            solution[i] = omega * ( temp / diag ) + ( 1.0 - omega ) * getSharedValue<ValueType>( shared, oldSolution, i );
        }
    }
}

template<typename ValueType>
void CUDACSRUtils::jacobi(
    ValueType* const solution,
    const IndexType* const csrIA,
    const IndexType* const csrJA,
    const ValueType* const csrValues,
    const ValueType* const oldSolution,
    const ValueType* const rhs,
    const ValueType omega,
    const IndexType numRows )
{
    SCAI_LOG_INFO( logger, "jacobi, #rows = " << numRows )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    bool useTexture = CUDASettings::useTexture();

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();

        useTexture = false; // not yet supported
    }

    const int blockSize = CUDASettings::getBlockSize();

    dim3 dimBlock( blockSize, 1, 1 );

    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    SCAI_LOG_INFO( logger, "Start csr_jacobi_kernel<" << getScalarType<ValueType>()
                   << ", useTexture = " << useTexture << ">" );

    if ( useTexture )
    {
        vectorBindTexture( oldSolution );

        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobi_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        csr_jacobi_kernel <ValueType, true> <<< dimGrid, dimBlock, 0, stream>>>( csrIA, csrJA, csrValues, numRows,
                rhs, solution, oldSolution, omega );
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobi_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        csr_jacobi_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream>>>( csrIA, csrJA, csrValues, numRows, rhs,
                solution, oldSolution, omega );
    }

    if ( !syncToken )
    {
        hipStreamSynchronize( stream );
    }

    if ( useTexture )
    {
        vectorUnbindTexture( oldSolution );
    }
}

/* --------------------------------------------------------------------------- */
/*                          Jacobi halo                                        */
/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void csr_jacobiHalo_kernel(
    ValueType* const solution,
    const IndexType* const localIA,
    const ValueType* const localValues,
    const IndexType* const haloIA,
    const IndexType* const haloJA,
    const ValueType* const haloValues,
    const IndexType* const rowIndexes,
    const IndexType numNonEmptyRows,
    const ValueType* const oldSolution,
    const ValueType omega )
{
    const IndexType ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( ii < numNonEmptyRows )
    {
        IndexType i = ii; // default: rowIndexes is identity

        if ( rowIndexes )
        {
            i = rowIndexes[ii];
        }

        ValueType temp = 0.0;

        const IndexType rowStart = haloIA[i];
        const IndexType rowEnd = haloIA[i + 1];

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            temp += haloValues[jj] * fetchVectorX<ValueType, useTexture>( oldSolution, haloJA[jj] );
        }

        const ValueType diag = localValues[localIA[i]];

        solution[i] -= temp * ( omega / diag );
    }
}

template<typename ValueType>
void CUDACSRUtils::jacobiHalo(
    ValueType solution[],
    const IndexType localIA[],
    const ValueType localValues[],
    const IndexType haloIA[],
    const IndexType haloJA[],
    const ValueType haloValues[],
    const IndexType haloRowIndexes[],
    const ValueType oldSolution[],
    const ValueType omega,
    const IndexType numNonEmptyRows )
{
    SCAI_LOG_INFO( logger, "jacobiHalo, #non-empty rows = " << numNonEmptyRows )

    SCAI_CHECK_CUDA_ACCESS

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numNonEmptyRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    useTexture = false;

    if ( useTexture )
    {
        vectorBindTexture( oldSolution );

        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobiHalo_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        csr_jacobiHalo_kernel <ValueType, true> <<< dimGrid, dimBlock>>>( solution, localIA, localValues, haloIA,
                haloJA, haloValues, haloRowIndexes,
                numNonEmptyRows, oldSolution, omega );
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobiHalo_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        csr_jacobiHalo_kernel<ValueType, false> <<< dimGrid, dimBlock>>>( solution, localIA, localValues, haloIA,
                haloJA, haloValues, haloRowIndexes, numNonEmptyRows,
                oldSolution, omega );
    }

    SCAI_CUDA_RT_CALL( hipGetLastError(), "LAMA_STATUS_CSRJACOBIHALO_CUDAKERNEL_FAILED" )
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "LAMA_STATUS_CSRJACOBIHALO_CUDAKERNEL_FAILED" )

    if ( useTexture )
    {
        vectorUnbindTexture( oldSolution );
    }
}

/* --------------------------------------------------------------------------- */
/*                          Jacobi halo with diagonal array                    */
/* --------------------------------------------------------------------------- */

template<typename ValueType, bool useTexture>
__global__
void csr_jacobiHaloWithDiag_kernel(
    ValueType* const solution,
    const ValueType* const localDiagValues,
    const IndexType* const haloIA,
    const IndexType* const haloJA,
    const ValueType* const haloValues,
    const IndexType* const rowIndexes,
    const IndexType numNonEmptyRows,
    const ValueType* const oldSolution,
    const ValueType omega )
{
    const IndexType ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( ii < numNonEmptyRows )
    {
        IndexType i = ii; // default: rowIndexes is identity

        if ( rowIndexes )
        {
            i = rowIndexes[ii];
        }

        ValueType temp = 0.0;

        const IndexType rowStart = haloIA[i];
        const IndexType rowEnd = haloIA[i + 1];

        for ( IndexType jj = rowStart; jj < rowEnd; ++jj )
        {
            temp += haloValues[jj] * fetchVectorX<ValueType, useTexture>( oldSolution, haloJA[jj] );
        }

        const ValueType diag = localDiagValues[i];

        solution[i] -= temp * ( omega / diag );
    }
}

template<typename ValueType>
void CUDACSRUtils::jacobiHaloWithDiag(
    ValueType solution[],
    const ValueType localDiagValues[],
    const IndexType haloIA[],
    const IndexType haloJA[],
    const ValueType haloValues[],
    const IndexType haloRowIndexes[],
    const ValueType oldSolution[],
    const ValueType omega,
    const IndexType numNonEmptyRows )
{
    SCAI_LOG_INFO( logger, "jacobiHaloWithDiag, #non-empty rows = " << numNonEmptyRows )

    SCAI_CHECK_CUDA_ACCESS

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numNonEmptyRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    useTexture = false;

    if ( useTexture )
    {
        vectorBindTexture( oldSolution );

        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobiHaloWithDiag_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr_jacobiHaloWithDiag_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

    }

    if ( useTexture )
    {
        csr_jacobiHaloWithDiag_kernel <ValueType, true> <<< dimGrid, dimBlock>>>( solution, localDiagValues, haloIA,
                haloJA, haloValues, haloRowIndexes,
                numNonEmptyRows, oldSolution, omega );
    }
    else
    {
        csr_jacobiHaloWithDiag_kernel<ValueType, false> <<< dimGrid, dimBlock>>>( solution, localDiagValues, haloIA,
                haloJA, haloValues, haloRowIndexes, numNonEmptyRows,
                oldSolution, omega );
    }

    SCAI_CUDA_RT_CALL( hipGetLastError(), "LAMA_STATUS_CSRJACOBIHALOWITHDIAG_CUDAKERNEL_FAILED" )
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "LAMA_STATUS_CSRJACOBIHALOWITHDIAG_CUDAKERNEL_FAILED" )

    if ( useTexture )
    {
        vectorUnbindTexture( oldSolution );
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             helper                                                                 */
/* ------------------------------------------------------------------------------------------------------------------ */

__device__ __inline__ IndexType multHlp_getNumActiveThreads(
    IndexType aColIt,
    IndexType aColEnd,
    const IndexType* aIA,
    IndexType aRowIt,
    IndexType offset )
{
#ifdef CUDA_CAP_20
    IndexType end = __popc ( __ballot ( aColIt < aColEnd ) );
#else
    IndexType aColStart = aIA[aRowIt] + offset;
    IndexType end = ( aColEnd - aColStart );
#endif
    return end;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAddSizes                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template<int nWarps>
__global__ void matrixAddSizesKernel(
    IndexType* cIa,
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const IndexType* aIa,
    const IndexType* aJa,
    const IndexType* bIa,
    const IndexType* bJa )
{
    __shared__ volatile IndexType sColA[nWarps];
    __shared__ volatile bool sFound[nWarps];

    IndexType localWarpId = threadIdx.x / warpSize;
    IndexType globalWarpId = ( blockIdx.x * blockDim.x + threadIdx.x ) / warpSize;
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
//IndexType numWarpsLocal  = blockDim.x / warpSize;
    IndexType numWarpsGlobal = ( blockDim.x * gridDim.x ) / warpSize;
    IndexType rowIt = globalWarpId;

    for ( ; __any( rowIt < numRows ); rowIt += numWarpsGlobal )
    {
        if ( rowIt < numRows )
        {
            if ( diagonalProperty && rowIt >= numColumns )
            {
                diagonalProperty = false;
            }

            IndexType aColIt = aIa[rowIt] + laneId;
            IndexType aColEnd = aIa[rowIt + 1];

            IndexType bColIt = bIa[rowIt] + laneId;
            IndexType bColEnd = bIa[rowIt + 1];

            if ( laneId == 0 )
            {
                cIa[rowIt] = bColEnd - bColIt;
            }

            for ( IndexType aColItOffset = 0; __any( aColIt < aColEnd ); aColIt += warpSize, aColItOffset += warpSize )
            {
                IndexType colA = aColIt < aColEnd ? aJa[aColIt] : -1;
                IndexType end = multHlp_getNumActiveThreads( aColIt, aColEnd, aIa, rowIt, aColItOffset );

                for ( IndexType k = 0; k < end && k < warpSize; k++ )
                {
                    if ( laneId == k )
                    {
                        sColA[localWarpId] = colA;
                    }

                    sFound[localWarpId] = false;

                    for ( IndexType bColItOffset = 0; !sFound[localWarpId] && __any( ( bColIt + bColItOffset ) < bColEnd );
                            bColItOffset += warpSize )
                    {
                        IndexType colB = ( bColIt + bColItOffset ) < bColEnd ? bJa[bColIt + bColItOffset] : -1;

                        if ( sColA[localWarpId] == colB )
                        {
                            sFound[localWarpId] = true;
                        }
                    }

                    if ( laneId == 0 && !sFound[localWarpId] )
                    {
                        cIa[rowIt]++;
                    }
                }
            }
        }
    }
}

IndexType CUDACSRUtils::matrixAddSizes(
    IndexType cIa[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const IndexType aIa[],
    const IndexType aJa[],
    const IndexType bIa[],
    const IndexType bJa[] )
{
    SCAI_REGION( "CUDA.CSRUtils.matrixAddSizes" )

    SCAI_LOG_INFO(
        logger,
        "matrixAddSizes for " << numRows << " x " << numColumns << " matrix" << ", diagonalProperty = " << diagonalProperty )

    SCAI_CHECK_CUDA_ACCESS

// Reset cIa
    thrust::device_ptr<IndexType> cIaPtr( cIa );
    thrust::fill( cIaPtr, cIaPtr + numRows, 0 );

// TODO: Check if diagonal property needs special attention
    matrixAddSizesKernel<NUM_WARPS> <<< NUM_BLOCKS, NUM_THREADS>>>( cIa, numRows, numColumns, diagonalProperty,
            aIa, aJa, bIa, bJa );

    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR

// Convert sizes array to offset array
    thrust::exclusive_scan( cIaPtr, cIaPtr + numRows + 1, cIaPtr );

// Copy numValues from cIa to Host
// TODO: use cuMem cpy
    thrust::device_ptr<IndexType> iaPtr( cIa );
    thrust::host_vector<IndexType> numValues( iaPtr + numRows, iaPtr + numRows + 1 );

    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR

// TODO: write it!
    return numValues[0];
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             hashTable Methods                                                      */
/* ------------------------------------------------------------------------------------------------------------------ */

__device__
inline bool multHlp_insertIndexex( IndexType colB,
                                   IndexType sHashTableIndexes[],
                                   IndexType aRowIt,
                                   IndexType* chunkPtr,
                                   volatile int chunkList[],
                                   int numReservedChunks,
                                   IndexType* cIA )
{
    unsigned int fx = HASH_A * colB;
    unsigned int gx = ( fx + HASH_B ) % HASH_P;

    if ( numReservedChunks == 0 )
    {
        for ( IndexType i = 0; i < NUM_HASH_RETRIES; i++ )
        {
            int hash = ( gx + HASH_C0 * i + HASH_C1 * ( IndexType ) i * i ) % NUM_ELEMENTS_IN_SHARED;

            IndexType val = atomicCAS( &sHashTableIndexes[hash], -1, colB );

            if ( val == -1 )
            {
                atomicAdd( &cIA[aRowIt], 1 );
                return true;
            }

            if ( val == colB )
            {
                return true;
            }
        }

        return false;
    }

    for ( IndexType i = 0; i < NUM_HASH_RETRIES; i++ )
    {
        int globalHash = ( gx + HASH_C0 * i + HASH_C1 * ( IndexType ) i * i ) % ( NUM_ELEMENTS_PER_CHUNK * numReservedChunks );
        int localHash = globalHash % NUM_ELEMENTS_PER_CHUNK;
        int chunk = globalHash / NUM_ELEMENTS_PER_CHUNK;

        IndexType val = atomicCAS( &chunkPtr[chunkList[chunk] * NUM_ELEMENTS_PER_CHUNK + localHash], -1, colB );

        if ( val == -1 )
        {
            atomicAdd( &cIA[aRowIt], 1 );
            return true;
        }

        if ( val == colB )
        {
            return true;
        }
    }

    return false;
}

template <typename ValueType>
__device__
inline bool multHlp_insertValues( IndexType colB,
                                  IndexType* sHashTableIndexes,
                                  ValueType* sHashTableValues,
                                  IndexType* indexChunks,
                                  ValueType* valueChunks,
                                  volatile int chunkList[],
                                  int numReservedChunks,
                                  ValueType valB,
                                  ValueType sValA )
{
    unsigned int fx = HASH_A * colB;
    unsigned int gx = ( fx + HASH_B ) % HASH_P;

    if ( numReservedChunks == 0 )
    {
        for ( IndexType i = 0; i < NUM_HASH_RETRIES; i++ )
        {
            int hash = ( gx + HASH_C0 * i + HASH_C1 * ( IndexType ) i * i ) % NUM_ELEMENTS_IN_SHARED;

            IndexType val = atomicCAS( &sHashTableIndexes[hash], -1, colB );

            if ( val == -1 )
            {
                sHashTableValues[hash] = valB * sValA;
                return true;
            }

            if ( val == colB )
            {
                sHashTableValues[hash] += valB * sValA;
                return true;
            }
        }

        return false;
    }

    for ( IndexType i = 0; i < NUM_HASH_RETRIES; i++ )
    {
        int globalHash = ( gx + HASH_C0 * i + HASH_C1 * ( IndexType ) i * i ) % ( NUM_ELEMENTS_PER_CHUNK * numReservedChunks );
        int localHash = globalHash % NUM_ELEMENTS_PER_CHUNK;
        int chunk = globalHash / NUM_ELEMENTS_PER_CHUNK;

        IndexType val = atomicCAS( &indexChunks[chunkList[chunk] * NUM_ELEMENTS_PER_CHUNK + localHash], -1, colB );

        if ( val == -1 )
        {
            valueChunks[chunkList[chunk] * NUM_ELEMENTS_PER_CHUNK + localHash] = sValA * valB;
            return true;
        }

        if ( val == colB )
        {
            valueChunks[chunkList[chunk] * NUM_ELEMENTS_PER_CHUNK + localHash] += sValA * valB;
            return true;
        }
    }

    return false;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiplySizes                                                    */
/* ------------------------------------------------------------------------------------------------------------------ */

__device__
inline bool multHlp_nextRow( IndexType* row,
                             IndexType numRows
#ifdef USE_LOAD_BALANCING
                             , IndexType* rowCounter
#endif
                           )
{
#ifdef USE_LOAD_BALANCING
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
    IndexType localWarpId = threadIdx.x / warpSize;
    __shared__ volatile int sRowIt[NUM_WARPS];

    if ( laneId == 0 )
    {
        sRowIt[localWarpId] = atomicAdd( rowCounter, 1 );
    }

    *row = sRowIt[localWarpId];

    if ( *row < numRows )
    {
        return true;
    }
    else
    {
        return false;
    }

#else
    IndexType numWarpsGlobal = ( blockDim.x * gridDim.x ) / warpSize;

    *row += numWarpsGlobal;

    if ( *row < numRows )
    {
        return true;
    }
    else
    {
        return false;
    }

#endif
}

__device__
inline void multHlp_releaseChunks ( IndexType* chunkList,
                                    volatile IndexType* sChunkList,
                                    volatile IndexType* sReservedChunks,
                                    IndexType chunkCount )
{
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;

    if ( laneId == 0 )
    {
        for ( int i = *sReservedChunks - 1; i >= *sReservedChunks - chunkCount; --i )
        {
            IndexType headItem;
            IndexType old;

            do
            {
                headItem = chunkList[0];
                chunkList[sChunkList[i] + 1] = headItem;
                old = atomicCAS( const_cast<int*>( &chunkList[0] ), headItem, sChunkList[i] );
            }
            while ( old != headItem );
        }
    }

    *sReservedChunks = *sReservedChunks - chunkCount;
}

__device__
inline bool multHlp_reserveChunks( IndexType* chunkList,
                                   volatile IndexType* sChunkList,
                                   volatile IndexType* sReservedChunks,
                                   IndexType chunkCount )
{
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;

    if ( chunkCount > NUM_CHUNKS_PER_WARP )
    {
//        printf("to many chunks %i\n", chunkCount);
        return false;
    }

    if ( laneId == 0 && chunkCount > 0 && *sReservedChunks != chunkCount )
    {
        if ( *sReservedChunks < chunkCount )
        {
            for ( int i = *sReservedChunks; i < chunkCount; ++i )
            {
                IndexType headItem;
                IndexType nextItem;
                IndexType old;

                do
                {
                    headItem = chunkList[0];

                    if ( headItem != -1 )
                    {
                        __threadfence();
                        nextItem = chunkList[headItem + 1];
                        old = atomicCAS( const_cast<int*>( &chunkList[0] ), headItem, nextItem );

                        if ( old == headItem )
                        {
                            sChunkList[i] = headItem;
                        }
                    }
                    else
                    {
//                        printf("no more chunks!\n");
                        return false;
                    }

                }
                while ( old != headItem );
            }

            *sReservedChunks = chunkCount;
            return true;
        }
        else
        {
            multHlp_releaseChunks ( chunkList, sChunkList, sReservedChunks, *sReservedChunks - chunkCount );
            return true;
        }
    }
    else
    {
        return true;
    }
}

__device__
inline void multHlp_initializeChunks ( IndexType* sHashTable,
                                       IndexType* chunks,
                                       const IndexType numElementsPerChunk,
                                       volatile IndexType* sChunkList,
                                       volatile IndexType sReservedChunks )
{
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;

    if ( sReservedChunks == 0 )
    {
        for ( IndexType i = 0; i < NUM_ELEMENTS_IN_SHARED; i += warpSize )
        {
            if ( i + laneId < NUM_ELEMENTS_IN_SHARED )
            {
                sHashTable[i + laneId] = -1;
            }
        }

        return;
    }

    for ( int i = 0; i < sReservedChunks; ++i )
    {
        int chunkId = sChunkList[i];

        for ( int j = laneId; j < numElementsPerChunk; j += warpSize )
        {
            chunks[chunkId * numElementsPerChunk + j] = -1;
        }
    }
}

__device__
inline IndexType multHlp_growth ( IndexType numChunks )
{
    if ( numChunks == 0 )
    {
        return 2;
    }
    else
    {
        return numChunks * 2;
    }
}

__device__
inline IndexType multHlp_calcOptChunkCount ( IndexType row,
        const IndexType* cIA,
        const IndexType numElementsPerChunk )
{
    IndexType numElements = cIA[row + 1] - cIA[row];

    if ( numElements * 2 < NUM_ELEMENTS_IN_SHARED )
    {
        return 0;
    }
    else
    {
        return ( ( ( cIA[row + 1] - cIA[row] ) * 2 ) / numElementsPerChunk ) + 1;
    }
}

__global__
void matrixMultiplySizesKernel(
    const IndexType* aIA,
    const IndexType* aJA,
    const IndexType* bIA,
    const IndexType* bJA,
    IndexType* cIA,
    const IndexType numRows,
    const IndexType numColumns,
    IndexType* chunkPtr,
    IndexType* chunkList,
    IndexType numChunks,
    bool* hashError,
    bool diagonalProperty )
{
    __shared__ IndexType sHashTable[NUM_ELEMENTS_IN_SHARED];
    __shared__ volatile int sReservedChunks;
    __shared__ volatile int sChunkList[NUM_CHUNKS_PER_WARP];
    __shared__ volatile IndexType sColA;
    __shared__ volatile int sRowIt;
    __shared__ volatile bool sInsertMiss;

    IndexType globalWarpId = ( blockIdx.x * blockDim.x + threadIdx.x ) / warpSize;
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
    IndexType colB;
    IndexType aRowIt = globalWarpId;
    bool localSystemError = false;

    sReservedChunks = 0;

    if ( aRowIt < numRows )
    {
        do
        {
            do
            {
                sInsertMiss = false;

                IndexType aColIt = aIA[aRowIt] + laneId;
                IndexType aColEnd = aIA[aRowIt + 1];

                if ( laneId == 0 && diagonalProperty )
                {
                    cIA[aRowIt]++;
                }

                multHlp_initializeChunks( sHashTable,
                                          chunkPtr,
                                          NUM_ELEMENTS_PER_CHUNK,
                                          sChunkList,
                                          sReservedChunks );

                for ( IndexType offset = 0; __any( aColIt < aColEnd ); aColIt += warpSize, offset += warpSize )
                {
                    IndexType colA = aColIt < aColEnd ? aJA[aColIt] : -1;

                    IndexType end = multHlp_getNumActiveThreads( aColIt, aColEnd, aIA, aRowIt, offset );

                    for ( IndexType k = 0; k < end && k < warpSize; k++ )
                    {
                        if ( laneId == k )
                        {
                            sColA = colA;
                        }

                        IndexType bColIt = bIA[sColA] + laneId;
                        IndexType bColEnd = bIA[sColA + 1];

                        for ( ; __any( bColIt < bColEnd ); bColIt += warpSize )
                        {
                            colB = bColIt < bColEnd ? bJA[bColIt] : -1;

                            if ( colB != -1 && ( !diagonalProperty || colB != aRowIt ) )
                            {
                                bool inserted = multHlp_insertIndexex( colB,
                                                                       sHashTable,
                                                                       aRowIt,
                                                                       chunkPtr,
                                                                       sChunkList,
                                                                       sReservedChunks,
                                                                       cIA );

                                if ( !inserted )
                                {
                                    sInsertMiss = true;
                                }
                            }
                        }
                    }
                }

                // only release if insertion was ok, otherwire reserve some more
                // STEP x: release reserved chunks
                if ( laneId == 0 )
                {
                    if ( sInsertMiss )
                    {
                        cIA[aRowIt] = 0;

                        if ( !multHlp_reserveChunks( chunkList, sChunkList, &sReservedChunks, multHlp_growth( sReservedChunks ) ) )
                        {
                            // ABORT KERNEL HERE;
                            localSystemError = true;
                        }
                    }
                }

                if ( __any( localSystemError ) )
                {
                    *hashError = true;
                    return;
                }
            }
            while ( sInsertMiss );
        }
        while ( multHlp_nextRow( &aRowIt, numRows ) );
    }

    // release all remaining chunks
    multHlp_releaseChunks( chunkList, sChunkList, &sReservedChunks, sReservedChunks );
}

struct multHlp_chunkFill
{
    const int n;
    multHlp_chunkFill( int _n )
        : n( _n )
    {
    }
    __device__
    IndexType operator()( int i )
    {
        if ( i == ( n - 1 ) )
        {
            return -1;
        }

        return i;
    }
};

IndexType CUDACSRUtils::matrixMultiplySizes(
    IndexType cIa[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType /* k */,
    bool diagonalProperty,
    const IndexType aIa[],
    const IndexType aJa[],
    const IndexType bIa[],
    const IndexType bJa[] )
{

    SCAI_REGION( "CUDA.CSR.matrixMultiplySizes" )

    SCAI_LOG_INFO(
        logger,
        "matrixMutliplySizes for " << numRows << " x " << numColumns << " matrix" << ", diagonalProperty = " << diagonalProperty )

    SCAI_CHECK_CUDA_ACCESS

    // Reset cIa
    thrust::device_ptr<IndexType> cIaPtr( cIa );
    thrust::fill( cIaPtr, cIaPtr + numRows, 0 );

    ContextPtr loc = Context::getContextPtr( context::CUDA );
    MemoryPtr mem = loc->getMemoryPtr();

    bool hashErrorHost = false;
    bool* hashError = ( bool* ) mem->allocate( sizeof( bool ) );
    hipMemcpy( hashError, &hashErrorHost, sizeof( bool ), hipMemcpyHostToDevice );

    size_t free;
    size_t total;
    hipMemGetInfo( &free, &total );

    int nnz_a;
    int nnz_b;
    hipMemcpy( &nnz_a, &aIa[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnz_b, &bIa[numColumns], sizeof( IndexType ), hipMemcpyDeviceToHost );
    int avgDensity = ( nnz_a / numRows + nnz_b / numColumns ) / 2;

    int numChunks;
    int maxNumChunks = ( free - ( 100 * 1024 * 1024 ) ) / ( NUM_ELEMENTS_PER_CHUNK * sizeof ( IndexType ) * 2 );
    int chunksPerWarp = NUM_BLOCKS * ( ( avgDensity * 8 ) / NUM_ELEMENTS_PER_CHUNK + 1 );

    if ( chunksPerWarp > maxNumChunks )
    {
        numChunks = maxNumChunks;
    }
    else
    {
        numChunks = chunksPerWarp;
    }

    unsigned int hashTableAllocatedBytes = numChunks * NUM_ELEMENTS_PER_CHUNK * sizeof( IndexType );
    IndexType* hashTable = ( IndexType* ) mem->allocate( hashTableAllocatedBytes );

    // chunkList table needs one integers per chunk plus 1 start pointer
    unsigned int chunkListAllocatedBytes = numChunks * sizeof( IndexType ) + sizeof( IndexType );
    IndexType* chunkList = ( IndexType* ) mem->allocate( chunkListAllocatedBytes );

    thrust::device_ptr<IndexType> chunkListPtr( chunkList );
    thrust::transform( thrust::make_counting_iterator( 0 ),
                       thrust::make_counting_iterator( numChunks + 1 ),
                       chunkListPtr,
                       multHlp_chunkFill( numChunks + 1 ) );

    matrixMultiplySizesKernel <<< NUM_BLOCKS, NUM_THREADS>>>( aIa,
            aJa,
            bIa,
            bJa,
            cIa,
            numRows,
            numColumns,
            hashTable,
            chunkList,
            numChunks,
            hashError,
            diagonalProperty );

    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR

    hipMemcpy( &hashErrorHost, hashError, sizeof( bool ), hipMemcpyDeviceToHost );

    if ( hashErrorHost )
    {
        COMMON_THROWEXCEPTION( "Multiplication failed!" );
    }

    // Free hashTable and hashError
    mem->free( ( void* ) hashError, sizeof( bool ) );
    mem->free( ( void* ) hashTable, hashTableAllocatedBytes );
    mem->free( ( void* ) chunkList, chunkListAllocatedBytes );

    // Convert sizes array to offset array
    thrust::exclusive_scan( cIaPtr, cIaPtr + numRows + 1, cIaPtr );

    IndexType numValues;
    hipMemcpy( &numValues, &cIa[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );

    return numValues;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAdd                                                              */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType, int nWarps>
__global__
void matrixAddKernel(
    IndexType* cJA,
    ValueType* cValues,
    const IndexType* cIA,
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const ValueType alpha,
    const IndexType* aIA,
    const IndexType* aJA,
    const ValueType* aValues,
    const ValueType beta,
    const IndexType* bIA,
    const IndexType* bJA,
    const ValueType* bValues )
{
// TODO: Just naive implementation, could be done faster, but works!
// TODO: Check if diagonal property needs special attention
    __shared__ volatile IndexType sColA[nWarps];
    __shared__ volatile ValueType sValA[nWarps];
    __shared__ volatile IndexType sFoundJa[nWarps];

    IndexType localWarpId = threadIdx.x / warpSize;
    IndexType globalWarpId = ( blockIdx.x * blockDim.x + threadIdx.x ) / warpSize;
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;
//IndexType numWarpsLocal  = blockDim.x / warpSize;
    IndexType numWarpsGlobal = ( blockDim.x * gridDim.x ) / warpSize;
    IndexType rowIt = globalWarpId;

    for ( ; __any( rowIt < numRows ); rowIt += numWarpsGlobal )
    {
        if ( rowIt < numRows )
        {
            if ( diagonalProperty && rowIt >= numColumns )
            {
                diagonalProperty = false;
            }

            IndexType aColIt = aIA[rowIt] + laneId;
            IndexType aColEnd = aIA[rowIt + 1];

            IndexType bColIt = bIA[rowIt] + laneId;
            IndexType bColEnd = bIA[rowIt + 1];

            IndexType cColIt = cIA[rowIt] + laneId;

// Copy values of b to C
            for ( IndexType bColOffset = 0; __any( ( bColIt + bColOffset ) < bColEnd ); bColOffset += warpSize )
            {
                IndexType colB = ( bColIt + bColOffset ) < bColEnd ? bJA[bColIt + bColOffset] : -1;
                ValueType valB = ( bColIt + bColOffset ) < bColEnd ? bValues[bColIt + bColOffset] : static_cast<ValueType>( 0 );

                if ( colB != -1 )
                {
                    cJA[cColIt + bColOffset] = colB;
                    cValues[cColIt + bColOffset] = valB * beta;
                }
            }

// Offset in c after coping b to c
            IndexType cColOffset = bIA[rowIt + 1] - bIA[rowIt];

// Add values of a to c
            for ( IndexType aColItOffset = 0; __any( aColIt < aColEnd ); aColIt += warpSize, aColItOffset += warpSize )
            {
                IndexType colA = aColIt < aColEnd ? aJA[aColIt] : -1;
                ValueType valA = aColIt < aColEnd ? aValues[aColIt] : static_cast<ValueType>( 0 );
                IndexType end = multHlp_getNumActiveThreads( aColIt, aColEnd, aIA, rowIt, aColItOffset );

                for ( IndexType k = 0; k < end && k < warpSize; k++ )
                {
                    if ( laneId == k )
                    {
                        sColA[localWarpId] = colA;
                        sValA[localWarpId] = valA;
                        sFoundJa[localWarpId] = -1;
                    }

                    for ( IndexType bColItOffset = 0; ( sFoundJa[localWarpId] == -1 ) && __any( ( bColIt + bColItOffset ) < bColEnd );
                            bColItOffset += warpSize )
                    {
                        IndexType colB = ( bColIt + bColItOffset ) < bColEnd ? bJA[bColIt + bColItOffset] : -1;

                        if ( sColA[localWarpId] == colB )
                        {
                            sFoundJa[localWarpId] = laneId + bColItOffset;
                        }
                    }

                    if ( laneId == 0 )
                    {
                        if ( sFoundJa[localWarpId] == -1 )
                        {
                            // Element is new element, add new element
                            cJA[cColIt + cColOffset] = colA;
                            cValues[cColIt + cColOffset] = sValA[localWarpId] * alpha;
                            cColOffset++;
                        }
                        else
                        {
                            // Element exists, add values
                            // We can use cColIt, because this is thread with laneId = 0!
                            cValues[cColIt + sFoundJa[localWarpId]] += sValA[localWarpId] * alpha;
                        }
                    }
                }
            }
        }
    }
}

template<typename ValueType>
void CUDACSRUtils::matrixAdd(
    IndexType cJA[],
    ValueType cValues[],
    const IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const ValueType alpha,
    const IndexType aIA[],
    const IndexType aJA[],
    const ValueType aValues[],
    const ValueType beta,
    const IndexType bIA[],
    const IndexType bJA[],
    const ValueType bValues[] )
{
    SCAI_REGION( "CUDA.CSRUtils.matrixAdd" )

    SCAI_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )

    SCAI_CHECK_CUDA_ACCESS

    matrixAddKernel<ValueType, NUM_WARPS> <<< NUM_BLOCKS, NUM_THREADS>>>( cJA, cValues, cIA, numRows, numColumns,
            diagonalProperty, alpha, aIA, aJA, aValues, beta, bIA, bJA, bValues );

    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR
}
/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiply                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template <typename ValueType>
__device__
inline void multHlp_copyHashtable ( volatile IndexType* sColA,
                                    const IndexType* cIA,
                                    IndexType laneId,
                                    IndexType aRowIt,
                                    const ValueType alpha,
                                    IndexType* cJA,
                                    ValueType* cValues,
                                    IndexType* sHashTableIndexes,
                                    ValueType* sHashTableValues,
                                    IndexType* indexChunks,
                                    ValueType* valueChunks,
                                    volatile int chunkList[],
                                    int numReservedChunks,
                                    bool diagonalProperty,
                                    ValueType diagonalElement )

{
    // TODO: rename sColA => destinationOffset!
    *sColA = 0;
    IndexType rowOffset = cIA[aRowIt];

    IndexType hashCol;
    ValueType hashVal;

    if ( diagonalProperty && laneId == 0 )
    {
        cJA[rowOffset] = aRowIt;
        cValues[rowOffset] = diagonalElement * alpha;
        *sColA = 1;
    }

    if ( numReservedChunks == 0 )
    {
        for ( int j = laneId; j < NUM_ELEMENTS_IN_SHARED; j += warpSize )
        {
            hashCol = sHashTableIndexes[j];
            hashVal = sHashTableValues[j];

#if SCAI_CUDA_COMPUTE_CAPABILITY >= 20
            IndexType localOffset;

            // TODO: be carefull here, ballot is warpsize Bit's long!
            IndexType ballot = __ballot ( hashCol != -1 );
            localOffset = __popc( ballot << ( warpSize - laneId ) );

            if ( hashCol != -1 )
            {
                cJA[rowOffset + *sColA + localOffset] = hashCol;
                cValues[rowOffset + *sColA + localOffset] = hashVal * alpha;
            }

            *sColA += __popc( ballot );
#else

            if ( hashCol != -1 )
            {
                IndexType offset = atomicAdd( ( int* )sColA, 1 );

                cJA[rowOffset + offset] = hashCol;
                cValues[rowOffset + offset] = hashVal * alpha;

            }

#endif
        }

        return;
    }

    for ( int i = 0; i < numReservedChunks; ++i )
    {
        for ( int j = laneId; j < NUM_ELEMENTS_PER_CHUNK; j += warpSize )
        {
            hashCol = indexChunks[chunkList[i] * NUM_ELEMENTS_PER_CHUNK + j];
            hashVal = valueChunks[chunkList[i] * NUM_ELEMENTS_PER_CHUNK + j];

#if SCAI_CUDA_COMPUTE_CAPABILITY >= 20
            IndexType localOffset;

            // TODO: be carefull here, ballot is warpsize Bit's long!
            IndexType ballot = __ballot ( hashCol != -1 );

            localOffset = __popc( ballot << ( warpSize - laneId ) );

            if ( hashCol != -1 )
            {
                cJA[rowOffset + *sColA + localOffset] = hashCol;
                cValues[rowOffset + *sColA + localOffset] = hashVal * alpha;
            }

            if ( laneId == 0 )
            {
                *sColA += __popc( ballot );
            }

#else

            if ( hashCol != -1 )
            {
                IndexType offset = atomicAdd( ( int* )sColA, 1 );

                cJA[rowOffset + offset] = hashCol;
                cValues[rowOffset + offset] = hashVal * alpha;

            }

#endif
        }
    }
}

template<typename ValueType>
__global__
void matrixMultiplyKernel(
    const IndexType* aIA,
    const IndexType* aJA,
    const ValueType* aValues,
    const IndexType* bIA,
    const IndexType* bJA,
    const ValueType* bValues,
    const IndexType* cIA,
    const ValueType alpha,
    IndexType* cJA,
    ValueType* cValues,
    const IndexType numRows,
    const IndexType numColumns,
    IndexType* indexChunks,
    ValueType* valueChunks,
    IndexType* chunkList,
    const int numChunks,
    bool* hashError,
    bool diagonalProperty )
{
    __shared__ IndexType sHashTableIndexes[NUM_ELEMENTS_IN_SHARED];
    __shared__ ValueType sHashTableValues[NUM_ELEMENTS_IN_SHARED];
    __shared__ volatile int sReservedChunks;
    __shared__ volatile IndexType sChunkList[NUM_CHUNKS_PER_WARP];
    __shared__ volatile IndexType sColA;
    __shared__ volatile ValueType sValA;
    __shared__ volatile int sRowIt;
    __shared__ volatile bool sInsertMiss;
    __shared__ volatile ValueType diagonalElement;

    IndexType globalWarpId = ( blockIdx.x * blockDim.x + threadIdx.x ) / warpSize;
    IndexType laneId = ( blockIdx.x * blockDim.x + threadIdx.x ) % warpSize;

    IndexType colB;
    IndexType aRowIt = globalWarpId;
    bool localSystemError = false;

    sReservedChunks = 0;

    if ( aRowIt < numRows )
    {
        do
        {
            IndexType optimalChunkCount = multHlp_calcOptChunkCount ( aRowIt, cIA, NUM_ELEMENTS_PER_CHUNK );

            // reserve Chunks
            if ( !multHlp_reserveChunks( chunkList, sChunkList, &sReservedChunks, optimalChunkCount ) )
            {
                // ABORT KERNEL HERE;
                localSystemError = true;
            }

            if ( __any( localSystemError ) )
            {
                *hashError = true;
                return;
            }

            do
            {
                sInsertMiss = false;
                IndexType aColIt = aIA[aRowIt] + laneId;
                IndexType aColEnd = aIA[aRowIt + 1];

                if ( laneId == 0 && diagonalProperty )
                {
                    diagonalElement = 0.0;
                }

                multHlp_initializeChunks( sHashTableIndexes,
                                          indexChunks,
                                          NUM_ELEMENTS_PER_CHUNK,
                                          sChunkList,
                                          sReservedChunks );

                for ( IndexType offset = 0; __any( aColIt < aColEnd ); aColIt += warpSize, offset += warpSize )
                {
                    IndexType colA = aColIt < aColEnd ? aJA[aColIt] : -1;
                    ValueType valA = aColIt < aColEnd ? aValues[aColIt] : static_cast<ValueType>( 0 );

                    IndexType end = multHlp_getNumActiveThreads( aColIt, aColEnd, aIA, aRowIt, offset );

                    for ( IndexType k = 0; k < end && k < warpSize; k++ )
                    {
                        if ( laneId == k )
                        {
                            sColA = colA;
                            sValA = valA;
                        }

                        IndexType bColIt = bIA[sColA] + laneId;
                        IndexType bColEnd = bIA[sColA + 1];

                        for ( ; __any( bColIt < bColEnd ); bColIt += warpSize )
                        {
                            colB = bColIt < bColEnd ? bJA[bColIt] : -1;
                            ValueType valB = bColIt < bColEnd ? bValues[bColIt] : static_cast<ValueType>( 0 );

                            if ( diagonalProperty && colB == aRowIt )
                            {
                                diagonalElement += sValA * valB;
                            }
                            else
                            {

                                if ( colB != -1 && ( !diagonalProperty || colB != aRowIt ) )
                                {
                                    bool inserted = multHlp_insertValues( colB,
                                                                          sHashTableIndexes,
                                                                          sHashTableValues,
                                                                          indexChunks,
                                                                          valueChunks,
                                                                          sChunkList,
                                                                          sReservedChunks,
                                                                          valB,
                                                                          sValA );

                                    if ( !inserted )
                                    {
                                        sInsertMiss = true;
                                    }
                                }
                            }
                        }
                    }
                }

                if ( !sInsertMiss )
                {
                    multHlp_copyHashtable ( &sColA,
                                            cIA,
                                            laneId,
                                            aRowIt,
                                            alpha,
                                            cJA,
                                            cValues,
                                            sHashTableIndexes,
                                            sHashTableValues,
                                            indexChunks,
                                            valueChunks,
                                            sChunkList,
                                            sReservedChunks,
                                            diagonalProperty,
                                            diagonalElement );
                }
                else
                {
                    if ( !multHlp_reserveChunks( chunkList, sChunkList, &sReservedChunks, multHlp_growth( sReservedChunks ) ) )
                    {
                        // ABORT KERNEL HERE;
                        localSystemError = true;
                    }

                    if ( __any( localSystemError ) )
                    {
                        *hashError = true;
                        return;
                    }
                }
            }
            while ( sInsertMiss );

        }
        while ( multHlp_nextRow( &aRowIt, numRows ) );
    }

    // release all remaining chunks
    multHlp_releaseChunks( chunkList, sChunkList, &sReservedChunks, sReservedChunks );
}

template<typename ValueType>
void CUDACSRUtils::matrixMultiply(
    const IndexType cIa[],
    IndexType cJa[],
    ValueType cValues[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType /* k */,
    const ValueType alpha,
    bool diagonalProperty,
    const IndexType aIa[],
    const IndexType aJa[],
    const ValueType aValues[],
    const IndexType bIa[],
    const IndexType bJa[],
    const ValueType bValues[] )
{
    SCAI_REGION( "CUDA.CSRUtils.matrixMultiply" )

    SCAI_LOG_INFO( logger, "matrixMultiply for " << numRows << "x" << numColumns << " matrix" )

    SCAI_CHECK_CUDA_ACCESS

    ContextPtr loc = Context::getContextPtr( context::CUDA );
    MemoryPtr mem = loc->getMemoryPtr();

    bool hashErrorHost = false;
    bool* hashError = ( bool* ) mem->allocate( sizeof( bool ) );
    hipMemcpy( hashError, &hashErrorHost, sizeof( bool ), hipMemcpyHostToDevice );

    size_t free;
    size_t total;
    hipMemGetInfo( &free, &total );

    int nnz_a;
    int nnz_b;
    hipMemcpy( &nnz_a, &aIa[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnz_b, &bIa[numColumns], sizeof( IndexType ), hipMemcpyDeviceToHost );
    int avgDensity = ( nnz_a / numRows + nnz_b / numColumns ) / 2;

    int numChunks;
    int maxNumChunks = ( free - ( 100 * 1024 * 1024 ) ) / ( NUM_ELEMENTS_PER_CHUNK * sizeof ( IndexType ) * 2 );
    int chunksPerWarp = NUM_BLOCKS * ( ( avgDensity * 8 ) / NUM_ELEMENTS_PER_CHUNK + 1 );

    if ( chunksPerWarp > maxNumChunks )
    {
        numChunks = maxNumChunks;
    }
    else
    {
        numChunks = chunksPerWarp;
    }

    unsigned int hashTableAllocatedBytes = numChunks * NUM_ELEMENTS_PER_CHUNK * ( sizeof( IndexType ) + sizeof( ValueType ) );
    void* chunks = ( void* ) mem->allocate( hashTableAllocatedBytes );

    IndexType* indexChunks = ( IndexType* ) chunks;
    ValueType* valueChunks = ( ValueType* ) ( indexChunks + numChunks * NUM_ELEMENTS_PER_CHUNK );

    // chunkList table needs one integers per chunk plus 1 start pointer
    unsigned int chunkListAllocatedBytes = numChunks * sizeof( IndexType ) + sizeof( IndexType );
    IndexType* chunkList = ( IndexType* ) mem->allocate( chunkListAllocatedBytes );

    thrust::device_ptr<IndexType> chunkListPtr( chunkList );
    thrust::transform( thrust::make_counting_iterator( 0 ),
                       thrust::make_counting_iterator( numChunks + 1 ),
                       chunkListPtr,
                       multHlp_chunkFill( numChunks + 1 ) );

    matrixMultiplyKernel <<< NUM_BLOCKS, NUM_THREADS>>>( aIa,
            aJa,
            aValues,
            bIa,
            bJa,
            bValues,
            cIa,
            alpha,
            cJa,
            cValues,
            numRows,
            numColumns,
            indexChunks,
            valueChunks,
            chunkList,
            numChunks,
            hashError,
            diagonalProperty );

    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR

    hipMemcpy( &hashErrorHost, hashError, sizeof( bool ), hipMemcpyDeviceToHost );

    if ( hashErrorHost )
    {
        COMMON_THROWEXCEPTION( "Multiplication failed!" );
    }

    // Free hashTable and hashError
    mem->free( ( void* ) hashError, sizeof( bool ) );
    mem->free( ( void* ) chunks, hashTableAllocatedBytes );
    mem->free( ( void* ) chunkList, chunkListAllocatedBytes );

    hipStreamSynchronize( 0 );
    SCAI_CHECK_CUDA_ERROR
}

/* ------------------------------------------------------------------------------------------------------------------ */

/* --------------------------------------------------------------------------- */
/*     Template instantiations via registration routine                        */
/* --------------------------------------------------------------------------- */

void CUDACSRUtils::registerKernels( bool deleteFlag )
{
    SCAI_LOG_INFO( logger, "set CSR routines for CUDA in Interface" )

    using kregistry::KernelRegistry;
    using common::context::CUDA;

    KernelRegistry::KernelRegistryFlag flag = KernelRegistry::KERNEL_ADD ;   // lower priority

    if ( deleteFlag )
    {
        flag = KernelRegistry::KERNEL_ERASE;
    }

    // Instantations for IndexType, not done by ARITHMETIC_TYPE macrods

    KernelRegistry::set<CSRKernelTrait::sizes2offsets>( sizes2offsets, CUDA, flag );
    KernelRegistry::set<CSRKernelTrait::offsets2sizes>( offsets2sizes, CUDA, flag );
    KernelRegistry::set<CSRKernelTrait::hasDiagonalProperty>( hasDiagonalProperty, CUDA, flag );

    KernelRegistry::set<CSRKernelTrait::matrixAddSizes>( matrixAddSizes, CUDA, flag );
    KernelRegistry::set<CSRKernelTrait::matrixMultiplySizes>( matrixMultiplySizes, CUDA, flag );

#define LAMA_CSR_UTILS2_REGISTER(z, J, TYPE )                                                                   \
    KernelRegistry::set<CSRKernelTrait::scaleRows<TYPE, ARITHMETIC_CUDA_TYPE_##J> >( scaleRows, CUDA, flag );   \

#define LAMA_CSR_UTILS_REGISTER(z, I, _)                                                                                  \
    KernelRegistry::set<CSRKernelTrait::convertCSR2CSC<ARITHMETIC_CUDA_TYPE_##I> >( convertCSR2CSC, CUDA, flag );         \
    KernelRegistry::set<CSRKernelTrait::normalGEMV<ARITHMETIC_CUDA_TYPE_##I> >( normalGEMV, CUDA, flag );                 \
    KernelRegistry::set<CSRKernelTrait::sparseGEMV<ARITHMETIC_CUDA_TYPE_##I> >( sparseGEMV, CUDA, flag );                 \
    KernelRegistry::set<CSRKernelTrait::normalGEVM<ARITHMETIC_CUDA_TYPE_##I> >( normalGEVM, CUDA, flag );                 \
    KernelRegistry::set<CSRKernelTrait::sparseGEVM<ARITHMETIC_CUDA_TYPE_##I> >( sparseGEVM, CUDA, flag );                 \
    KernelRegistry::set<CSRKernelTrait::matrixAdd<ARITHMETIC_CUDA_TYPE_##I> >( matrixAdd, CUDA, flag );                   \
    KernelRegistry::set<CSRKernelTrait::matrixMultiply<ARITHMETIC_CUDA_TYPE_##I> >( matrixMultiply, CUDA, flag );         \
    KernelRegistry::set<CSRKernelTrait::jacobi<ARITHMETIC_CUDA_TYPE_##I> >( jacobi, CUDA, flag );                         \
    KernelRegistry::set<CSRKernelTrait::jacobiHalo<ARITHMETIC_CUDA_TYPE_##I> >( jacobiHalo, CUDA, flag );                 \
    KernelRegistry::set<CSRKernelTrait::jacobiHaloWithDiag<ARITHMETIC_CUDA_TYPE_##I> >( jacobiHaloWithDiag, CUDA, flag ); \
                                                                                                                          \
    BOOST_PP_REPEAT( ARITHMETIC_CUDA_TYPE_CNT,                                                                            \
                     LAMA_CSR_UTILS2_REGISTER,                                                                            \
                     ARITHMETIC_CUDA_TYPE_##I )                                                                           \

    BOOST_PP_REPEAT( ARITHMETIC_CUDA_TYPE_CNT, LAMA_CSR_UTILS_REGISTER, _ )

#undef LAMA_CSR_UTILS_REGISTER
#undef LAMA_CSR_UTILS2_REGISTER

}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUDACSRUtils::CUDACSRUtils()
{
    bool deleteFlag = false;
    registerKernels( deleteFlag );
}

CUDACSRUtils::~CUDACSRUtils()
{
    bool deleteFlag = true;
    registerKernels( deleteFlag );
}

CUDACSRUtils CUDACSRUtils::guard;    // guard variable for registration

/* --------------------------------------------------------------------------- */
/*    Static initialiazion at program start                                    */
/* --------------------------------------------------------------------------- */

unsigned int CUDACSRUtils::lastHashTableSize = 1024;

} /* end namespace lama */

} /* end namespace scai */
