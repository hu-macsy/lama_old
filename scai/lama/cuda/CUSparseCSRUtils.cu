/**
 * @file CUSparseCSRUtils.cu
 *
 * @license
 * Copyright (c) 2009-2013
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Implementation of some CSR routines with CUSparse library 5.0
 * @author Thomas Brandes
 * @date 11.06.2013
 * @since 1.0.1
 */

// hpp
#include <scai/lama/cuda/CUSparseCSRUtils.hpp>

// local library
#include <scai/lama/UtilKernelTrait.hpp>
#include <scai/lama/CSRKernelTrait.hpp>

// internal scai libraries
#include <scai/hmemo/cuda/CUDAStreamSyncToken.hpp>
#include <scai/kregistry/KernelRegistry.hpp>

#include <scai/tracing.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/Settings.hpp>

// CUDA
#include <hip/hip_runtime.h>
#include <hipsparse.h>

namespace scai
{

using tasking::CUDAStreamSyncToken;

/* --------------------------------------------------------------------------- */
/*     cusparse handle is needed, set by HIPContext                           */
/* --------------------------------------------------------------------------- */

extern hipsparseHandle_t CUDAContext_cusparseHandle;

namespace lama
{

SCAI_LOG_DEF_LOGGER( CUSparseCSRUtils::logger, "CUDA.CSRUtilsSparse" )

/* --------------------------------------------------------------------------- */
/*     Template specialization convertCSR2CSC<float>                           */
/* --------------------------------------------------------------------------- */

template<>
void CUSparseCSRUtils::convertCSR2CSC(
    IndexType cscIA[],
    IndexType cscJA[],
    float cscValues[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const float csrValues[],
    IndexType numRows,
    IndexType numColumns,
    IndexType numValues )
{
    SCAI_LOG_INFO( logger,
                   "convertCSR2CSC<float> -> hipsparseScsr2csc" << ", matrix size = "
                   << numRows << " x " << numColumns << ", nnz = " << numValues )

    SCAI_CUSPARSE_CALL(
        hipsparseScsr2csc( CUDAContext_cusparseHandle,
                          numRows, numColumns, numValues,
                          csrValues, csrIA, csrJA,
                          cscValues, cscJA, cscIA,
                          HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO ),
        "convertCSR2SCC<float>" )

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )
}

/* --------------------------------------------------------------------------- */
/*     Template specialization convertCSR2CSC<double>                          */
/* --------------------------------------------------------------------------- */

template<>
void CUSparseCSRUtils::convertCSR2CSC(
    IndexType cscIA[],
    IndexType cscJA[],
    double cscValues[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const double csrValues[],
    IndexType numRows,
    IndexType numColumns,
    IndexType numValues )
{
    SCAI_LOG_INFO( logger,
                   "convertCSR2CSC<double> -> hipsparseDcsr2csc" << ", matrix size = "
                   << numRows << " x " << numColumns << ", nnz = " << numValues )

    SCAI_CUSPARSE_CALL(
        hipsparseDcsr2csc( CUDAContext_cusparseHandle,
                          numRows, numColumns, numValues,
                          csrValues, csrIA, csrJA,
                          cscValues, cscJA, cscIA,
                          HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO ),
        "convertCSR2SCC<double>" )

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             normalGEMV                                                             */
/* ------------------------------------------------------------------------------------------------------------------ */

template<>
void CUSparseCSRUtils::normalGEMV(
    float result[],
    const float alpha,
    const float x[],
    const float beta,
    const float y[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType nnz,
    const IndexType csrIA[],
    const IndexType csrJA[],
    const float csrValues[] )
{
    SCAI_LOG_INFO( logger, "normalGEMV<float>" <<
                   " result[ " << numRows << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )

    SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0; // default stream if no syncToken is given

    hipsparseMatDescr_t descrCSR;

    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    if ( y != result && beta != 0.0f )
    {
        SCAI_CUDA_RT_CALL( hipMemcpy( result, y, numRows * sizeof( float ), hipMemcpyDeviceToDevice ),
                           "hipMemcpy for result = y" )
    }

    // call result = alpha * op(A) * x + beta * result of cusparse
    // Note: alpha, beta are passed as pointers

    SCAI_LOG_INFO( logger, "Start hipsparseScsrmv, stream = " << stream )

    SCAI_CUSPARSE_CALL( hipsparseScsrmv( CUDAContext_cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        numRows, numColumns, nnz, &alpha, descrCSR,
                                        csrValues, csrIA, csrJA, x, &beta, result ),
                        "hipsparseScsrmv" )

    if ( syncToken )
    {
        // set back stream for cusparse

        SCAI_CUSPARSE_CALL( hipsparseSetStream( CUDAContext_cusparseHandle, 0 ),
                            "hipsparseSetStream" )
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseXcsrgeamNnz" )
    }
}

template<>
void CUSparseCSRUtils::normalGEMV(
    double result[],
    const double alpha,
    const double x[],
    const double beta,
    const double y[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType nnz,
    const IndexType csrIA[],
    const IndexType csrJA[],
    const double csrValues[] )
{
    SCAI_LOG_INFO( logger, "normalGEMV<double>" <<
                   " result[ " << numRows << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )

    SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )

    SCAI_CHECK_CUDA_ACCESS

    hipStream_t stream = 0; // default stream if no syncToken is given

    hipsparseMatDescr_t descrCSR;

    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

    CUDAStreamSyncToken* syncToken = CUDAStreamSyncToken::getCurrentSyncToken();

    if ( syncToken )
    {
        stream = syncToken->getCUDAStream();
    }

    if ( y != result && beta != 0.0 )
    {
        SCAI_CUDA_RT_CALL( hipMemcpy( result, y, numRows * sizeof( double ), hipMemcpyDeviceToDevice ),
                           "hipMemcpy for result = y" )
    }

    // call result = alpha * op(A) * x + beta * result of cusparse
    // Note: alpha, beta are passed as pointers

    SCAI_LOG_INFO( logger, "Start hipsparseDcsrmv, stream = " << stream )

    SCAI_CUSPARSE_CALL( hipsparseDcsrmv( CUDAContext_cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        numRows, numColumns, nnz, &alpha, descrCSR,
                                        csrValues, csrIA, csrJA, x, &beta, result ),
                        "hipsparseScsrmv" )

    if ( syncToken )
    {
        // set back stream for cusparse

        SCAI_CUSPARSE_CALL( hipsparseSetStream( CUDAContext_cusparseHandle, 0 ),
                            "hipsparseSetStream" )
    }
    else
    {
        SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseXcsrgeamNnz" )
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAddSizes                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

IndexType CUSparseCSRUtils::matrixAddSizes(
    IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const IndexType bIA[],
    const IndexType bJA[] )
{
    SCAI_REGION( "CUDA.CSR.matrixAddSizes" )

    SCAI_LOG_INFO(
        logger,
        "matrixAddSizes for " << numRows << " x " << numColumns << " matrix" << ", diagonalProperty = " << diagonalProperty )

    SCAI_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ numColumns ]

    // we have not passed the values, so copy it from device to host

    hipMemcpy( &nnzA, &aIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );

    int nnzC;

    SCAI_CUSPARSE_CALL(
        hipsparseXcsrgeamNnz( CUDAContext_cusparseHandle,
                             numRows, numColumns,
                             descrCSR, nnzA, aIA, aJA,
                             descrCSR, nnzB, bIA, bJA,
                             descrCSR, cIA, &nnzC ),
        "hipsparseXcsrgeamNnz" )

    // synchronization might be redundant due to the return value

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseXcsrgeamNnz" )

    return nnzC;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiplySizes                                                    */
/* ------------------------------------------------------------------------------------------------------------------ */

IndexType CUSparseCSRUtils::matrixMultiplySizes(
    IndexType cIA[],
    const IndexType m,
    const IndexType n,
    const IndexType k,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const IndexType bIA[],
    const IndexType bJA[] )
{
    SCAI_REGION( "CUDA.CSR.matrixMultiplySizes" )

    SCAI_LOG_INFO(
        logger,
        "matrixMutliplySizes for " << m << " x " << n << " matrix" << ", diagonalProperty = " << diagonalProperty )

    SCAI_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ numColumns ]

    // we have not passed the values, so copy it

    hipMemcpy( &nnzA, &aIA[m], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[k], sizeof( IndexType ), hipMemcpyDeviceToHost );

    int nnzC;

    SCAI_LOG_DEBUG( logger, "multSizes, A is " << m << " x " << k << ", nnz = " << nnzA
                    << ", B is " << k << " x " << n << ", nnz = " << nnzB
                    << ", C = " << m << " x " << n )

    SCAI_CUSPARSE_CALL(
        hipsparseXcsrgemmNnz( CUDAContext_cusparseHandle,
                             HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             m, n, k,
                             descrCSR, nnzA, aIA, aJA,
                             descrCSR, nnzB, bIA, bJA,
                             descrCSR, cIA, &nnzC ),
        "hipsparseXcsrgemmNnz" )

    // synchronization might be redundant due to the return value

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )

    SCAI_LOG_DEBUG( logger, "matrixMultiplySizes, nnzC = " << nnzC )

    return nnzC;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAdd                                                              */
/* ------------------------------------------------------------------------------------------------------------------ */

template<>
void CUSparseCSRUtils::matrixAdd(
    IndexType cJA[],
    float cValues[],
    const IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const float alpha,
    const IndexType aIA[],
    const IndexType aJA[],
    const float aValues[],
    const float beta,
    const IndexType bIA[],
    const IndexType bJA[],
    const float bValues[] )
{
    SCAI_REGION( "CUDA.CSR.matrixAdd" )

    SCAI_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )

    SCAI_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ numColumns ]

    // we have not passed the values, so copy it

    hipMemcpy( &nnzA, &aIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );

    // cIA requires const_cast, but will not be modified

    SCAI_CUSPARSE_CALL(
        hipsparseScsrgeam( CUDAContext_cusparseHandle,
                          numRows, numColumns,
                          &alpha, descrCSR, nnzA, aValues, aIA, aJA,
                          &beta, descrCSR, nnzB, bValues, bIA, bJA,
                          descrCSR, cValues, const_cast<IndexType*>( cIA ), cJA ),
        "hipsparseScsrgeam" )

    // synchronization might be redundant

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseScsrgeam" )
}

template<>
void CUSparseCSRUtils::matrixAdd(
    IndexType cJA[],
    double cValues[],
    const IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const double alpha,
    const IndexType aIA[],
    const IndexType aJA[],
    const double aValues[],
    const double beta,
    const IndexType bIA[],
    const IndexType bJA[],
    const double bValues[] )
{
    SCAI_REGION( "CUDA.CSR.matrixAdd" )

    SCAI_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )

    SCAI_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

    int nnzA = 0; // aIA[ numRows ]
    int nnzB = 0;// bIA[ numColumns ]

    // we have not passed the number of non-zero values for A, B, so copy it

    hipMemcpy( &nnzA, &aIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );

    // cIA requires const_cast, but will not be modified

    SCAI_CUSPARSE_CALL(
        hipsparseDcsrgeam( CUDAContext_cusparseHandle,
                          numRows, numColumns,
                          &alpha, descrCSR, nnzA, aValues, aIA, aJA,
                          &beta, descrCSR, nnzB, bValues, bIA, bJA,
                          descrCSR, cValues, const_cast<IndexType*>( cIA ), cJA ),
        "hipsparseDcsrgeam" )

    // synchronization might be redundant

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseDcsrgeam" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiply                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template<>
void CUSparseCSRUtils::matrixMultiply(
    const IndexType cIA[],
    IndexType cJA[],
    float cValues[],
    const IndexType m,
    const IndexType n,
    const IndexType k,
    const float alpha,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const float aValues[],
    const IndexType bIA[],
    const IndexType bJA[],
    const float bValues[] )
{
    SCAI_REGION( "CUDA.CSR.matrixMultiply" )

    SCAI_LOG_INFO( logger, "matrixMultiply, result is " << m << "x" << n << " CSR storage" )

    SCAI_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ numColumns ]

    // we have not passed the number of non-zero values for A, B, so copy it

    hipMemcpy( &nnzA, &aIA[m], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[k], sizeof( IndexType ), hipMemcpyDeviceToHost );

    SCAI_ASSERT_EQUAL_ERROR( 0.0f, alpha );

    SCAI_CUSPARSE_CALL(
        hipsparseScsrgemm( CUDAContext_cusparseHandle,
                          HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          m, n, k,
                          descrCSR, nnzA, aValues, aIA, aJA,
                          descrCSR, nnzB, bValues, bIA, bJA,
                          descrCSR, cValues, cIA, cJA ),
        "hipsparseScsrgemm" )

    // synchronization might be redundant d

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csrSparseMatmulS" )
}

template<>
void CUSparseCSRUtils::matrixMultiply(
    const IndexType cIA[],
    IndexType cJA[],
    double cValues[],
    const IndexType m,
    const IndexType n,
    const IndexType k,
    const double alpha,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const double aValues[],
    const IndexType bIA[],
    const IndexType bJA[],
    const double bValues[] )
{
    SCAI_REGION( "CUDA.CSR.matrixMultiply" )

    SCAI_LOG_INFO( logger, "matrixMultiply, result is " << m << "x" << n << " CSR storage" )

    SCAI_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    SCAI_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

    int nnzA = 0; // aIA[ m ]
    int nnzB = 0;// bIA[ n ]

    // we have not passed the number of non-zero values for A, B, so copy it

    hipMemcpy( &nnzA, &aIA[m], sizeof( IndexType ), hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[k], sizeof( IndexType ), hipMemcpyDeviceToHost );

    SCAI_ASSERT_EQUAL_ERROR( 0.0, alpha );

    SCAI_CUSPARSE_CALL(
        hipsparseDcsrgemm( CUDAContext_cusparseHandle,
                          HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          m, n, k,
                          descrCSR, nnzA, aValues, aIA, aJA,
                          descrCSR, nnzB, bValues, bIA, bJA,
                          descrCSR, cValues, cIA, cJA ),
        "hipsparseDcsrgemm" )

    // synchronization might be redundant d

    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csrSparseMatmulD" )
}

/* ------------------------------------------------------------------------------------------------------------------ */

/* --------------------------------------------------------------------------- */
/*     Template instantiations via registration routine                        */
/* --------------------------------------------------------------------------- */

void CUSparseCSRUtils::registerKernels( bool deleteFlag )
{
    SCAI_LOG_INFO( logger, "set CSR routines for CUSparse in Interface" )

    bool useCUSparse = true;

    // using CUSparse for CSR might be disabled explicitly by environment variable

    common::Settings::getEnvironment( useCUSparse, "USE_CUSPARSE" );

    if ( !useCUSparse )
    {
        return;
    }

    // REGISTER1: overwrites previous settings

    using kregistry::KernelRegistry;
    using common::context::CUDA;

    KernelRegistry::KernelRegistryFlag flag = KernelRegistry::KERNEL_REPLACE;   // priority over OpenMPBLAS

    if ( deleteFlag )
    {
        flag = KernelRegistry::KERNEL_ERASE;
    }

    KernelRegistry::set<CSRKernelTrait::normalGEMV<float> >( normalGEMV, CUDA, flag );
    KernelRegistry::set<CSRKernelTrait::normalGEMV<double> >( normalGEMV, CUDA, flag );

    KernelRegistry::set<CSRKernelTrait::convertCSR2CSC<float> >( convertCSR2CSC, CUDA, flag );
    KernelRegistry::set<CSRKernelTrait::convertCSR2CSC<double> >( convertCSR2CSC, CUDA, flag );

    KernelRegistry::set<CSRKernelTrait::matrixAddSizes>( matrixAddSizes, CUDA, flag );
    KernelRegistry::set<CSRKernelTrait::matrixMultiplySizes>( matrixMultiplySizes, CUDA, flag );

    KernelRegistry::set<CSRKernelTrait::matrixAdd<float> >( matrixAdd, CUDA, flag );
    KernelRegistry::set<CSRKernelTrait::matrixAdd<double> >( matrixAdd, CUDA, flag );

    KernelRegistry::set<CSRKernelTrait::matrixMultiply<float> >( matrixMultiply, CUDA, flag );
    KernelRegistry::set<CSRKernelTrait::matrixMultiply<double> >( matrixMultiply, CUDA, flag );
}

/* --------------------------------------------------------------------------- */
/*    Constructor/Desctructor with registration                                */
/* --------------------------------------------------------------------------- */

CUSparseCSRUtils::CUSparseCSRUtils()
{
    bool deleteFlag = false;
    registerKernels( deleteFlag );
}

CUSparseCSRUtils::~CUSparseCSRUtils()
{
    bool deleteFlag = true;
    registerKernels( deleteFlag );
}

CUSparseCSRUtils CUSparseCSRUtils::guard;    // guard variable for registration

} /* end namespace lama */

} /* end namespace scai */
