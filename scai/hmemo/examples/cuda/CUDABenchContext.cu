#include "hip/hip_runtime.h"
/**
 * @file hmemo/examples/cuda/CUDABenchContext.cu
 *
 * @license
 * Copyright (c) 2009-2018
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * This file is part of the SCAI framework LAMA.
 *
 * LAMA is free software: you can redistribute it and/or modify it under the
 * terms of the GNU Lesser General Public License as published by the Free
 * Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * LAMA is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License for
 * more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with LAMA. If not, see <http://www.gnu.org/licenses/>.
 * @endlicense
 *
 * @brief ToDo: Missing description in ./hmemo/examples/cuda/CUDABenchContext.cu
 * @author Thomas Brandes
 * @date 21.07.2015
 */

#include <scai/hmemo.hpp>

#include <thrust/reduce.h>
#include <thrust/device_vector.h>

#include <scai/logging.hpp>

#include <scai/common/cuda/CUDAError.hpp>
#include <scai/common/Walltime.hpp>

#include <iostream>

using namespace scai;
using namespace scai::hmemo;

SCAI_LOG_DEF_LOGGER( logger, "CudaExample" )

template<typename ValueType>
ValueType sum( const ValueType array[], const IndexType n )
{
    thrust::device_ptr<ValueType> data( const_cast<ValueType*>( array ) );
    ValueType zero = static_cast<ValueType>( 0 );
    ValueType result = thrust::reduce( data, data + n, zero, thrust::plus<ValueType>() );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipStreamSynchronize( 0 )" );
    SCAI_LOG_INFO( logger, "sum of " << n << " values = " << result )
    return result;
}

template<typename ValueType>
__global__
void add_kernel( ValueType* array, IndexType n )
{
    const IndexType i = blockIdx.x * blockDim.x + threadIdx.x;
    ValueType one = 1;

    if ( i < n )
    {
        array[i] += one;
    }
}

template<typename ValueType>
void add( ValueType array[], const IndexType n )
{
    const int blockSize = 256;
    const int nblocks   = ( n + blockSize - 1 ) / blockSize;
    dim3 block( blockSize, 1, 1 );
    dim3 grid( nblocks, 1, 1 );
    add_kernel <<< grid, block>>>( array, n );
    SCAI_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "cuda failure" );
}

template<typename ValueType>
void addHost( ValueType array[], const IndexType n )
{
    for ( IndexType i = 0; i < n; ++i )
    {
        array[i] += 1;
    }
}

template<typename ValueType>
void doBench( HArray<ValueType>& array, const IndexType N )
{
    ContextPtr hostContext = Context::getContextPtr( common::ContextType::Host );
    ContextPtr cudaContext = Context::getContextPtr( common::ContextType::CUDA );
    int nhost = 1;
    int ncuda = 1;
    int niter = 50;
    double time = common::Walltime::get();
    // init on host
    {
        WriteOnlyAccess<double> write( array, hostContext, N );
        double* v = write.get();

        for ( IndexType i = 0; i < N; ++i )
        {
            v[i] = 0.0;
        }
    }

    for ( int iter = 0; iter < niter; ++iter )
    {
        // do some work on cuda
        for ( int k = 0; k < ncuda; ++k )
        {
            WriteAccess<double> write( array, cudaContext );
            SCAI_CONTEXT_ACCESS( cudaContext )
            add( write.get(), N );
        }

        // do some work on host

        for ( int k = 0; k < nhost; ++k )
        {
            WriteAccess<double> write( array, hostContext );
            addHost( write.get(), N );
        }
    }

    // compute result
    double res = 0.0;
    {
        ReadAccess<double> read( array, cudaContext );
        SCAI_CONTEXT_ACCESS( cudaContext )
        res = sum( read.get(), N );
    }
    double resExpected = N;
    resExpected *= double ( niter * ( ncuda + nhost ) );
    SCAI_ASSERT_EQUAL( res, resExpected, "wrong result, N = " << N
                       << ", niter = " << niter << ", ncuda = " << ncuda << ", nhost = " << nhost )
    time = common::Walltime::get() - time;
    std::cout << "Time = " << time << " seconds" << std::endl;
}

int main()
{
    const IndexType N = 8 * 1024 * 1024;  // 8 MB data
    ContextPtr hostContextPtr = Context::getContextPtr( common::ContextType::Host );
    ContextPtr cudaContextPtr = Context::getContextPtr( common::ContextType::CUDA );
    // First touch on host memory, never uses CUDA host memory
    std::cout << "Benchmark for array, first touch on host memory" << std::endl;
    HArray<double> data1( hostContextPtr->getMemoryPtr() );
    doBench( data1, N );
    std::cout << "Benchmark for array, first touch on cuda memory" << std::endl;
    HArray<double> data2( cudaContextPtr->getMemoryPtr() );
    doBench( data2, N );
    std::cout << "Benchmark for array, first touch on cuda host memory" << std::endl;
    HArray<double> data3( cudaContextPtr->getHostMemoryPtr() );
    doBench( data3, N );
}

